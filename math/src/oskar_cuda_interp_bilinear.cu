#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "math/oskar_cuda_interp_bilinear.h"
#include "utility/oskar_cuda_eclipse.h"

// Texture references must be static global variables,
// hence kernel and all device code is in this file.
static texture<float, 2> texture_ref_float;
static texture<float2, 2> texture_ref_float2;

// Template declaration: returns a reference to the texture.
template <typename T>
inline __device__ __host__ texture<T, 2>& texture_ref();

// Template specialisation: float.
template <>
inline __device__ __host__ texture<float, 2>& texture_ref<float>()
{
    return texture_ref_float;
}

// Template specialisation: float2.
template <>
inline __device__ __host__ texture<float2, 2>& texture_ref<float2>()
{
    return texture_ref_float2;
}

// Kernel.
template <typename InputType, typename CoordType, typename OutputType>
__global__ void oskar_bilinear_kernel(int n, const CoordType* pos_x,
        const CoordType* pos_y, OutputType* out)
{
    const int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i < n)
    {
        CoordType p_x = pos_x[i];
        CoordType p_y = pos_y[i];
        out[i] = tex2D(texture_ref<InputType>(), p_x, p_y);
    }
}

// Kernel template specialisation for complex double output.
template <>
__global__ void oskar_bilinear_kernel<float2, double, double2>(int n,
        const double* pos_x, const double* pos_y, double2* out)
{
    const int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (i < n)
    {
        float p_x = pos_x[i];
        float p_y = pos_y[i];
        float2 t1 = tex2D(texture_ref<float2>(), p_x, p_y);
        double2 t2 = make_double2(t1.x, t1.y);
        out[i] = t2;
    }
}

// Kernel wrapper.
template <typename InputType, typename CoordType, typename OutputType>
int oskar_math_cuda_interp_bilinear(int width, int height, int pitch,
        const InputType* input, int n, const CoordType* pos_x,
        const CoordType* pos_y, OutputType* output)
{
    // Prepare the texture reference from the look-up table.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<InputType>();
    texture<InputType, 2>& ref = texture_ref<InputType>();
    ref.filterMode = hipFilterModeLinear;
    ref.normalized = true;
    hipError_t errCuda = hipBindTexture2D(0, &ref, input, &channelDesc,
            width, height, pitch);
    if (errCuda != hipSuccess) return errCuda;

    // Launch the kernel.
    //const int thd = 768;
    const int thd = 512; // 768 dosn't work with CUDA 1.3
    const int blk = (n + thd - 1) / thd;
    oskar_bilinear_kernel<InputType, CoordType, OutputType> <<< blk, thd >>>
            (n, pos_x, pos_y, output);
    hipDeviceSynchronize();
    errCuda = hipPeekAtLastError();
    if (errCuda != hipSuccess) return errCuda;

    // Unbind texture.
    hipUnbindTexture(&ref);

    // Return 0 on success.
    return 0;
}

extern "C"
int oskar_cuda_interp_bilinear_f(int width, int height, int pitch,
        const float* input, int n, const float* pos_x, const float* pos_y,
        float* output)
{
    return oskar_math_cuda_interp_bilinear<float, float, float>(width,
            height, pitch, input, n, pos_x, pos_y, output);
}

extern "C"
int oskar_cuda_interp_bilinear_complex_f(int width, int height, int pitch,
        const float2* input, int n, const float* pos_x, const float* pos_y,
        float2* output)
{
    return oskar_math_cuda_interp_bilinear<float2, float, float2>(width,
            height, pitch, input, n, pos_x, pos_y, output);
}

extern "C"
int oskar_cuda_interp_bilinear_d(int width, int height, int pitch,
        const float* input, int n, const double* pos_x, const double* pos_y,
        double* output)
{
    return oskar_math_cuda_interp_bilinear<float, double, double>(width,
            height, pitch, input, n, pos_x, pos_y, output);
}

extern "C"
int oskar_cuda_interp_bilinear_complex_d(int width, int height, int pitch,
        const float2* input, int n, const double* pos_x, const double* pos_y,
        double2* output)
{
    return oskar_math_cuda_interp_bilinear<float2, double, double2>(width,
            height, pitch, input, n, pos_x, pos_y, output);
}
