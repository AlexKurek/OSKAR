#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "math/oskar_dierckx_bispev.h"
#include "math/oskar_spline_data_evaluate.h"
#include "math/cudak/oskar_cudak_dierckx_bispev_bicubic.h"
#include "utility/oskar_mem_type_check.h"

#ifdef __cplusplus
extern "C" {
#endif

int oskar_spline_data_evaluate(oskar_Mem* output, int offset, int stride,
        const oskar_SplineData* spline, const oskar_Mem* x, const oskar_Mem* y)
{
    int err = 0, nx, ny, num_points, type, location;

    /* Check arrays are consistent. */
    num_points = x->num_elements;
    if (y->num_elements != num_points)
        return OSKAR_ERR_DIMENSION_MISMATCH;

    /* Check type. */
    type = x->type;
    if (type != y->type)
        return OSKAR_ERR_TYPE_MISMATCH;

    /* Check location. */
    location = output->location;
    if (location != spline->coeff.location ||
            location != spline->knots_x.location ||
            location != spline->knots_y.location ||
            location != x->location ||
            location != y->location)
        return OSKAR_ERR_BAD_LOCATION;

    /* Check that the spline data has been set up. */
    if (!spline->coeff.data || !spline->knots_x.data || !spline->knots_y.data)
        return OSKAR_ERR_MEMORY_NOT_ALLOCATED;

    /* Check data type. */
    if (type == OSKAR_SINGLE)
    {
        const float *knots_x, *knots_y, *coeff;
        float *out;
        nx      = spline->num_knots_x;
        ny      = spline->num_knots_y;
        knots_x = (const float*)spline->knots_x.data;
        knots_y = (const float*)spline->knots_y.data;
        coeff   = (const float*)spline->coeff.data;
        out     = (float*)output->data + offset;

        /* Check if data are in CPU memory. */
        if (location == OSKAR_LOCATION_CPU)
        {
            /* Set up workspace. */
            float wrk[8];
            int i, iwrk1[2], kwrk1 = 2, lwrk = 8;

            /* Evaluate surface at the points. */
            for (i = 0; i < num_points; ++i)
            {
                float x1, y1;
                x1 = ((const float*)x->data)[i];
                y1 = ((const float*)y->data)[i];
                oskar_dierckx_bispev_f(knots_x, nx, knots_y, ny, coeff,
                        3, 3, &x1, 1, &y1, 1, &out[i * stride],
                        wrk, lwrk, iwrk1, kwrk1, &err);
                if (err != 0) return OSKAR_ERR_SPLINE_EVAL_FAIL;
            }
        }
        else if (location == OSKAR_LOCATION_GPU)
        {
            /* Evaluate surface at the points by calling kernel. */
            int num_blocks, num_threads = 256;
            num_blocks = (num_points + num_threads - 1) / num_threads;
            oskar_cudak_dierckx_bispev_bicubic_f
            OSKAR_CUDAK_CONF(num_blocks, num_threads) (knots_x,
                    nx, knots_y, ny, coeff, num_points,
                    (const float*)x->data, (const float*)y->data,
                    stride, out);
        }
        else
            return OSKAR_ERR_BAD_LOCATION;
    }
    else if (type == OSKAR_DOUBLE)
    {
        const double *knots_x, *knots_y, *coeff;
        double* out;
        nx      = spline->num_knots_x;
        ny      = spline->num_knots_y;
        knots_x = (const double*)spline->knots_x.data;
        knots_y = (const double*)spline->knots_y.data;
        coeff   = (const double*)spline->coeff.data;
        out     = (double*)output->data + offset;

        /* Check if data are in CPU memory. */
        if (location == OSKAR_LOCATION_CPU)
        {
            /* Set up workspace. */
            double wrk[8];
            int i, iwrk1[2], kwrk1 = 2, lwrk = 8;

            /* Evaluate surface at the points. */
            for (i = 0; i < num_points; ++i)
            {
                double x1, y1;
                x1 = ((const double*)x->data)[i];
                y1 = ((const double*)y->data)[i];
                oskar_dierckx_bispev_d(knots_x, nx, knots_y, ny, coeff,
                        3, 3, &x1, 1, &y1, 1, &out[i * stride],
                        wrk, lwrk, iwrk1, kwrk1, &err);
                if (err != 0) return OSKAR_ERR_SPLINE_EVAL_FAIL;
            }
        }
        else if (location == OSKAR_LOCATION_GPU)
        {
            /* Evaluate surface at the points by calling kernel. */
            int num_blocks, num_threads = 256;
            num_blocks = (num_points + num_threads - 1) / num_threads;
            oskar_cudak_dierckx_bispev_bicubic_d
            OSKAR_CUDAK_CONF(num_blocks, num_threads) (knots_x,
                    nx, knots_y, ny, coeff, num_points,
                    (const double*)x->data, (const double*)y->data,
                    stride, out);
        }
        else
            return OSKAR_ERR_BAD_LOCATION;
    }
    else
        return OSKAR_ERR_BAD_DATA_TYPE;

    if (location == OSKAR_LOCATION_GPU)
    {
        hipDeviceSynchronize();
        err = (int) hipPeekAtLastError();
    }
    return err;
}

#ifdef __cplusplus
}
#endif
