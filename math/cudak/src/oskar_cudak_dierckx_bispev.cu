#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "math/cudak/oskar_cudak_dierckx_bispev.h"
#include "math/cudak/oskar_cudaf_dierckx_fpbisp.h"
#include "math/cudak/oskar_cudaf_dierckx_fpbspl.h"

// Single precision.

__global__
void oskar_cudak_dierckx_bispev_f(const float* tx, int nx, const float* ty,
        int ny, const float* c, int kx, int ky, int n, const float* x,
        const float* y, float* z, float *wrk, int lwrk, int *iwrk, int kwrk)
{
    // Get the output position (pixel) ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;

    float theta1, phi1;
    theta1 = ((const float*)theta->data)[i];
    phi1 = ((const float*)phi->data)[i];
    bispev_f(knots_theta, nt, knots_phi, np, coeff, 3, 3,
            &theta1, 1, &phi1, 1, &out[i * 2 * stride],
            wrk, lwrk, iwrk1, kwrk1);
}
