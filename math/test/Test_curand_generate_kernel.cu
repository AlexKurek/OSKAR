#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_global.h>
#include <hiprand/hiprand_kernel.h>

// Test CURAND kernel.
__global__
void test_curand_generate_kernel(double* values, int num_values,
        int num_per_thread, hiprandStateXORWOW* state, int num_states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_states) return;

    for (int i = 0; i < num_per_thread; ++i)
    {
        int idx = num_per_thread * tid + i;
        if (idx >= num_values) continue;
        values[idx] = hiprand_normal_double(&state[tid]);
    }
}

// Kernel wrapper.
void test_curand_generate(double* d_values, int num_blocks, int num_threads,
        int num_values, int num_per_thread, hiprandStateXORWOW* state,
        int num_states)
{
    test_curand_generate_kernel OSKAR_CUDAK_CONF(num_blocks, num_threads)
        (d_values, num_values, num_per_thread, state, num_states);
}
