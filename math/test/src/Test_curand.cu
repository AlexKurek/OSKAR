#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */


#include "math/test/Test_curand.h"

#include "oskar_global.h"
#include "math/cudak/oskar_cudak_curand_init.h"
#include "math/test/cudak/test_curand_generate.h"
#include "math/oskar_allocate_curand_states.h"
#include "utility/oskar_get_error_string.h"
#include "utility/oskar_Mem.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <omp.h>

void Test_curand::test()
{
    int offset = 0;
    int seed   = 0;
    int num_threads = 10;
    int num_values_per_thread = 2;
    int device_offset = 0;
    FILE* file = NULL;
    //const char* filename = "temp_test_curand.txt";
    //file = fopen(filename, "w");

    int num_values = num_threads * num_values_per_thread;
    double* d_values;
    double* h_values;
    hipMalloc(&d_values, num_values * sizeof(double));
    h_values = (double*) malloc(num_values * sizeof(double));

    int num_blocks  = (num_values + num_threads - 1) / num_threads;

    hiprandState* d_states;
    hipMalloc((void**)&d_states, num_threads * sizeof(hiprandState));

    if (file)
    {
        fprintf(file, "--------\n");
        fprintf(file, "num_threads           = %i\n", num_threads);
        fprintf(file, "num_values_per_thread = %i\n", num_values_per_thread);
        fprintf(file, "num_values            = %i\n", num_values);
        fprintf(file, "num_blocks            = %i\n", num_blocks);
        fprintf(file, "offset                = %i\n", offset);
        fprintf(file, "seed                  = %i\n", seed);
        fprintf(file, "--------\n");
    }

    // Initialise the random number generator.
    oskar_cudak_curand_init
        OSKAR_CUDAK_CONF(num_blocks, num_threads)
        (d_states, seed, offset, device_offset);


    // Generate some random numbers.
    int num_sets = 3;
    for (int j = 0; j < num_sets; ++j)
    {
        test_curand_generate
            OSKAR_CUDAK_CONF(num_blocks, num_threads)
            (d_values, num_values, num_values_per_thread, d_states);
        hipMemcpy(h_values, d_values, num_values * sizeof(double), hipMemcpyDeviceToHost);
        if (file)
        {
            for (int i = 0; i < num_values; ++i)
            {
                fprintf(file, "%i %f\n", i, h_values[i]);
            }
            fprintf(file, "\n");
        }
    }

    if (file) fclose(file);
    hipFree(d_states);
    hipFree(d_values);
    free(h_values);
}


void Test_curand::test_state_allocation()
{
    {
        FILE* file = NULL;
//        const char* filename = "temp_test_curand_1.txt";
//        file = fopen(filename, "w");

        // Allocate a number of hiprand states.
        int offset = 0;
        int seed = 0;
        int num_states = (int)2e5;
        hiprandState* d_states;
        hipMalloc(&d_states, num_states * sizeof(hiprandState));
        int error = oskar_allocate_curand_states(d_states, num_states, seed, offset);
        CPPUNIT_ASSERT_MESSAGE(oskar_get_error_string(error), error == OSKAR_SUCCESS);

        int num_iter = 1;
        int num_blocks  = 2;
        int num_threads = 20;
        int num_per_thread = 1;
        int num_values = num_blocks * num_threads * num_per_thread;
        oskar_Mem d_values(OSKAR_DOUBLE, OSKAR_LOCATION_GPU, num_values);

        for (int i = 0; i < num_iter; ++i)
        {
            test_curand_generate
            OSKAR_CUDAK_CONF(num_blocks, num_threads)
            (d_values, num_values, num_per_thread, d_states);

            oskar_Mem h_values(&d_values, OSKAR_LOCATION_CPU);

            if (file)
            {
                for (int i = 0; i < num_values; ++i)
                {
                    fprintf(file, "%i %f\n", i, ((double*)h_values.data)[i]);
                }
            }
        }
        if (file) fclose(file);
        hipFree(d_states);
    }
}


void Test_curand::test_multi_device()
{
    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    omp_set_num_threads(min(num_devices, 4));
    int use_device[4] = {0, 1, 2, 3};

    #pragma omp parallel
    {
        int thread_id = omp_get_thread_num();
        int error = (int)hipSetDevice(use_device[thread_id]);
        CPPUNIT_ASSERT_MESSAGE(hipGetErrorString((hipError_t)error), error == 0);

        int device_id = 0;
        hipGetDevice(&device_id);

        char filename[100];
        sprintf(filename, "temp_test_curand_device_%i.txt", device_id);
        FILE* file = NULL;

        // Allocate a number of hiprand states.
        int offset = 0;
        int seed = 0;
        int num_states = (int)2e5;
        hiprandState* d_states;
        hipMalloc(&d_states, num_states * sizeof(hiprandState));
        error = oskar_allocate_curand_states(d_states, num_states, seed, offset);
        CPPUNIT_ASSERT_MESSAGE(oskar_get_error_string(error), error == OSKAR_SUCCESS);

        int num_iter = 1;
        int num_blocks  = 2;
        int num_threads = 20;
        int num_per_thread = 1;
        int num_values = num_blocks * num_threads * num_per_thread;
        oskar_Mem d_values(OSKAR_DOUBLE, OSKAR_LOCATION_GPU, num_values);

        file = fopen(filename, "w");

        for (int i = 0; i < num_iter; ++i)
        {
            test_curand_generate
            OSKAR_CUDAK_CONF(num_blocks, num_threads)
            (d_values, num_values, num_per_thread, d_states);

            oskar_Mem h_values(&d_values, OSKAR_LOCATION_CPU);

            if (file)
            {
                for (int i = 0; i < num_values; ++i)
                {
                    fprintf(file, "%i %f\n", i, ((double*)h_values.data)[i]);
                }
            }
        }
        fclose(file);
        hipFree(d_states);
    }
}

