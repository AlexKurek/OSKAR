#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "sky/cudak/oskar_sky_cudak_ha_dec_to_az_el.h"

// Single precision.

__global__
void oskar_sky_cudakf_ha_dec_to_az_el(int ns, const float2* hadec,
        float cosLat, float sinLat, float2* azel)
{
    // Get the source ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;

    // Copy source local equatorial coordinates from global memory.
    float2 src;
    if (s < ns)
        src = hadec[s];
    __syncthreads(); // Coalesce memory accesses.

    // Find azimuth and elevation.
    float cosDec, sinDec, cosHA, sinHA, t, X1, Y2;
    sincosf(src.x, &sinHA, &cosHA);
    sincosf(src.y, &sinDec, &cosDec);
    t = cosDec * cosHA;
    X1 = cosLat * sinDec - sinLat * t;
    Y2 = sinLat * sinDec + cosLat * t;
    t = -cosDec * sinHA;
    src.x = atan2f(t, X1); // Azimuth.
    t = hypotf(X1, t);
    src.y = atan2f(Y2, t); // Elevation.

    // Copy source horizontal coordinates into global memory.
    __syncthreads(); // Coalesce memory accesses.
    if (s < ns)
        azel[s] = src;
}

// Double precision.

__global__
void oskar_sky_cudakd_ha_dec_to_az_el(int ns, const double2* hadec,
        double cosLat, double sinLat, double2* azel)
{
    // Get the source ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;

    // Copy source local equatorial coordinates from global memory.
    double2 src;
    if (s < ns)
        src = hadec[s];
    __syncthreads(); // Coalesce memory accesses.

    // Find azimuth and elevation.
    double cosDec, sinDec, cosHA, sinHA, t, X1, Y2;
    sincos(src.x, &sinHA, &cosHA);
    sincos(src.y, &sinDec, &cosDec);
    t = cosDec * cosHA;
    X1 = cosLat * sinDec - sinLat * t;
    Y2 = sinLat * sinDec + cosLat * t;
    t = -cosDec * sinHA;
    src.x = atan2(t, X1); // Azimuth.
    t = hypot(X1, t);
    src.y = atan2(Y2, t); // Elevation.

    // Copy source horizontal coordinates into global memory.
    __syncthreads(); // Coalesce memory accesses.
    if (s < ns)
        azel[s] = src;
}
