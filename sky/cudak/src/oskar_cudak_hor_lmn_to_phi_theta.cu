#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "sky/cudak/oskar_cudak_hor_lmn_to_phi_theta.h"

#define PIf 3.14159265358979323846f
#define PI  3.14159265358979323846

// Single precision.
__global__
void oskar_cudak_hor_lmn_to_phi_theta_f(int num_points, const float* l,
        const float* m, const float* n, float* phi, float* theta)
{
    // Get the position ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_points) return;

    // Get the data.
    float x = l[i];
    float y = m[i];
    float z = n[i];

    // Cartesian to spherical.
    float p = atan2f(y, x) + PIf; // Phi in range 0 to 2 pi.
    x = sqrtf(x*x + y*y);
    y = atan2f(x, z); // Theta.
    phi[i] = p;
    theta[i] = y;
}

// Double precision.
__global__
void oskar_cudak_hor_lmn_to_phi_theta_d(int num_points, const double* l,
        const double* m, const double* n, double* phi, double* theta)
{
    // Get the position ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_points) return;

    // Get the data.
    double x = l[i];
    double y = m[i];
    double z = n[i];

    // Cartesian to spherical.
    double p = atan2(y, x) + PI; // Phi in range 0 to 2 pi.
    x = sqrt(x*x + y*y);
    y = atan2(x, z); // Theta.
    phi[i] = p;
    theta[i] = y;
}
