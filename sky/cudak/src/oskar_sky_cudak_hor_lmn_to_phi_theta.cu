#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "sky/cudak/oskar_sky_cudak_hor_lmn_to_phi_theta.h"

// Single precision.

__global__
void oskar_sky_cudakf_hor_lmn_to_phi_theta(int n, const float* p_l,
        const float* p_m, const float* p_n, float* phi, float* theta)
{
    // Get the position ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;

    // Get the data.
    float x = p_l[i];
    float y = p_m[i];
    float z = p_n[i];
    __syncthreads();

    // Cartesian to spherical.
    float p = atan2f(y, x); // Phi.
    x = sqrtf(x*x + y*y);
    y = atan2f(x, z); // Theta.
    phi[i] = p;
    theta[i] = y;
}

// Double precision.

__global__
void oskar_sky_cudakd_hor_lmn_to_phi_theta(int n, const double* p_l,
        const double* p_m, const double* p_n, double* phi, double* theta)
{
    // Get the position ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;

    // Get the data.
    double x = p_l[i];
    double y = p_m[i];
    double z = p_n[i];
    __syncthreads();

    // Cartesian to spherical.
    double p = atan2(y, x); // Phi.
    x = sqrt(x*x + y*y);
    y = atan2(x, z); // Theta.
    phi[i] = p;
    theta[i] = y;
}
