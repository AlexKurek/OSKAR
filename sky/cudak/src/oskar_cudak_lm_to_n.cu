#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "sky/cudak/oskar_cudak_lm_to_n.h"

// Single precision.

__global__
void oskar_cudak_lm_to_n_f(int n, const float* p_l, const float* p_m,
        float* p_n)
{
    // Get the position ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;

    float l = p_l[i];
    float m = p_m[i];
    float a = 1.0f - l*l - m*m;
    if (a < 0.0f)
    {
        p_n[i] = -1.0f;
    }
    else
    {
        float x = sqrtf(a) - 1.0f;
        p_n[i] = x;
    }
}

// Double precision.

__global__
void oskar_cudak_lm_to_n_d(int n, const double* p_l, const double* p_m,
        double* p_n)
{
    // Get the position ID that this thread is working on.
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;

    double l = p_l[i];
    double m = p_m[i];
    double a = 1.0 - l*l - m*m;
    if (a < 0.0)
    {
        p_n[i] = -1.0;
    }
    else
    {
        double x = sqrt(a) - 1.0;
        p_n[i] = x;
    }
}
