#include <mex.h>

#include "sky/oskar_cuda_ra_dec_to_az_el.h"

#include <math.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

void mexFunction(int /*num_outputs*/, mxArray ** output, int num_inputs,
        const mxArray ** input)
{
    // Parse Inputs.
    if (num_inputs != 4)
    {
        mexPrintf("(ra, dec, lst, lat)");
        mexErrMsgTxt("Four inputs required.");
    }

    // Get matlab inputs.
    double ra  = mxGetScalar(input[0]);
    double dec = mxGetScalar(input[1]);
    double lst = mxGetScalar(input[2]);
    double lat = mxGetScalar(input[3]);

//    mexPrintf("- ra  = %f\n", ra);
//    mexPrintf("- dec = %f\n", dec);
//    mexPrintf("- lst = %f\n", lst);
//    mexPrintf("- lat = %f\n", lat);

    int n = 1;
    mwSize n_dims  = 1;
    mwSize dims[1] = {n};
    output[0] = mxCreateNumericArray(n_dims, dims, mxDOUBLE_CLASS, mxREAL);
    output[1] = mxCreateNumericArray(n_dims, dims, mxDOUBLE_CLASS, mxREAL);
    double* az = (double*) mxGetPr(output[0]);
    double* el = (double*) mxGetPr(output[1]);
    size_t mem_size = n * sizeof(double);
    double* d_ra;
    hipMalloc((void**)&d_ra, mem_size);
    hipMemcpy(d_ra, &ra, mem_size, hipMemcpyHostToDevice);
    double* d_dec;
    hipMalloc((void**)&d_dec, mem_size);
    hipMemcpy(d_dec, &dec, mem_size, hipMemcpyHostToDevice);
    double* d_az;
    hipMalloc((void**)&d_az, mem_size);
    double* d_el;
    hipMalloc((void**)&d_el, mem_size);
    double* d_work;
    hipMalloc((void**)&d_work, mem_size);

    int error = (int)hipSuccess;
    error = oskar_cuda_ra_dec_to_az_el_d(n, d_ra, d_dec, lst, lat, d_work,
            d_az, d_el);
    hipMemcpy(az, d_az, mem_size, hipMemcpyDeviceToHost);
    hipMemcpy(el, d_el, mem_size, hipMemcpyDeviceToHost);

    hipFree(d_ra);
    hipFree(d_dec);
    hipFree(d_az);
    hipFree(d_el);
    hipFree(d_work);

    if (error != hipSuccess)
    {
        mexPrintf("****************************************************\n");
        mexPrintf("** CUDA ERROR[%i]: %s.\n", error,
                hipGetErrorString((hipError_t)error));
        mexPrintf("****************************************************\n");
        mexErrMsgTxt("** ERROR: oskar_ra_dec_to_az_el_d()");
    }
}
