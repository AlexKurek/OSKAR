#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_evaluate_average_cross_power_beam_cuda.h>
#include <oskar_correlate_functions_inline.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_evaluate_average_cross_power_beam_cuda_f(int num_sources,
        int num_stations, const float4c* d_jones, float4c* d_beam)
{
    int num_blocks, num_threads = 128;
    size_t shared_mem = num_threads * sizeof(float4c);
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_average_cross_power_beam_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem) (num_sources,
            num_stations, d_jones, d_beam);
}

/* Double precision. */
void oskar_evaluate_average_cross_power_beam_cuda_d(int num_sources,
        int num_stations, const double4c* d_jones, double4c* d_beam)
{
    int num_blocks, num_threads = 128;
    size_t shared_mem = num_threads * sizeof(double4c);
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_evaluate_average_cross_power_beam_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem) (num_sources,
            num_stations, d_jones, d_beam);
}

/* Kernels. ================================================================ */

extern __shared__ float4c  smem_f[];
extern __shared__ double4c smem_d[];

/* Single precision. */
__global__
void oskar_evaluate_average_cross_power_beam_cudak_f(const int num_sources,
        const int num_stations, const float4c* restrict jones,
        float4c* restrict beam)
{
    float4c val, *p, q;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate cross-power beam at the source. */
    p = &smem_f[threadIdx.x];
    oskar_clear_complex_matrix_f(&val);
    for (int SP = 0; SP < num_stations; ++SP)
    {
        /* Load data for first station into shared memory. */
        OSKAR_LOAD_MATRIX(smem_f[threadIdx.x], jones, SP * num_sources + i);

        /* Cross-correlate. */
        for (int SQ = SP + 1; SQ < num_stations; ++SQ)
        {
            /* Load data for second station into registers. */
            OSKAR_LOAD_MATRIX(q, jones, SQ * num_sources + i);

            /* Multiply-add: val += p * conj(q). */
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.a, p->a, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.a, p->b, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.b, p->a, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.b, p->b, q.d);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.c, p->c, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.c, p->d, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.d, p->c, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.d, p->d, q.d);
        }
    }

    /* Calculate average. */
    int num_baselines = (num_stations * (num_stations - 1)) / 2;
    val.a.x /= num_baselines;
    val.a.y /= num_baselines;
    val.b.x /= num_baselines;
    val.b.y /= num_baselines;
    val.c.x /= num_baselines;
    val.c.y /= num_baselines;
    val.d.x /= num_baselines;
    val.d.y /= num_baselines;

    /* Store result. */
    beam[i] = val;
}

/* Double precision. */
__global__
void oskar_evaluate_average_cross_power_beam_cudak_d(const int num_sources,
        const int num_stations, const double4c* restrict jones,
        double4c* restrict beam)
{
    double4c val, *p, q;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_sources) return;

    /* Calculate cross-power beam at the source. */
    p = &smem_d[threadIdx.x];
    oskar_clear_complex_matrix_d(&val);
    for (int SP = 0; SP < num_stations; ++SP)
    {
        /* Load data for first station into shared memory. */
        OSKAR_LOAD_MATRIX(smem_d[threadIdx.x], jones, SP * num_sources + i);

        /* Cross-correlate. */
        for (int SQ = SP + 1; SQ < num_stations; ++SQ)
        {
            /* Load data for second station into registers. */
            OSKAR_LOAD_MATRIX(q, jones, SQ * num_sources + i);

            /* Multiply-add: val += p * conj(q). */
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.a, p->a, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.a, p->b, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.b, p->a, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.b, p->b, q.d);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.c, p->c, q.a);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.c, p->d, q.b);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.d, p->c, q.c);
            OSKAR_MULTIPLY_ADD_COMPLEX_CONJUGATE(val.d, p->d, q.d);
        }
    }

    /* Calculate average. */
    int num_baselines = (num_stations * (num_stations - 1)) / 2;
    val.a.x /= num_baselines;
    val.a.y /= num_baselines;
    val.b.x /= num_baselines;
    val.b.y /= num_baselines;
    val.c.x /= num_baselines;
    val.c.y /= num_baselines;
    val.d.x /= num_baselines;
    val.d.y /= num_baselines;

    /* Store result. */
    beam[i] = val;
}

#ifdef __cplusplus
}
#endif
