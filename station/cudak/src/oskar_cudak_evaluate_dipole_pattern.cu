#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "station/cudak/oskar_cudak_evaluate_dipole_pattern.h"
#include <math.h>

#if 0
// Single precision.
__global__
void oskar_cudak_evaluate_dipole_pattern_f(const int num_sources,
        const float* theta, const float* phi, const float orientation_x,
        const float orientation_y, const int taper_func, float4c* pattern)
{
    // Source index being processed by the thread.
    const int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= num_sources) return;

    // Get source direction cosines.
    const float ln = cosf(theta[s]); // Component along z-axis.

    // Get sine and cosine of angle phi.
    float sin_phi, cos_phi;
    sincosf(phi[s], &sin_phi, &cos_phi);

    // Evaluate unit vectors e_theta and e_phi at source position.
    // cos_theta = ln
    const float e_theta_x = ln * cos_phi; // Component of e_theta in x.
    const float e_theta_y = ln * sin_phi; // Component of e_theta in y.
    // e_phi_x = -sin_phi;
    // e_phi_y = cos_phi;

    // Dot products:
    // g_theta_a = a_x * e_theta_x + a_y * e_theta_y;
    // g_phi_a   = a_x * e_phi_x   + a_y * e_phi_y;
    // g_theta_b = b_x * e_theta_x + b_y * e_theta_y;
    // g_phi_b   = b_x * e_phi_x   + b_y * e_phi_y;
    const float g_theta_a = sin_orientation_x * e_theta_x
            + cos_orientation_x * e_theta_y;
    const float g_phi_a   = sin_orientation_x * -sin_phi
            + cos_orientation_x * cos_phi;
    const float g_theta_b = sin_orientation_y * e_theta_x
            + cos_orientation_y * e_theta_y;
    const float g_phi_b   = sin_orientation_y * -sin_phi
            + cos_orientation_y * cos_phi;

    // Store components.
    pattern[s].a.x += g_theta_a;
    pattern[s].b.x += g_phi_a;
    pattern[s].c.x += g_theta_b;
    pattern[s].d.x += g_phi_b;
}
#endif

// Single precision.
__global__
void oskar_cudak_evaluate_dipole_pattern_f(const int num_sources,
        const float* l, const float* m, const float* n,
        const float cos_orientation_x, const float sin_orientation_x,
        const float cos_orientation_y, const float sin_orientation_y,
        float4c* pattern)
{
    // Source index being processed by the thread.
    const int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= num_sources) return;

    // Get source direction cosines.
    const float ll = l[s]; // Component along x-axis.
    const float lm = m[s]; // Component along y-axis.
    const float ln = n[s]; // Component along z-axis.

    // Evaluate phi, the angle (co-azimuth) from East (x) towards North (y).
    const float phi = atan2f(lm, ll);
    float sin_phi, cos_phi;
    sincosf(phi, &sin_phi, &cos_phi); // Cannot use direction cosines here.

    // Evaluate unit vectors e_theta and e_phi at source position.
    // cos_theta = ln
    const float e_theta_x = ln * cos_phi; // Component of e_theta in x.
    const float e_theta_y = ln * sin_phi; // Component of e_theta in y.
    // e_phi_x = -sin_phi;
    // e_phi_y = cos_phi;

    // Dot products:
    // g_phi_a   = a_x * e_phi_x   + a_y * e_phi_y;
    // g_theta_a = a_x * e_theta_x + a_y * e_theta_y;
    // g_phi_b   = b_x * e_phi_x   + b_y * e_phi_y;
    // g_theta_b = b_x * e_theta_x + b_y * e_theta_y;
    const float g_phi_a   = sin_orientation_x * -sin_phi
            + cos_orientation_x * cos_phi;
    const float g_theta_a = sin_orientation_x * e_theta_x
            + cos_orientation_x * e_theta_y;
    const float g_phi_b   = sin_orientation_y * -sin_phi
            + cos_orientation_y * cos_phi;
    const float g_theta_b = sin_orientation_y * e_theta_x
            + cos_orientation_y * e_theta_y;

    // Store components.
    pattern[s].a.x = g_theta_a;
    pattern[s].a.y = 0.0f;
    pattern[s].b.x = g_phi_a;
    pattern[s].b.y = 0.0f;
    pattern[s].c.x = g_theta_b;
    pattern[s].c.y = 0.0f;
    pattern[s].d.x = g_phi_b;
    pattern[s].d.y = 0.0f;
}

// Double precision.
__global__
void oskar_cudak_evaluate_dipole_pattern_d(const int num_sources,
        const double* l, const double* m, const double* n,
        const double cos_orientation_x, const double sin_orientation_x,
        const double cos_orientation_y, const double sin_orientation_y,
        double4c* pattern)
{
    // Source index being processed by the thread.
    const int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= num_sources) return;

    // Get source direction cosines.
    const double ll = l[s]; // Component along x-axis.
    const double lm = m[s]; // Component along y-axis.
    const double ln = n[s]; // Component along z-axis.

    // Evaluate phi, the angle (co-azimuth) from East (x) towards North (y).
    const double phi = atan2(lm, ll);
    double sin_phi, cos_phi;
    sincos(phi, &sin_phi, &cos_phi); // Cannot use direction cosines here.

    // Evaluate unit vectors e_theta and e_phi at source position.
    // cos_theta = ln
    const double e_theta_x = ln * cos_phi; // Component of e_theta in x.
    const double e_theta_y = ln * sin_phi; // Component of e_theta in y.
    // e_phi_x = -sin_phi;
    // e_phi_y = cos_phi;

    // Dot products:
    // g_phi_a   = a_x * e_phi_x   + a_y * e_phi_y;
    // g_theta_a = a_x * e_theta_x + a_y * e_theta_y;
    // g_phi_b   = b_x * e_phi_x   + b_y * e_phi_y;
    // g_theta_b = b_x * e_theta_x + b_y * e_theta_y;
    const double g_phi_a   = sin_orientation_x * -sin_phi
            + cos_orientation_x * cos_phi;
    const double g_theta_a = sin_orientation_x * e_theta_x
            + cos_orientation_x * e_theta_y;
    const double g_phi_b   = sin_orientation_y * -sin_phi
            + cos_orientation_y * cos_phi;
    const double g_theta_b = sin_orientation_y * e_theta_x
            + cos_orientation_y * e_theta_y;

    // Store components.
    pattern[s].a.x = g_theta_a;
    pattern[s].a.y = 0.0;
    pattern[s].b.x = g_phi_a;
    pattern[s].b.y = 0.0;
    pattern[s].c.x = g_theta_b;
    pattern[s].c.y = 0.0;
    pattern[s].d.x = g_phi_b;
    pattern[s].d.y = 0.0;
}
