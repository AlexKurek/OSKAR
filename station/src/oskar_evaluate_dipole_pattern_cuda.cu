#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2013, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_mem.h>
#include "station/oskar_evaluate_dipole_pattern_cuda.h"

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_evaluate_dipole_pattern_cuda_f(int num_points,
        const float* d_theta, const float* d_phi, int return_x_dipole,
        float4c* d_pattern)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_points + num_threads - 1) / num_threads;
    oskar_evaluate_dipole_pattern_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_points, d_theta, d_phi,
            return_x_dipole, d_pattern);
}

/* Double precision. */
void oskar_evaluate_dipole_pattern_cuda_d(int num_points,
        const double* d_theta, const double* d_phi, int return_x_dipole,
        double4c* d_pattern)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_points + num_threads - 1) / num_threads;
    oskar_evaluate_dipole_pattern_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (num_points, d_theta, d_phi,
            return_x_dipole, d_pattern);
}


/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_evaluate_dipole_pattern_cudak_f(const int num_points,
        const float* theta, const float* phi, const int return_x_dipole,
        float4c* pattern)
{
    float theta_c, sin_phi, cos_phi, cos_theta, e_theta, e_phi;

    /* Source index being processed by the thread. */
    const int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= num_points) return;

    /* Get (modified) source vector components, relative to a dipole along x. */
    theta_c = theta[s];
    cos_theta = cosf(theta_c);
    sincosf(phi[s], &sin_phi, &cos_phi);

    /* Evaluate vectors e_theta and e_phi in x-direction at source position. */
    e_theta = cos_theta * cos_phi;
    e_phi = -sin_phi;

    /* Store components. */
    if (return_x_dipole)
    {
        pattern[s].a.x = e_theta;
        pattern[s].a.y = 0.0f;
        pattern[s].b.x = e_phi;
        pattern[s].b.y = 0.0f;
    }
    else
    {
        pattern[s].c.x = e_theta;
        pattern[s].c.y = 0.0f;
        pattern[s].d.x = e_phi;
        pattern[s].d.y = 0.0f;
    }
}

/* Double precision. */
__global__
void oskar_evaluate_dipole_pattern_cudak_d(const int num_points,
        const double* theta, const double* phi, const int return_x_dipole,
        double4c* pattern)
{
    double theta_c, sin_phi, cos_phi, cos_theta, e_theta, e_phi;

    /* Source index being processed by the thread. */
    const int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= num_points) return;

    /* Get (modified) source vector components, relative to a dipole along x. */
    theta_c = theta[s];
    cos_theta = cos(theta_c);
    sincos(phi[s], &sin_phi, &cos_phi);

    /* Evaluate vectors e_theta and e_phi in x-direction at source position. */
    e_theta = cos_theta * cos_phi;
    e_phi = -sin_phi;

    /* Store components. */
    if (return_x_dipole)
    {
        pattern[s].a.x = e_theta;
        pattern[s].a.y = 0.0;
        pattern[s].b.x = e_phi;
        pattern[s].b.y = 0.0;
    }
    else
    {
        pattern[s].c.x = e_theta;
        pattern[s].c.y = 0.0;
        pattern[s].d.x = e_phi;
        pattern[s].d.y = 0.0;
    }
}

#ifdef __cplusplus
}
#endif
