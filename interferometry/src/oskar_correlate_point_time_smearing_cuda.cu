#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2013, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_accumulate_baseline_visibility_for_source.h>
#include <oskar_correlate_point_time_smearing_cuda.h>
#include <oskar_sinc.h>

#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_correlate_point_time_smearing_cuda_f(int num_sources,
        int num_stations, const float4c* d_jones,
        const float* d_source_I, const float* d_source_Q,
        const float* d_source_U, const float* d_source_V,
        const float* d_source_l, const float* d_source_m,
        const float* d_source_n, const float* d_station_u,
        const float* d_station_v, const float* d_station_x,
        const float* d_station_y, float inv_wavelength, float frac_bandwidth,
        float time_int_sec, float gha0_rad, float dec0_rad, float4c* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(float4c);
    oskar_correlate_point_time_smearing_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_Q, d_source_U,
            d_source_V, d_source_l, d_source_m, d_source_n, d_station_u,
            d_station_v, d_station_x, d_station_y, inv_wavelength,
            frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
}

/* Double precision. */
void oskar_correlate_point_time_smearing_cuda_d(int num_sources,
        int num_stations, const double4c* d_jones,
        const double* d_source_I, const double* d_source_Q,
        const double* d_source_U, const double* d_source_V,
        const double* d_source_l, const double* d_source_m,
        const double* d_source_n, const double* d_station_u,
        const double* d_station_v, const double* d_station_x,
        const double* d_station_y, double inv_wavelength, double frac_bandwidth,
        double time_int_sec, double gha0_rad, double dec0_rad, double4c* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(double4c);
    oskar_correlate_point_time_smearing_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_Q, d_source_U,
            d_source_V, d_source_l, d_source_m, d_source_n, d_station_u,
            d_station_v, d_station_x, d_station_y, inv_wavelength,
            frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
}

#ifdef __cplusplus
}
#endif


/* Kernels. ================================================================ */

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifndef M_PIf
#define M_PIf 3.14159265358979323846f
#endif

#define OMEGA_EARTH  7.272205217e-5  /* radians/sec */
#define OMEGA_EARTHf 7.272205217e-5f /* radians/sec */

/* Indices into the visibility/baseline matrix. */
#define SI blockIdx.x /* Column index. */
#define SJ blockIdx.y /* Row index. */

extern __shared__ float4c  smem_f[];
extern __shared__ double4c smem_d[];

/* Single precision. */
__global__
void oskar_correlate_point_time_smearing_cudak_f(const int num_sources,
        const int num_stations, const float4c* __restrict__ jones,
        const float* __restrict__ source_I,
        const float* __restrict__ source_Q,
        const float* __restrict__ source_U,
        const float* __restrict__ source_V,
        const float* __restrict__ source_l,
        const float* __restrict__ source_m,
        const float* __restrict__ source_n,
        const float* __restrict__ station_u,
        const float* __restrict__ station_v,
        const float* __restrict__ station_x,
        const float* __restrict__ station_y, const float inv_wavelength,
        const float frac_bandwidth, const float time_int_sec,
        const float gha0_rad, const float dec0_rad,
        float4c* __restrict__ vis)
{
    /* Local variables. */
    float4c sum;
    float l, m, n, rb, rt;
    int i;

    /* Common values per thread block. */
    __shared__ float uu, vv, du_dt, dv_dt, dw_dt;
    __shared__ const float4c* __restrict__ station_i;
    __shared__ const float4c* __restrict__ station_j;

    /* Return immediately if in the wrong half of the visibility matrix. */
    if (SJ >= SI) return;

    /* Use thread 0 to set up the block. */
    if (threadIdx.x == 0)
    {
        float factor;

        /* Baseline lengths. */
        factor = M_PIf * inv_wavelength;
        uu = (station_u[SI] - station_u[SJ]) * factor;
        vv = (station_v[SI] - station_v[SJ]) * factor;

        /* Modify the baseline distance to include the common components
         * of the bandwidth smearing term. */
        uu *= frac_bandwidth;
        vv *= frac_bandwidth;

        /* Compute the derivatives for time-average smearing. */
        {
            float xx, yy, rot_angle, temp;
            float sin_HA, cos_HA, sin_Dec, cos_Dec;
            sincosf(gha0_rad, &sin_HA, &cos_HA);
            sincosf(dec0_rad, &sin_Dec, &cos_Dec);
            xx = (station_x[SI] - station_x[SJ]) * factor;
            yy = (station_y[SI] - station_y[SJ]) * factor;
            rot_angle = OMEGA_EARTHf * time_int_sec;
            temp = (xx * sin_HA + yy * cos_HA) * rot_angle;
            du_dt = (xx * cos_HA - yy * sin_HA) * rot_angle;
            dv_dt = temp * sin_Dec;
            dw_dt = -temp * cos_Dec;
        }

        /* Get pointers to source vectors for both stations. */
        station_i = &jones[num_sources * SI];
        station_j = &jones[num_sources * SJ];
    }
    __syncthreads();

    /* Partial sum per thread. */
    sum.a = make_float2(0.0f, 0.0f);
    sum.b = make_float2(0.0f, 0.0f);
    sum.c = make_float2(0.0f, 0.0f);
    sum.d = make_float2(0.0f, 0.0f);

    /* Each thread loops over a subset of the sources. */
    for (i = threadIdx.x; i < num_sources; i += blockDim.x)
    {
        /* Get source direction cosines. */
        l = source_l[i];
        m = source_m[i];
        n = source_n[i];

        /* Compute bandwidth- and time-smearing terms. */
        rb = oskar_sinc_f(uu * l + vv * m);
        rt = oskar_sinc_f(du_dt * l + dv_dt * m + dw_dt * n);
        rb *= rt;

        /* Accumulate baseline visibility response for source. */
        oskar_accumulate_baseline_visibility_for_source_f(&sum, i,
                source_I, source_Q, source_U, source_V,
                station_i, station_j, rb);
    }

    /* Store partial sum for the thread in shared memory and synchronise. */
    smem_f[threadIdx.x] = sum;
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_f[i].a.x;
            sum.a.y += smem_f[i].a.y;
            sum.b.x += smem_f[i].b.x;
            sum.b.y += smem_f[i].b.y;
            sum.c.x += smem_f[i].c.x;
            sum.c.y += smem_f[i].c.y;
            sum.d.x += smem_f[i].d.x;
            sum.d.y += smem_f[i].d.y;
        }

        /* Determine 1D visibility index for global memory store. */
        i = SJ*(num_stations-1) - (SJ-1)*SJ/2 + SI - SJ - 1;

        /* Add result of this thread block to the baseline visibility. */
        vis[i].a.x += sum.a.x;
        vis[i].a.y += sum.a.y;
        vis[i].b.x += sum.b.x;
        vis[i].b.y += sum.b.y;
        vis[i].c.x += sum.c.x;
        vis[i].c.y += sum.c.y;
        vis[i].d.x += sum.d.x;
        vis[i].d.y += sum.d.y;
    }
}

/* Double precision. */
__global__
void oskar_correlate_point_time_smearing_cudak_d(const int num_sources,
        const int num_stations, const double4c* __restrict__ jones,
        const double* __restrict__ source_I,
        const double* __restrict__ source_Q,
        const double* __restrict__ source_U,
        const double* __restrict__ source_V,
        const double* __restrict__ source_l,
        const double* __restrict__ source_m,
        const double* __restrict__ source_n,
        const double* __restrict__ station_u,
        const double* __restrict__ station_v,
        const double* __restrict__ station_x,
        const double* __restrict__ station_y, const double inv_wavelength,
        const double frac_bandwidth, const double time_int_sec,
        const double gha0_rad, const double dec0_rad,
        double4c* __restrict__ vis)
{
    /* Local variables. */
    double4c sum;
    double l, m, n, r1, r2;
    int i;

    /* Common values per thread block. */
    __shared__ double uu, vv, du_dt, dv_dt, dw_dt;
    __shared__ const double4c* __restrict__ station_i;
    __shared__ const double4c* __restrict__ station_j;

    /* Return immediately if in the wrong half of the visibility matrix. */
    if (SJ >= SI) return;

    /* Use thread 0 to set up the block. */
    if (threadIdx.x == 0)
    {
        double factor;

        /* Baseline lengths. */
        factor = M_PI * inv_wavelength;
        uu = (station_u[SI] - station_u[SJ]) * factor;
        vv = (station_v[SI] - station_v[SJ]) * factor;

        /* Modify the baseline distance to include the common components
         * of the bandwidth smearing term. */
        uu *= frac_bandwidth;
        vv *= frac_bandwidth;

        /* Compute the derivatives for time-average smearing. */
        {
            double xx, yy, rot_angle, temp;
            double sin_HA, cos_HA, sin_Dec, cos_Dec;
            sincos(gha0_rad, &sin_HA, &cos_HA);
            sincos(dec0_rad, &sin_Dec, &cos_Dec);
            xx = (station_x[SI] - station_x[SJ]) * factor;
            yy = (station_y[SI] - station_y[SJ]) * factor;
            rot_angle = OMEGA_EARTH * time_int_sec;
            temp = (xx * sin_HA + yy * cos_HA) * rot_angle;
            du_dt = (xx * cos_HA - yy * sin_HA) * rot_angle;
            dv_dt = temp * sin_Dec;
            dw_dt = -temp * cos_Dec;
        }

        /* Get pointers to source vectors for both stations. */
        station_i = &jones[num_sources * SI];
        station_j = &jones[num_sources * SJ];
    }
    __syncthreads();

    /* Partial sum per thread. */
    sum.a = make_double2(0.0, 0.0);
    sum.b = make_double2(0.0, 0.0);
    sum.c = make_double2(0.0, 0.0);
    sum.d = make_double2(0.0, 0.0);

    /* Each thread loops over a subset of the sources. */
    for (i = threadIdx.x; i < num_sources; i += blockDim.x)
    {
        /* Get source direction cosines. */
        l = source_l[i];
        m = source_m[i];
        n = source_n[i];

        /* Compute bandwidth- and time-smearing terms. */
        r1 = oskar_sinc_d(uu * l + vv * m);
        r2 = oskar_sinc_d(du_dt * l + dv_dt * m + dw_dt * n);
        r1 *= r2;

        /* Accumulate baseline visibility response for source. */
        oskar_accumulate_baseline_visibility_for_source_d(&sum, i,
                source_I, source_Q, source_U, source_V,
                station_i, station_j, r1);
    }

    /* Store partial sum for the thread in shared memory and synchronise. */
    smem_d[threadIdx.x] = sum;
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_d[i].a.x;
            sum.a.y += smem_d[i].a.y;
            sum.b.x += smem_d[i].b.x;
            sum.b.y += smem_d[i].b.y;
            sum.c.x += smem_d[i].c.x;
            sum.c.y += smem_d[i].c.y;
            sum.d.x += smem_d[i].d.x;
            sum.d.y += smem_d[i].d.y;
        }

        /* Determine 1D visibility index for global memory store. */
        i = SJ*(num_stations-1) - (SJ-1)*SJ/2 + SI - SJ - 1;

        /* Add result of this thread block to the baseline visibility. */
        vis[i].a.x += sum.a.x;
        vis[i].a.y += sum.a.y;
        vis[i].b.x += sum.b.x;
        vis[i].b.y += sum.b.y;
        vis[i].c.x += sum.c.x;
        vis[i].c.y += sum.c.y;
        vis[i].d.x += sum.d.x;
        vis[i].d.y += sum.d.y;
    }
}
