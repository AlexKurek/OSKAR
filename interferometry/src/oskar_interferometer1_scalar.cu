#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/oskar_interferometer1_scalar.h"
#include "interferometry/oskar_cuda_correlator_scalar.h"
#include "interferometry/oskar_compute_baselines.h"
#include "interferometry/oskar_xyz_to_uvw.h"

#include "sky/oskar_cuda_horizon_clip.h"
#include "sky/oskar_cuda_ra_dec_to_hor_lmn.h"
#include "sky/oskar_cuda_ra_dec_to_relative_lmn.h"
#include "sky/oskar_ra_dec_to_hor_lmn.h"

#include "station/oskar_mult_beampattern_by_sources.h"
#include "station/oskar_evaluate_station_beam.h"

#include "sky/oskar_mjd_to_last_fast.h"

#include <cstdio>
#include <cmath>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifdef __cplusplus
extern "C" {
#endif

// =============================================================================
void alloc_beamforming_weights_buffer_d(const unsigned num_stations,
        const oskar_StationModel_d* stations, double2** d_weights);

void alloc_beamforming_weights_buffer_f(const unsigned num_stations,
        const oskar_StationModel_f* stations, float2** d_weights);
// =============================================================================


int oskar_interferometer1_scalar_d(
        const oskar_TelescopeModel_d telescope,
        const oskar_StationModel_d * stations,
        const oskar_SkyModelGlobal_d sky,
        const double ra0_rad,
        const double dec0_rad,
        const double obs_start_mjd_utc,
        const double obs_length_days,
        const unsigned num_vis_dumps,
        const unsigned num_vis_ave,
        const unsigned num_fringe_ave,
        const double frequency,
        const double bandwidth,
        const bool disable_e_jones,
        oskar_VisData_d* h_vis
){
    const double sec_per_day = 86400.0;

    // === Evaluate number of stations and number of baselines.
    const unsigned num_stations = telescope.num_antennas;
    const unsigned num_baselines = num_stations * (num_stations - 1) / 2;

    const double lambda = 299792458.0 / frequency;
    const double wavenumber = 2.0 * M_PI / lambda;

    // === Allocate device memory for telescope and transfer to device.
    oskar_TelescopeModel_d hd_telescope;
    oskar_copy_telescope_to_device_d(&telescope, &hd_telescope);

    // === Scale telescope coordinates to wavenumber units.
    oskar_scale_device_telescope_coords_d(&hd_telescope, wavenumber);

    // === Allocate memory to hold station uvw coordinates.
    double* station_u = (double*)malloc(num_stations * sizeof(double));
    double* station_v = (double*)malloc(num_stations * sizeof(double));
    double* station_w = (double*)malloc(num_stations * sizeof(double));

    // === Allocate device memory for antennas and transfer to the device.
    size_t mem_size = num_stations * sizeof(oskar_StationModel_d);
    oskar_StationModel_d* hd_stations = (oskar_StationModel_d*)malloc(mem_size);
    oskar_copy_stations_to_device_d(stations, num_stations, hd_stations);

    // === Scale station coordinates to wavenumber units.
    oskar_scale_station_coords_d(num_stations, hd_stations, wavenumber);

    // === Allocate device memory for source model and transfer to device.
    oskar_SkyModelGlobal_d hd_sky_global;
    for (int i = 0; i < sky.num_sources; ++i)
        sky.I[i] = sqrt(sky.I[i]);
    oskar_copy_gobal_sky_to_device_d(&sky, &hd_sky_global);

    // === Allocate local sky structure.
    oskar_SkyModelLocal_d hd_sky_local;
    oskar_allocate_device_local_sky_d(sky.num_sources, &hd_sky_local);

    // === Allocate device memory for station beam patterns.
    double2 * d_e_jones = NULL;
    size_t mem_e_jones = telescope.num_antennas * sky.num_sources * sizeof(double2);
    hipMalloc((void**)&d_e_jones, mem_e_jones);

    // === Allocate device memory for beamforming weights buffer.
    double2* d_weights_work;
    alloc_beamforming_weights_buffer_d(telescope.num_antennas, stations,
            &d_weights_work);

    // === Allocate device memory for source positions in relative lmn coordinates.
    double *d_l, *d_m, *d_n;
    hipMalloc((void**)&d_l, sky.num_sources * sizeof(double));
    hipMalloc((void**)&d_m, sky.num_sources * sizeof(double));
    hipMalloc((void**)&d_n, sky.num_sources * sizeof(double));

    // === Allocate device memory for visibilities.
    double2* d_vis;
    size_t mem_size_vis = num_baselines * sizeof(double2);
    hipMalloc((void**)&d_vis, mem_size_vis);

    // === Allocate device memory for correlator work buffer.
    double* d_work_uvw;
    hipMalloc((void**)&d_work_uvw, 3 * num_stations * sizeof(double));
    double2* d_work_k;
    hipMalloc((void**)&d_work_k, num_stations * hd_sky_global.num_sources *  sizeof(double2));

    // === Calculate time increments.
    unsigned total_samples   = num_vis_dumps * num_fringe_ave * num_vis_ave;
    double dt_days           = obs_length_days / total_samples;
    double dt_vis_days       = obs_length_days / num_vis_dumps;
    double dt_vis_ave_days   = dt_vis_days / num_vis_ave;
    double dt_vis_offset     = dt_vis_days / 2.0;
    double dt_vis_ave_offset = dt_vis_ave_days / 2.0;

    // === Loop over number of visibility snapshots.
    for (unsigned j = 0; j < num_vis_dumps; ++j)
    {
        printf("--> simulating visibility snapshot %i of %i ...\n", j+1,
                num_vis_dumps);

        // Start time for the visibility dump (in mjd utc)
        double t_vis_dump_start = obs_start_mjd_utc + (j * dt_vis_days);

        // Initialise visibilities for the dump to zero.
        hipMemset((void*)d_vis, 0, mem_size_vis);

        // Loop over evaluations of the visibility average with changing
        // E-Jones within the dump.
        for (unsigned i = 0; i < num_vis_ave; ++i)
        {
            // Evaluate lst
            double t_ave_start = t_vis_dump_start + i * dt_vis_ave_days;
            double t_ave_mid   = t_ave_start + dt_vis_ave_offset;
            double lst = oskar_mjd_to_last_fast_d(t_ave_mid, telescope.longitude);

            // Find sources above horizon.
            oskar_cuda_horizon_clip_d(&hd_sky_global, lst, telescope.latitude,
                    &hd_sky_local);

            if (hd_sky_local.num_sources == 0)
                fprintf(stderr, "WARNING: no sources above horizon! (this will fail!)\n");

            // Evaluate horizontal lm for the beam phase centre.
            double h_beam_l, h_beam_m, h_beam_n;
            oskar_ra_dec_to_hor_lmn_d(1, &ra0_rad, &dec0_rad, lst,
                    telescope.latitude, &h_beam_l, &h_beam_m, &h_beam_n);

            // Evaluate E-Jones for each source position per station
            oskar_evaluate_station_beams_d(num_stations, hd_stations,
                    &hd_sky_local, h_beam_l, h_beam_m, d_weights_work,
                    disable_e_jones, telescope.identical_stations, d_e_jones);

            // Multiply e-jones by source brightness.
            oskar_mult_beampattern_by_source_field_amp_d(num_stations,
                    &hd_sky_local, d_e_jones);

            // Convert source positions in ra, dec to lmn relative to the phase
            // centre.
            oskar_cuda_ra_dec_to_relative_lmn_d(hd_sky_local.num_sources,
                    hd_sky_local.RA, hd_sky_local.Dec, ra0_rad, dec0_rad,
                    d_l, d_m, d_n);

            // Correlator which updates phase matrix.
            double lst_start = oskar_mjd_to_last_fast_d(t_ave_start,
                    telescope.longitude);
            oskar_cuda_correlator_scalar_d(num_stations, hd_telescope.antenna_x,
                    hd_telescope.antenna_y, hd_telescope.antenna_z,
                    hd_sky_local.num_sources, d_l, d_m, d_n,
                    d_e_jones, ra0_rad, dec0_rad, lst_start,
                    num_fringe_ave, dt_days * sec_per_day, lambda * bandwidth,
                    d_work_k, d_work_uvw, d_vis);
        }

        // copy back the vis dump into host memory.
        hipMemcpy(&(h_vis->amp[num_baselines * j]), d_vis, mem_size_vis,
                hipMemcpyDeviceToHost);

        // Evaluate baseline coordinates for the visibility dump.
        double* u = &h_vis->u[num_baselines * j];
        double* v = &h_vis->v[num_baselines * j];
        double* w = &h_vis->w[num_baselines * j];
        double t_vis = t_vis_dump_start + dt_vis_offset;
        double lst_vis_dump = oskar_mjd_to_last_fast_d(t_vis, telescope.longitude);
        double ha_vis = lst_vis_dump - ra0_rad;
        oskar_xyz_to_uvw_d(num_stations, telescope.antenna_x, telescope.antenna_y,
                telescope.antenna_z, ha_vis, dec0_rad, station_u, station_v,
                station_w);
        oskar_compute_baselines_d(num_stations, station_u, station_v, station_w,
                u, v, w);
    }

    // free memory
    oskar_free_device_telescope_d(&hd_telescope);
    oskar_free_device_global_sky_d(&hd_sky_global);
    oskar_free_device_local_sky_d(&hd_sky_local);
    free(station_u);
    free(station_v);
    free(station_w);
    for (unsigned i = 0; i < num_stations; ++i)
    {
        hipFree(hd_stations[i].antenna_x);
        hipFree(hd_stations[i].antenna_y);
    }
    free(hd_stations);
    hipFree(d_e_jones);
    hipFree(d_weights_work);
    hipFree(d_l);
    hipFree(d_m);
    hipFree(d_n);
    hipFree(d_vis);
    hipFree(d_work_uvw);
    hipFree(d_work_k);

    // Catch any CUDA errors and return
    hipError_t error = hipPeekAtLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA ERROR [%i] from oskar_interferometer1_scalar(): %s\n",
                (int)error, hipGetErrorString(error));
    }
    return (int)error;
}



int oskar_interferometer1_scalar_f(
        const oskar_TelescopeModel_f telescope,
        const oskar_StationModel_f * stations,
        const oskar_SkyModelGlobal_f sky,
        const float ra0_rad,
        const float dec0_rad,
        const float obs_start_mjd_utc,
        const float obs_length_days,
        const unsigned num_vis_dumps,
        const unsigned num_vis_ave,
        const unsigned num_fringe_ave,
        const float frequency,
        const float bandwidth,
        const bool disable_e_jones,
        oskar_VisData_f* h_vis
)
{
    const float sec_per_day = 86400.0f;

    // === Evaluate number of stations and number of baselines.
    const unsigned num_stations = telescope.num_antennas;
    const unsigned num_baselines = num_stations * (num_stations - 1) / 2;

    const float lambda     = 299792458.0f / frequency;
    const float wavenumber = 2.0f * M_PI / lambda;

    // === Allocate device memory for telescope and transfer to device.
    oskar_TelescopeModel_f hd_telescope;
    oskar_copy_telescope_to_device_f(&telescope, &hd_telescope);

    // === Scale telescope coordinates to wavenumber units.
    oskar_scale_device_telescope_coords_f(&hd_telescope, (float)wavenumber);

    // === Allocate memory to hold station uvw coordinates.
    float* station_u = (float*)malloc(num_stations * sizeof(float));
    float* station_v = (float*)malloc(num_stations * sizeof(float));
    float* station_w = (float*)malloc(num_stations * sizeof(float));

    // === Allocate device memory for antennas and transfer to the device.
    size_t mem_size = num_stations * sizeof(oskar_StationModel_f);
    oskar_StationModel_f* hd_stations = (oskar_StationModel_f*)malloc(mem_size);
    oskar_copy_stations_to_device_f(stations, num_stations, hd_stations);

    // === Scale station coordinates to wavenumber units.
    oskar_scale_station_coords_f(num_stations, hd_stations, (float)wavenumber);

    // === Allocate device memory for source model and transfer to device.
    oskar_SkyModelGlobal_f hd_sky_global;
    for (int i = 0; i < sky.num_sources; ++i)
        sky.I[i] = sqrt(sky.I[i]);
    oskar_copy_gobal_sky_to_device_f(&sky, &hd_sky_global);

    // === Allocate local sky structure.
    oskar_SkyModelLocal_f hd_sky_local;
    oskar_allocate_device_local_sky_f(sky.num_sources, &hd_sky_local);

    // === Allocate device memory for station beam patterns.
    float2 * d_e_jones = NULL;
    size_t mem_e_jones = telescope.num_antennas * sky.num_sources * sizeof(float2);
    hipMalloc((void**)&d_e_jones, mem_e_jones);

    // === Allocate device memory for beamforming weights buffer.
    float2* d_weights_work;
    alloc_beamforming_weights_buffer_f(telescope.num_antennas, stations,
            &d_weights_work);

    // === Allocate device memory for source positions in relative lmn coordinates.
    float *d_l, *d_m, *d_n;
    hipMalloc((void**)&d_l, sky.num_sources * sizeof(float));
    hipMalloc((void**)&d_m, sky.num_sources * sizeof(float));
    hipMalloc((void**)&d_n, sky.num_sources * sizeof(float));

    // === Allocate device memory for visibilities.
    float2* d_vis;
    size_t mem_size_vis = num_baselines * sizeof(float2);
    hipMalloc((void**)&d_vis, mem_size_vis);

    // === Allocate device memory for correlator work buffer.
    float* d_work_uvw;
    hipMalloc((void**)&d_work_uvw, 3 * num_stations * sizeof(float));
    float2* d_work_k;
    hipMalloc((void**)&d_work_k, num_stations * hd_sky_global.num_sources *  sizeof(float2));

    // === Calculate time increments.
    unsigned total_samples  = num_vis_dumps * num_fringe_ave * num_vis_ave;
    float dt_days           = obs_length_days / total_samples;
    float dt_vis_days       = obs_length_days / num_vis_dumps;
    float dt_vis_ave_days   = dt_vis_days / num_vis_ave;
    float dt_vis_offset     = dt_vis_days / 2.0;
    float dt_vis_ave_offset = dt_vis_ave_days / 2.0;

    // === Loop over number of visibility snapshots.
    for (unsigned j = 0; j < num_vis_dumps; ++j)
    {
        printf("--> simulating visibility snapshot %i of %i ...\n", j+1, num_vis_dumps);

        // Start time for the visibility dump (in mjd utc)
        float t_vis_dump_start = obs_start_mjd_utc + (j * dt_vis_days);

        // Initialise visibilities for the dump to zero.
        hipMemset((void*)d_vis, 0, mem_size_vis);

        // Loop over evaluations of the visibility average with changing
        // E-Jones within the dump.
        for (unsigned i = 0; i < num_vis_ave; ++i)
        {
            // Evaluate lst
            float t_ave_start = t_vis_dump_start + i * dt_vis_ave_days;
            float t_ave_mid   = t_ave_start + dt_vis_ave_offset;
            float lst = oskar_mjd_to_last_fast_d(t_ave_mid, telescope.longitude);

            // Find sources above horizon.
            oskar_cuda_horizon_clip_f(&hd_sky_global, lst, telescope.latitude,
                    &hd_sky_local);

            if (hd_sky_local.num_sources == 0)
            {
                fprintf(stderr, "WARNING: no sources above horizon! (this will fail!)\n");

            }

            // Evaluate horizontal lm for the beam phase centre.
            float h_beam_l, h_beam_m, h_beam_n;
            oskar_ra_dec_to_hor_lmn_f(1, &ra0_rad, &dec0_rad, lst,
                    telescope.latitude, &h_beam_l, &h_beam_m, &h_beam_n);

            // Evaluate E-Jones for each source position per station
            oskar_evaluate_station_beams_f(num_stations, hd_stations,
                    &hd_sky_local, h_beam_l, h_beam_m, d_weights_work,
                    disable_e_jones, telescope.identical_stations, d_e_jones);

            // Multiply e-jones by source brightness.
            oskar_mult_beampattern_by_source_field_amp_f(num_stations,
                    &hd_sky_local, d_e_jones);

            // Convert source positions in ra, dec to lmn relative to the phase
            // centre.
            oskar_cuda_ra_dec_to_relative_lmn_f(hd_sky_local.num_sources,
                    hd_sky_local.RA, hd_sky_local.Dec, ra0_rad, dec0_rad,
                    d_l, d_m, d_n);

            // Correlator which updates phase matrix.
            float lst_start = oskar_mjd_to_last_fast_f(t_ave_start,
                    telescope.longitude);
            oskar_cuda_correlator_scalar_f((int)num_stations,
                    hd_telescope.antenna_x, hd_telescope.antenna_y,
                    hd_telescope.antenna_z, hd_sky_local.num_sources,
                    d_l, d_m, d_n, d_e_jones, ra0_rad, dec0_rad, lst_start,
                    num_fringe_ave, dt_days * sec_per_day, lambda * bandwidth,
                    d_work_k, d_work_uvw, d_vis);
        }

        // copy back the vis dump into host memory.
        hipMemcpy(&(h_vis->amp[num_baselines * j]), d_vis, mem_size_vis,
                hipMemcpyDeviceToHost);

        // Evaluate baseline coordinates for the visibility dump.
        float* u = &(h_vis->u[num_baselines * j]);
        float* v = &(h_vis->v[num_baselines * j]);
        float* w = &(h_vis->w[num_baselines * j]);
        float t_vis = t_vis_dump_start + dt_vis_offset;
        float lst_vis_dump = oskar_mjd_to_last_fast_d(t_vis, telescope.longitude);
        float ha_vis = lst_vis_dump - ra0_rad;
        oskar_xyz_to_uvw_f(num_stations, telescope.antenna_x, telescope.antenna_y,
                telescope.antenna_z, ha_vis, dec0_rad, station_u, station_v,
                station_w);
        oskar_compute_baselines_f(num_stations, station_u, station_v, station_w,
                u, v, w);
    }


    // free memory
    oskar_free_device_telescope_f(&hd_telescope);
    free(station_u);
    free(station_v);
    free(station_w);
    for (unsigned i = 0; i < num_stations; ++i)
    {
        hipFree(hd_stations[i].antenna_x);
        hipFree(hd_stations[i].antenna_y);
    }
    free(hd_stations);

    oskar_free_device_global_sky_f(&hd_sky_global);
    oskar_free_device_local_sky_f(&hd_sky_local);

    hipFree(d_e_jones);
    hipFree(d_weights_work);

    hipFree(d_l);
    hipFree(d_m);
    hipFree(d_n);

    hipFree(d_vis);

    hipFree(d_work_uvw);
    hipFree(d_work_k);

    hipError_t error = hipPeekAtLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA ERROR [%i] from oskar_interferometer1_scalar(): %s\n",
                (int)error, hipGetErrorString(error));
    }

    return (int)error;
}






//==============================================================================
void alloc_beamforming_weights_buffer_d(const unsigned num_stations,
        const oskar_StationModel_d* stations, double2** d_weights)
{
    unsigned num_antennas_max = 0;
    for (unsigned i = 0; i < num_stations; ++i)
    {
        if (stations[i].num_antennas > num_antennas_max)
            num_antennas_max = stations[i].num_antennas;
    }
    hipMalloc((void**)d_weights, num_antennas_max * sizeof(double2));
}



void alloc_beamforming_weights_buffer_f(const unsigned num_stations,
        const oskar_StationModel_f* stations, float2** d_weights)
{
    unsigned num_antennas_max = 0;
    for (unsigned i = 0; i < num_stations; ++i)
    {
        if (stations[i].num_antennas > num_antennas_max)
            num_antennas_max = stations[i].num_antennas;
    }
    hipMalloc((void**)d_weights, num_antennas_max * sizeof(float2));
}
//==============================================================================


#ifdef __cplusplus
}
#endif
