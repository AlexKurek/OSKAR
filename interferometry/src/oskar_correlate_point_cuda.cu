#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011-2013, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/oskar_accumulate_baseline_visibility_for_source.h"
#include "interferometry/oskar_correlate_point_cuda.h"
#include "math/oskar_sinc.h"

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_correlate_point_cuda_f(int num_sources,
        int num_stations, const float4c* d_jones,
        const float* d_source_I, const float* d_source_Q,
        const float* d_source_U, const float* d_source_V,
        const float* d_source_l, const float* d_source_m,
        const float* d_station_u, const float* d_station_v,
        float frac_bandwidth, float4c* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(float4c);
    oskar_correlate_point_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_Q, d_source_U,
            d_source_V, d_source_l, d_source_m, d_station_u,
            d_station_v, frac_bandwidth, d_vis);
}

/* Double precision. */
void oskar_correlate_point_cuda_d(int num_sources,
        int num_stations, const double4c* d_jones,
        const double* d_source_I, const double* d_source_Q,
        const double* d_source_U, const double* d_source_V,
        const double* d_source_l, const double* d_source_m,
        const double* d_station_u, const double* d_station_v,
        double frac_bandwidth, double4c* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(double4c);
    oskar_correlate_point_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_Q, d_source_U,
            d_source_V, d_source_l, d_source_m, d_station_u,
            d_station_v, frac_bandwidth, d_vis);
}

#ifdef __cplusplus
}
#endif


/* Kernels. ================================================================ */

/* Indices into the visibility/baseline matrix. */
#define AI blockIdx.x /* Column index. */
#define AJ blockIdx.y /* Row index. */

extern __shared__ float4c  smem_f[];
extern __shared__ double4c smem_d[];

/* Single precision. */
__global__
void oskar_correlate_point_cudak_f(const int num_sources,
        const int num_stations, const float4c* __restrict__ jones,
        const float* __restrict__ source_I,
        const float* __restrict__ source_Q,
        const float* __restrict__ source_U,
        const float* __restrict__ source_V,
        const float* __restrict__ source_l,
        const float* __restrict__ source_m,
        const float* __restrict__ station_u,
        const float* __restrict__ station_v, const float frac_bandwidth,
        float4c* __restrict__ vis)
{
    /* Return immediately if in the wrong half of the visibility matrix. */
    if (AJ >= AI) return;

    /* Common values per thread block. */
    __shared__ float uu, vv;
    if (threadIdx.x == 0)
    {
        /* Determine UV-distance for baseline modified by the bandwidth
         * smearing parameters. */
        uu = 0.5f * frac_bandwidth * (station_u[AI] - station_u[AJ]);
        vv = 0.5f * frac_bandwidth * (station_v[AI] - station_v[AJ]);
    }
    __syncthreads();

    /* Get pointers to both source vectors for station i and j. */
    const float4c* __restrict__ station_i = &jones[num_sources * AI];
    const float4c* __restrict__ station_j = &jones[num_sources * AJ];

    /* Each thread loops over a subset of the sources. */
    {
        float4c sum; /* Partial sum per thread. */
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int t = threadIdx.x; t < num_sources; t += blockDim.x)
        {
            /* Compute bandwidth-smearing term. */
            float rb = oskar_sinc_f(uu * source_l[t] + vv * source_m[t]);

            /* Accumulate baseline visibility response for source. */
            oskar_accumulate_baseline_visibility_for_source_f(&sum, t,
                    source_I, source_Q, source_U, source_V,
                    station_i, station_j, rb);
        }
        smem_f[threadIdx.x] = sum;
    }
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        float4c sum;
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_f[i].a.x;
            sum.a.y += smem_f[i].a.y;
            sum.b.x += smem_f[i].b.x;
            sum.b.y += smem_f[i].b.y;
            sum.c.x += smem_f[i].c.x;
            sum.c.y += smem_f[i].c.y;
            sum.d.x += smem_f[i].d.x;
            sum.d.y += smem_f[i].d.y;
        }

        /* Determine 1D index. */
        int idx = AJ*(num_stations-1) - (AJ-1)*AJ/2 + AI - AJ - 1;

        /* Modify existing visibility. */
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}

/* Double precision. */
__global__
void oskar_correlate_point_cudak_d(const int num_sources,
        const int num_stations, const double4c* __restrict__ jones,
        const double* __restrict__ source_I,
        const double* __restrict__ source_Q,
        const double* __restrict__ source_U,
        const double* __restrict__ source_V,
        const double* __restrict__ source_l,
        const double* __restrict__ source_m,
        const double* __restrict__ station_u,
        const double* __restrict__ station_v, const double frac_bandwidth,
        double4c* __restrict__ vis)
{
    /* Return immediately if in the wrong half of the visibility matrix. */
    if (AJ >= AI) return;

    /* Common values per thread block. */
    __shared__ double uu, vv;
    if (threadIdx.x == 0)
    {
        /* Determine UV-distance for baseline modified by the bandwidth
         * smearing parameters. */
        uu = 0.5 * frac_bandwidth * (station_u[AI] - station_u[AJ]);
        vv = 0.5 * frac_bandwidth * (station_v[AI] - station_v[AJ]);
    }
    __syncthreads();

    /* Get pointers to both source vectors for station i and j. */
    const double4c* __restrict__ station_i = &jones[num_sources * AI];
    const double4c* __restrict__ station_j = &jones[num_sources * AJ];

    /* Each thread loops over a subset of the sources. */
    {
        double4c sum; /* Partial sum per thread. */
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int t = threadIdx.x; t < num_sources; t += blockDim.x)
        {
            /* Compute bandwidth-smearing term. */
            double rb = oskar_sinc_d(uu * source_l[t] + vv * source_m[t]);

            /* Accumulate baseline visibility response for source. */
            oskar_accumulate_baseline_visibility_for_source_d(&sum, t,
                    source_I, source_Q, source_U, source_V,
                    station_i, station_j, rb);
        }
        smem_d[threadIdx.x] = sum;
    }
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        double4c sum;
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_d[i].a.x;
            sum.a.y += smem_d[i].a.y;
            sum.b.x += smem_d[i].b.x;
            sum.b.y += smem_d[i].b.y;
            sum.c.x += smem_d[i].c.x;
            sum.c.y += smem_d[i].c.y;
            sum.d.x += smem_d[i].d.x;
            sum.d.y += smem_d[i].d.y;
        }

        /* Determine 1D index. */
        int idx = AJ*(num_stations-1) - (AJ-1)*AJ/2 + AI - AJ - 1;

        /* Modify existing visibility. */
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}
