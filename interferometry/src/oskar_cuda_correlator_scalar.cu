#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/oskar_cuda_correlator_scalar.h"

#include "interferometry/cudak/oskar_cudak_correlator_scalar.h"
#include "interferometry/cudak/oskar_cudak_xyz_to_uvw.h"

#include "math/cudak/oskar_cudak_dftw_3d_seq_out.h"
#include "math/cudak/oskar_cudak_mat_mul_cc.h"

#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// Single precision.
int oskar_cuda_correlator_scalar_f(int na, const float* ax,
        const float* ay, const float* az, int ns, const float* l,
        const float* m, const float* n, const float2* eb, float ra0,
        float dec0, float lst0, int nsdt, float sdt,
        float lambda_bandwidth, float2* work_k, float* work_uvw, float2* vis)
{
    // Initialise.
    hipError_t errCuda = hipSuccess;
    double tOffset = (double)sdt / 2.0;
    float* u = work_uvw;
    float* v = u + na;
    float* w = v + na;

    // Set up thread blocks.
    dim3 kThd(64, 4); // Sources, antennas.
    dim3 kBlk((ns + kThd.x - 1) / kThd.x, (na + kThd.y - 1) / kThd.y);
    size_t sMem = (kThd.x + kThd.y) * sizeof(float3);
    dim3 mThd(64, 4); // Sources, antennas.
    dim3 mBlk((ns + mThd.x - 1) / mThd.x, (na + mThd.y - 1) / mThd.y);
    dim3 vThd(256, 1); // Antennas, antennas.
    dim3 vBlk(na, na);
    size_t vsMem = vThd.x * sizeof(float2);
    dim3 rThd(256, 1); // Antennas.
    dim3 rBlk((na + rThd.x - 1) / rThd.x, 1);

    // Loop over integrations.
    for (int i = 0; i < nsdt; ++i)
    {
        // Compute the current LST and hour angle of the phase centre.
        double tInc = i * sdt + tOffset;
        double lst = lst0 + 2 * M_PI * tInc / 86400.0; // Must be double.
        double ha0 = lst - ra0; // Must be double.

        // Compute the station u,v,w coordinates.
        oskar_cudak_xyz_to_uvw_f <<<rThd, rBlk>>>
                (na, ax, ay, az, ha0, dec0, u, v, w);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;

        // Compute K-matrix of 3D DFT weights.
        oskar_cudak_dftw_3d_seq_out_f <<<kBlk, kThd, sMem>>>
                (na, u, v, w, ns, l, m, n, work_k);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;

        // Perform complex matrix element multiply of K with E * B.
        oskar_cudak_mat_mul_cc_f <<<mBlk, mThd>>>
                (ns, na, work_k, eb, work_k);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;

        // Call the correlator kernel.
        oskar_cudak_correlator_scalar_f <<<vBlk, vThd, vsMem>>>
                (ns, na, work_k, u, v, l, m, lambda_bandwidth, vis);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;
    }

    return 0;
}




// Double precision.
int oskar_cuda_correlator_scalar_d(int na, const double* ax,
        const double* ay, const double* az, int ns, const double* l,
        const double* m, const double* n, const double2* eb, double ra0,
        double dec0, double lst0, int nsdt, double sdt,
        double lambda_bandwidth, double2* work_k, double* work_uvw, double2* vis)
{
    // Initialise.
    hipError_t errCuda = hipSuccess;
    double tOffset = (double)sdt / 2.0;
    double* u = work_uvw;
    double* v = u + na;
    double* w = v + na;

    // Set up thread blocks.
    dim3 kThd(64, 4); // Sources, antennas.
    dim3 kBlk((ns + kThd.x - 1) / kThd.x, (na + kThd.y - 1) / kThd.y);
    size_t sMem = (kThd.x + kThd.y) * sizeof(double3);
    dim3 mThd(64, 4); // Sources, antennas.
    dim3 mBlk((ns + mThd.x - 1) / mThd.x, (na + mThd.y - 1) / mThd.y);
    dim3 vThd(256, 1); // Antennas, antennas.
    dim3 vBlk(na, na);
    size_t vsMem = vThd.x * sizeof(double2);
    dim3 rThd(256, 1); // Antennas.
    dim3 rBlk((na + rThd.x - 1) / rThd.x, 1);

    // Loop over integrations.
    for (int i = 0; i < nsdt; ++i)
    {
        // Compute the current LST and hour angle of the phase centre.
        double tInc = i * sdt + tOffset;
        double lst = lst0 + 2 * M_PI * tInc / 86400.0; // Must be double.
        double ha0 = lst - ra0; // Must be double.

        // Compute the station u,v,w coordinates.
        oskar_cudak_xyz_to_uvw_d <<<rThd, rBlk>>>
                (na, ax, ay, az, ha0, dec0, u, v, w);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;

        // Compute K-matrix of 3D DFT weights.
        oskar_cudak_dftw_3d_seq_out_d <<<kBlk, kThd, sMem>>>
                (na, u, v, w, ns, l, m, n, work_k);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;

        // Perform complex matrix element multiply of K with E * B.
        oskar_cudak_mat_mul_cc_d <<<mBlk, mThd>>> (ns, na, work_k, eb, work_k);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;

        // Call the correlator kernel.
        oskar_cudak_correlator_scalar_d <<<vBlk, vThd, vsMem>>>
                (ns, na, work_k, u, v, l, m, lambda_bandwidth, vis);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;
    }

    return 0;
}

#ifdef __cplusplus
}
#endif
