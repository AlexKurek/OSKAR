#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2013, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/oskar_accumulate_baseline_visibility_for_source.h"
#include "interferometry/oskar_correlate_extended_cuda.h"
#include "math/oskar_sinc.h"
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_correlate_extended_cuda_f(int num_sources,
        int num_stations, const float4c* d_jones,
        const float* d_source_I, const float* d_source_Q,
        const float* d_source_U, const float* d_source_V,
        const float* d_source_l, const float* d_source_m,
        const float* d_source_a, const float* d_source_b,
        const float* d_source_c, const float* d_station_u,
        const float* d_station_v, float freq_hz, float bandwidth_hz,
        float4c* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(float4c);
    oskar_correlate_extended_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_Q, d_source_U,
            d_source_V, d_source_l, d_source_m, d_source_a, d_source_b,
            d_source_c, d_station_u, d_station_v, freq_hz, bandwidth_hz, d_vis);
}

/* Double precision. */
void oskar_correlate_extended_cuda_d(int num_sources,
        int num_stations, const double4c* d_jones,
        const double* d_source_I, const double* d_source_Q,
        const double* d_source_U, const double* d_source_V,
        const double* d_source_l, const double* d_source_m,
        const double* d_source_a, const double* d_source_b,
        const double* d_source_c, const double* d_station_u,
        const double* d_station_v, double freq_hz, double bandwidth_hz,
        double4c* d_vis)
{
    dim3 num_threads(128, 1);
    dim3 num_blocks(num_stations, num_stations);
    size_t shared_mem = num_threads.x * sizeof(double4c);
    oskar_correlate_extended_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
    (num_sources, num_stations, d_jones, d_source_I, d_source_Q, d_source_U,
            d_source_V, d_source_l, d_source_m, d_source_a, d_source_b,
            d_source_c, d_station_u, d_station_v, freq_hz, bandwidth_hz, d_vis);
}

#ifdef __cplusplus
}
#endif


/* Kernels. ================================================================ */

#define ONE_OVER_2PI  0.159154943091895335768884   /* 1 / (2 * pi) */
#define ONE_OVER_2PIf 0.159154943091895335768884f  /* 1 / (2 * pi) */

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifndef M_PIf
#define M_PIf 3.14159265358979323846f
#endif

/* Indices into the visibility/baseline matrix. */
#define AI blockIdx.x /* Column index. */
#define AJ blockIdx.y /* Row index. */

extern __shared__ float4c  smem_f[];
extern __shared__ double4c smem_d[];


/* Single precision. */
__global__
void oskar_correlate_extended_cudak_f(const int num_sources,
        const int num_stations, const float4c* jones, const float* source_I,
        const float* source_Q, const float* source_U, const float* source_V,
        const float* source_l, const float* source_m,
        const float* source_a, const float* source_b, const float* source_c,
        const float* station_u, const float* station_v, const float freq_hz,
        const float bandwidth_hz, float4c* vis)
{
    /* Return immediately if in the wrong half of the visibility matrix. */
    if (AJ >= AI) return;

    /* Common things per thread block. */
    __device__ __shared__ float uu, vv, uu2, vv2, uuvv;
    if (threadIdx.x == 0)
    {
        /* Baseline UV-distance, in wavelengths. */
        uu   = (station_u[AI] - station_u[AJ]) * ONE_OVER_2PIf;
        vv   = (station_v[AI] - station_v[AJ]) * ONE_OVER_2PIf;

        /* Quantities needed for evaluating source with gaussian term. */
        uu2  = uu * uu;
        vv2  = vv * vv;
        uuvv = 2.0f * uu * vv;

        /* Modify the baseline UV-distance to include the common components
         * of the bandwidth smearing term. */
        uu *= M_PIf * bandwidth_hz / freq_hz;
        vv *= M_PIf * bandwidth_hz / freq_hz;
    }
    __syncthreads();

    /* Get pointers to both source vectors for station i and j. */
    const float4c* station_i = &jones[num_sources * AI];
    const float4c* station_j = &jones[num_sources * AJ];

    /* Each thread loops over a subset of the sources. */
    {
        float4c sum; /* Partial sum per thread. */
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int t = threadIdx.x; t < num_sources; t += blockDim.x)
        {
            /* Compute bandwidth-smearing term. */
            float rb = oskar_sinc_f(uu * source_l[t] + vv * source_m[t]);

            /* Evaluate gaussian source width term. */
            float f = expf(-(source_a[t] * uu2 +
                    source_b[t] * uuvv + source_c[t] * vv2));

            rb *= f;

            /* Accumulate baseline visibility response for source. */
            oskar_accumulate_baseline_visibility_for_source_f(&sum, t,
                    source_I, source_Q, source_U, source_V,
                    station_i, station_j, rb);
        }
        smem_f[threadIdx.x] = sum;
    }
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        float4c sum;
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_f[i].a.x;
            sum.a.y += smem_f[i].a.y;
            sum.b.x += smem_f[i].b.x;
            sum.b.y += smem_f[i].b.y;
            sum.c.x += smem_f[i].c.x;
            sum.c.y += smem_f[i].c.y;
            sum.d.x += smem_f[i].d.x;
            sum.d.y += smem_f[i].d.y;
        }

        /* Determine 1D index. */
        int idx = AJ*(num_stations-1) - (AJ-1)*AJ/2 + AI - AJ - 1;

        /* Modify existing visibility. */
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}

/* Double precision. */
__global__
void oskar_correlate_extended_cudak_d(const int num_sources,
        const int num_stations, const double4c* jones, const double* source_I,
        const double* source_Q, const double* source_U, const double* source_V,
        const double* source_l, const double* source_m,
        const double* source_a, const double* source_b, const double* source_c,
        const double* station_u, const double* station_v, const double freq_hz,
        const double bandwidth_hz, double4c* vis)
{
    /* Return immediately if in the wrong half of the visibility matrix. */
    if (AJ >= AI) return;

    /* Common things per thread block. */
    __device__ __shared__ double uu, vv, uu2, vv2, uuvv;
    if (threadIdx.x == 0)
    {
        /* Baseline UV-distance, in wavelengths. */
        uu   = (station_u[AI] - station_u[AJ]) * ONE_OVER_2PI;
        vv   = (station_v[AI] - station_v[AJ]) * ONE_OVER_2PI;

        /* Quantities needed for evaluating source with gaussian term. */
        uu2  = uu * uu;
        vv2  = vv * vv;
        uuvv = 2.0 * uu * vv;

        /* Modify the baseline UV-distance to include the common components
         * of the bandwidth smearing term. */
        uu *= M_PI * bandwidth_hz / freq_hz;
        vv *= M_PI * bandwidth_hz / freq_hz;
    }
    __syncthreads();

    /* Get pointers to both source vectors for station i and j. */
    const double4c* station_i = &jones[num_sources * AI];
    const double4c* station_j = &jones[num_sources * AJ];

    /* Each thread loops over a subset of the sources. */
    {
        double4c sum; /* Partial sum per thread. */
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int t = threadIdx.x; t < num_sources; t += blockDim.x)
        {
            /* Compute bandwidth-smearing term. */
            double rb = oskar_sinc_d(uu * source_l[t] + vv * source_m[t]);

            /* Evaluate gaussian source width term. */
            double f = exp(-(source_a[t] * uu2 +
                    source_b[t] * uuvv + source_c[t] * vv2));

            rb *= f;

            /* Accumulate baseline visibility response for source. */
            oskar_accumulate_baseline_visibility_for_source_d(&sum, t,
                    source_I, source_Q, source_U, source_V,
                    station_i, station_j, rb);
        }
        smem_d[threadIdx.x] = sum;
    }
    __syncthreads();

    /* Accumulate contents of shared memory. */
    if (threadIdx.x == 0)
    {
        /* Sum over all sources for this baseline. */
        double4c sum;
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_d[i].a.x;
            sum.a.y += smem_d[i].a.y;
            sum.b.x += smem_d[i].b.x;
            sum.b.y += smem_d[i].b.y;
            sum.c.x += smem_d[i].c.x;
            sum.c.y += smem_d[i].c.y;
            sum.d.x += smem_d[i].d.x;
            sum.d.y += smem_d[i].d.y;
        }

        /* Determine 1D index. */
        int idx = AJ*(num_stations-1) - (AJ-1)*AJ/2 + AI - AJ - 1;

        /* Modify existing visibility. */
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}
