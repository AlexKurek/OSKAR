#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2014, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_evaluate_jones_K_cuda.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_evaluate_jones_K_cuda_f(float2* d_jones, int num_stations,
        float wavenumber, const float* d_u, const float* d_v, const float* d_w,
        int num_sources, const float* d_l, const float* d_m, const float* d_n)
{
    /* Define block and grid sizes. */
    const dim3 num_threads(64, 4); /* Sources, stations. */
    const dim3 num_blocks((num_sources + num_threads.x - 1) / num_threads.x,
            (num_stations + num_threads.y - 1) / num_threads.y);
    const size_t s_mem = 3 * (num_threads.x + num_threads.y) * sizeof(float);

    /* Compute DFT phase weights for K. */
    oskar_evaluate_jones_K_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads, s_mem)
    (num_stations, wavenumber, d_u, d_v, d_w, num_sources, d_l, d_m, d_n,
            d_jones);
}

/* Double precision. */
void oskar_evaluate_jones_K_cuda_d(double2* d_jones, int num_stations,
        double wavenumber, const double* d_u, const double* d_v,
        const double* d_w, int num_sources, const double* d_l,
        const double* d_m, const double* d_n)
{
    /* Define block and grid sizes. */
    const dim3 num_threads(64, 4); /* Sources, stations. */
    const dim3 num_blocks((num_sources + num_threads.x - 1) / num_threads.x,
            (num_stations + num_threads.y - 1) / num_threads.y);
    const size_t s_mem = 3 * (num_threads.x + num_threads.y) * sizeof(double);

    /* Compute DFT phase weights for K. */
    oskar_evaluate_jones_K_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads, s_mem)
    (num_stations, wavenumber, d_u, d_v, d_w, num_sources, d_l, d_m, d_n,
            d_jones);
}


/* Kernels. ================================================================ */

/* Shared memory pointers used by the kernels. */
extern __shared__ float smem_f[];
extern __shared__ double smem_d[];

/* Single precision. */
__global__
void oskar_evaluate_jones_K_cudak_f(const int num_stations,
        const float wavenumber, const float* u, const float* v,
        const float* w, const int num_sources, const float* l,
        const float* m, const float* n, float2* jones)
{
    const int s = blockDim.x * blockIdx.x + threadIdx.x; /* Output index. */
    const int a = blockDim.y * blockIdx.y + threadIdx.y; /* Input index. */

    /* Cache input and output data from global memory. */
    float* l_ = smem_f;
    float* m_ = &l_[blockDim.x];
    float* n_ = &m_[blockDim.x];
    float* u_ = &n_[blockDim.x];
    float* v_ = &u_[blockDim.y];
    float* w_ = &v_[blockDim.y];
    if (s < num_sources && threadIdx.y == 0)
    {
        l_[threadIdx.x] = l[s];
        m_[threadIdx.x] = m[s];
        n_[threadIdx.x] = n[s] - 1.0f;
    }
    if (a < num_stations && threadIdx.x == 0)
    {
        u_[threadIdx.y] = wavenumber * u[a];
        v_[threadIdx.y] = wavenumber * v[a];
        w_[threadIdx.y] = wavenumber * w[a];
    }
    __syncthreads();

    /* Compute the geometric phase of the output direction. */
    float phase;
    phase =  u_[threadIdx.y] * l_[threadIdx.x];
    phase += v_[threadIdx.y] * m_[threadIdx.x];
    phase += w_[threadIdx.y] * n_[threadIdx.x];
    float2 weight;
    sincosf(phase, &weight.y, &weight.x);

    /* Write result to global memory. */
    if (s < num_sources && a < num_stations)
    {
        const int w = s + num_sources * a;
        jones[w] = weight;
    }
}

/* Double precision. */
__global__
void oskar_evaluate_jones_K_cudak_d(const int num_stations,
        const double wavenumber, const double* u, const double* v,
        const double* w, const int num_sources, const double* l,
        const double* m, const double* n, double2* jones)
{
    const int s = blockDim.x * blockIdx.x + threadIdx.x; /* Output index. */
    const int a = blockDim.y * blockIdx.y + threadIdx.y; /* Input index. */

    /* Cache input and output data from global memory. */
    double* l_ = smem_d;
    double* m_ = &l_[blockDim.x];
    double* n_ = &m_[blockDim.x];
    double* u_ = &n_[blockDim.x];
    double* v_ = &u_[blockDim.y];
    double* w_ = &v_[blockDim.y];
    if (s < num_sources && threadIdx.y == 0)
    {
        l_[threadIdx.x] = l[s];
        m_[threadIdx.x] = m[s];
        n_[threadIdx.x] = n[s] - 1.0;
    }
    if (a < num_stations && threadIdx.x == 0)
    {
        u_[threadIdx.y] = wavenumber * u[a];
        v_[threadIdx.y] = wavenumber * v[a];
        w_[threadIdx.y] = wavenumber * w[a];
    }
    __syncthreads();

    /* Compute the geometric phase of the output direction. */
    double phase;
    phase =  u_[threadIdx.y] * l_[threadIdx.x];
    phase += v_[threadIdx.y] * m_[threadIdx.x];
    phase += w_[threadIdx.y] * n_[threadIdx.x];
    double2 weight;
    sincos(phase, &weight.y, &weight.x);

    /* Write result to global memory. */
    if (s < num_sources && a < num_stations)
    {
        const int w = s + num_sources * a;
        jones[w] = weight;
    }
}

#ifdef __cplusplus
}
#endif

