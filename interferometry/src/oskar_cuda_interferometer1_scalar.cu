#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/oskar_cuda_interferometer1_scalar.h"
#include "interferometry/oskar_cuda_correlator_scalar.h"

#include "math/cudak/oskar_math_cudak_dftw_3d_seq_out.h"
#include "math/cudak/oskar_math_cudak_mat_mul_cc.h"

#include "interferometry/cudak/oskar_cudak_correlator.h"
#include "interferometry/cudak/oskar_cudak_xyz2uvw.h"

#include <cstdio>


#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define SEC_PER_DAY 86400.0

#ifdef __cplusplus
extern "C" {
#endif

//------------------------------------------------------------------------------
void oskar_cudad_copy_telescope_to_gpu(const struct TelescopeModel * h_telescope,
        struct TelescopeModel * d_telescope);

void oskar_cudad_copy_stations_to_gpu(const struct StationModel * h_stations,
        const unsigned num_stations, struct StationModel * d_stations);

void oskar_cudad_copy_sky_to_gpu(const struct SkyModel * h_sky,
        struct SkyModel * d_sky);
//------------------------------------------------------------------------------



int oskar_cudad_interferometer1_scalar(

        const struct TelescopeModel telescope, // NOTE: In ITRS coordinates

        const struct StationModel * stations,

        const struct SkyModel sky,

        const double ra0_rads,
        const double dec0_rads,

        const double start_date_utc,
        const unsigned nsdt,
        const double sdt,

        const double lambda_bandwidth,

        double * vis // FIXME float2?
){
    hipError_t cuda_error = hipSuccess;

    // === Evaluate number of stations and number of baselines.
    const unsigned num_stations = telescope.num_antennas;
    const unsigned num_baselines = num_stations * (num_stations - 1) / 2;

    // === Allocate device memory for telescope and transfer to device.
    struct TelescopeModel d_telescope;
    oskar_cudad_copy_telescope_to_gpu(&telescope, &d_telescope);

    // === Allocate device memory for antennas and transfer to the device.
    size_t mem_size = num_stations * sizeof(StationModel);
    struct StationModel * d_stations = (StationModel*)malloc(mem_size);
    oskar_cudad_copy_stations_to_gpu(stations, num_stations, d_stations);

    // === Allocate device memory for source model and transfer to device.
    struct SkyModel d_sky;
    oskar_cudad_copy_sky_to_gpu(&sky, &d_sky);









    // TODO: Transform the station positions to the local equatorial system.
    //[X, Y, Z] = horizon_plane_to_itrs(Xh, Yh, lat);

    int num_vis_snapshots = 0;
    int num_vis_averages = 0;


    // 4. Loop over number of visibility snapshots.
    for (int j = 0; j < num_vis_snapshots; ++j)
    {
        // 5. Evaluate LST from UTC.
        // TODO

        // 6. Loop over evaluations of the visibility average with changing E-Jones
        // within the dump
        for (int i = 0; i < num_vis_averages; ++i)
        {
            // 6. Find sources above horizon.
            // TODO

            // 7. Evaluate E-Jones for each source position per station.
            // TODO: optimisation if all stations are the same?
            //for each station
            //      oskar_cudad_bp2hc() <=== move this to beamforming folder.

            // 8. Correlator which updates phase matrix.
            // TODO
            // oskar_cudad_correlator_sclar()

            // 9. Accumulate visibilities.
            // TODO
        }

        // 10. Dump a new set of visibilities including baseline coordinates.
        // TODO
    }


    return (int)cuda_error;
}




void oskar_cudad_copy_telescope_to_gpu(const struct TelescopeModel * h_telescope,
        struct TelescopeModel * d_telescope)
{
    size_t mem_size = h_telescope->num_antennas * sizeof(double);

    d_telescope->num_antennas = h_telescope->num_antennas;

    hipMalloc((void**)&(d_telescope->antenna_x), mem_size);
    hipMalloc((void**)&(d_telescope->antenna_y), mem_size);
    hipMalloc((void**)&(d_telescope->antenna_z), mem_size);

    hipMemcpy(d_telescope->antenna_x, h_telescope->antenna_x, mem_size,
            hipMemcpyHostToDevice);
    hipMemcpy(d_telescope->antenna_y, h_telescope->antenna_y, mem_size,
            hipMemcpyHostToDevice);
    hipMemcpy(d_telescope->antenna_z, h_telescope->antenna_z, mem_size,
            hipMemcpyHostToDevice);
}


void oskar_cudad_copy_stations_to_gpu(const struct StationModel * h_stations,
        const unsigned num_stations, struct StationModel * d_stations)
{
    // Allocate and copy memory for each station.
    for (unsigned i = 0; i < num_stations; ++i)
    {
        d_stations[i].num_antennas = h_stations[i].num_antennas;

        size_t mem_size = d_stations[i].num_antennas * sizeof(double);
        hipMalloc((void**)&(d_stations[i].antenna_x), mem_size);
        hipMalloc((void**)&(d_stations[i].antenna_y), mem_size);

        hipMemcpy(d_stations[i].antenna_x, h_stations[i].antenna_x, mem_size,
                hipMemcpyHostToDevice);
        hipMemcpy(d_stations[i].antenna_y, h_stations[i].antenna_y, mem_size,
                hipMemcpyHostToDevice);
    }
}


void oskar_cudad_copy_sky_to_gpu(const struct SkyModel * h_sky,
        struct SkyModel * d_sky)
{
    // TODO: work out what needs to be in here...
}


#ifdef __cplusplus
}
#endif
