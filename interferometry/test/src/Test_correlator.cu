#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/test/Test_correlator.h"
#include "interferometry/cudak/oskar_cudak_correlator.h"
#include "utility/oskar_vector_types.h"
#include "utility/oskar_cuda_device_info_scan.h"

#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <vector>

#define TIMER_ENABLE 1
#include "utility/timer.h"

/**
 * @details
 * Constructor.
 */
Test_correlator::Test_correlator()
{
    device_ = new oskar_CudaDeviceInfo;
    oskar_cuda_device_info_scan(device_, 0);
}

/**
 * @details
 * Destructor.
 */
Test_correlator::~Test_correlator()
{
    delete device_;
}

/**
 * @details
 * Tests correlator kernel.
 */
void Test_correlator::test_kernel_float()
{
    int ns = 50000;
    int na = 25;
    int nb = na * (na - 1) / 2;
    float lambda_bandwidth = 0.0f;
    std::vector<float> h_I(ns);
    std::vector<float> h_Q(ns);
    std::vector<float> h_U(ns);
    std::vector<float> h_V(ns);
    std::vector<float> h_l(ns);
    std::vector<float> h_m(ns);
    std::vector<float> h_u(na);
    std::vector<float> h_v(na);
    std::vector<float4c> h_vis(nb);
    std::vector<float4c> h_jones(ns * na);

    for (int s = 0; s < ns; ++s)
    {
        h_I[0] = 2.0 * (s+1);
        h_Q[0] = 0.5 * (s+1);
        h_U[0] = 0.3 * (s+1);
        h_V[0] = 0.1 * (s+1);
    }

    for (int a = 0; a < na; ++a)
    {
        for (int s = 0; s < ns; ++s)
        {
            int i = s + a * ns;
            h_jones[i].a = make_float2(float(8*i + 0), float(8*i + 1));
            h_jones[i].b = make_float2(float(8*i + 2), float(8*i + 3));
            h_jones[i].c = make_float2(float(8*i + 4), float(8*i + 5));
            h_jones[i].d = make_float2(float(8*i + 6), float(8*i + 7));
        }
    }

    // Allocate device memory.
    float *d_I, *d_Q, *d_U, *d_V, *d_l, *d_m, *d_u, *d_v;
    float4c *d_jones, *d_vis;
    hipMalloc((void**)&d_I, ns * sizeof(float));
    hipMalloc((void**)&d_Q, ns * sizeof(float));
    hipMalloc((void**)&d_U, ns * sizeof(float));
    hipMalloc((void**)&d_V, ns * sizeof(float));
    hipMalloc((void**)&d_l, ns * sizeof(float));
    hipMalloc((void**)&d_m, ns * sizeof(float));
    hipMalloc((void**)&d_u, na * sizeof(float));
    hipMalloc((void**)&d_v, na * sizeof(float));
    hipMalloc((void**)&d_jones, ns * na * sizeof(float4c));
    hipMalloc((void**)&d_vis, nb * sizeof(float4c));
    hipMemcpy(d_I, &h_I[0], ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Q, &h_Q[0], ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_U, &h_U[0], ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, &h_V[0], ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_l, &h_l[0], ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, &h_m[0], ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_u, &h_u[0], na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, &h_v[0], na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_jones, &h_jones[0], ns * na * sizeof(float4c),
            hipMemcpyHostToDevice);
    int err = hipPeekAtLastError();
    if (err)
    {
        fprintf(stderr, "CUDA ERROR[%d]: %s.\n", err,
                hipGetErrorString((hipError_t)err));
        CPPUNIT_FAIL("CUDA Error allocating memory.");
    }

    // Call the correlator kernel.
    int num_threads = 0;
    if (device_->compute.capability.major < 2 &&
            device_->compute.capability.minor < 3)
        num_threads = 128;
    else
        num_threads = 256;

    dim3 vThd(num_threads, 1); // Antennas, antennas.
    dim3 vBlk(na, na);
    size_t vsMem = vThd.x * sizeof(float4c);
    TIMER_START
    oskar_cudak_correlator_f <<<vBlk, vThd, vsMem>>> (ns, na, d_jones,
            d_I, d_Q, d_U, d_V, d_u, d_v, d_l, d_m, lambda_bandwidth, d_vis);
    hipDeviceSynchronize();
    TIMER_STOP("Finished correlator kernel (float), %d sources", ns)
    err = hipPeekAtLastError();
    if (err)
    {
        fprintf(stderr, "CUDA ERROR[%d]: %s.\n", err,
                hipGetErrorString((hipError_t)err));
        CPPUNIT_FAIL("CUDA Error from oskar_cudak_correlator_f().");
    }

    // Copy memory back to host.
    hipMemcpy(&h_vis[0], d_vis, nb * sizeof(float4c), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_I);
    hipFree(d_Q);
    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_l);
    hipFree(d_m);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_jones);
    hipFree(d_vis);
}

/**
 * @details
 * Tests correlator kernel.
 */
void Test_correlator::test_kernel_double()
{
    if (!device_->supports_double)
        return;

    int ns = 50000;
    int na = 25;
    int nb = na * (na - 1) / 2;
    double lambda_bandwidth = 0.0;
    std::vector<double> h_I(ns);
    std::vector<double> h_Q(ns);
    std::vector<double> h_U(ns);
    std::vector<double> h_V(ns);
    std::vector<double> h_l(ns);
    std::vector<double> h_m(ns);
    std::vector<double> h_u(na);
    std::vector<double> h_v(na);
    std::vector<double4c> h_vis(nb);
    std::vector<double4c> h_jones(ns * na);

    for (int s = 0; s < ns; ++s)
    {
        h_I[0] = 2.0 * (s+1);
        h_Q[0] = 0.5 * (s+1);
        h_U[0] = 0.3 * (s+1);
        h_V[0] = 0.1 * (s+1);
    }

    for (int a = 0; a < na; ++a)
    {
        for (int s = 0; s < ns; ++s)
        {
            int i = s + a * ns;
            h_jones[i].a = make_double2(double(8*i + 0), double(8*i + 1));
            h_jones[i].b = make_double2(double(8*i + 2), double(8*i + 3));
            h_jones[i].c = make_double2(double(8*i + 4), double(8*i + 5));
            h_jones[i].d = make_double2(double(8*i + 6), double(8*i + 7));
        }
    }

    // Allocate device memory.
    double *d_I, *d_Q, *d_U, *d_V, *d_l, *d_m, *d_u, *d_v;
    double4c *d_jones, *d_vis;
    hipMalloc((void**)&d_I, ns * sizeof(double));
    hipMalloc((void**)&d_Q, ns * sizeof(double));
    hipMalloc((void**)&d_U, ns * sizeof(double));
    hipMalloc((void**)&d_V, ns * sizeof(double));
    hipMalloc((void**)&d_l, ns * sizeof(double));
    hipMalloc((void**)&d_m, ns * sizeof(double));
    hipMalloc((void**)&d_u, na * sizeof(double));
    hipMalloc((void**)&d_v, na * sizeof(double));
    hipMalloc((void**)&d_jones, ns * na * sizeof(double4c));
    hipMalloc((void**)&d_vis, nb * sizeof(double4c));
    hipMemcpy(d_I, &h_I[0], ns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Q, &h_Q[0], ns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_U, &h_U[0], ns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V, &h_V[0], ns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_l, &h_l[0], ns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_m, &h_m[0], ns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_u, &h_u[0], na * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, &h_v[0], na * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_jones, &h_jones[0], ns * na * sizeof(double4c),
            hipMemcpyHostToDevice);

    // Call the correlator kernel.
    dim3 vThd(192, 1); // Antennas, antennas. (Note: changed from 256 to 192 for 1.3 arch)
    dim3 vBlk(na, na);
    size_t vsMem = vThd.x * sizeof(double4c);
    TIMER_START
    oskar_cudak_correlator_d <<<vBlk, vThd, vsMem>>> (ns, na, d_jones,
            d_I, d_Q, d_U, d_V, d_u, d_v, d_l, d_m, lambda_bandwidth, d_vis);
    hipDeviceSynchronize();
    TIMER_STOP("Finished correlator kernel (double), %d sources", ns)
    int err = hipPeekAtLastError();
    if (err)
    {
        fprintf(stderr, "CUDA ERROR[%d]: %s.\n", err,
                hipGetErrorString((hipError_t)err));
        CPPUNIT_FAIL("CUDA Error");
    }

    // Copy memory back to host.
    hipMemcpy(&h_vis[0], d_vis, nb * sizeof(double4c), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_I);
    hipFree(d_Q);
    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_l);
    hipFree(d_m);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_jones);
    hipFree(d_vis);
}
