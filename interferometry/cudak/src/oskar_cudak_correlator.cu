#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/cudak/oskar_cudak_correlator.h"
#include "math/cudak/oskar_cudaf_mul_mat2c_mat2c.h"
#include "math/cudak/oskar_cudaf_mul_mat2c_mat2h.h"
#include "math/cudak/oskar_cudaf_mul_mat2c_mat2c_conj_trans.h"
#include "math/cudak/oskar_cudaf_sinc.h"


#define ONE_OVER_2C 1.66782047599076024788E-9   // 1 / (2c)
#define ONE_OVER_2Cf 1.66782047599076024788E-9f // 1 / (2c)

// Indices into the visibility/baseline matrix.
#define AI blockIdx.x // Column index.
#define AJ blockIdx.y // Row index.

extern __shared__ float4c  smem_f[];
extern __shared__ double4c smem_d[];

// Single precision.
__global__
void oskar_cudak_correlator_f(const int ns, const int na,
        const float4c* jones, const float* source_I, const float* source_Q,
        const float* source_U, const float* source_V, const float* u,
        const float* v, const float* l, const float* m,
        const float lambda_bandwidth, float4c* vis)
{
    // Return immediately if we're in the lower triangular half of the
    // visibility matrix.
    if (AJ >= AI) return;

    // Common things per thread block.
    __device__ __shared__ float uu, vv;
    if (threadIdx.x == 0)
    {
        // Determine UV-distance for baseline (common per thread block).
        uu = ONE_OVER_2Cf * lambda_bandwidth * (u[AI] - u[AJ]);
        vv = ONE_OVER_2Cf * lambda_bandwidth * (v[AI] - v[AJ]);
    }
    __syncthreads();

    // Get pointers to both source vectors for station i and j.
    const float4c* station_i = &jones[ns * AI];
    const float4c* station_j = &jones[ns * AJ];

    // Each thread loops over a subset of the sources.
    {
        float4c sum; // Partial sum per thread.
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int t = threadIdx.x; t < ns; t += blockDim.x)
        {
            // Compute bandwidth-smearing term first (register optimisation).
            float rb = oskar_cudaf_sinc_f(uu * l[t] + vv * m[t]);

            // Construct source brightness matrix.
            float4c c_b;
            {
                float s_I = source_I[t];
                float s_Q = source_Q[t];
                c_b.b.x = source_U[t];
                c_b.b.y = source_V[t];
                c_b.a.x = s_I + s_Q;
                c_b.d.x = s_I - s_Q;
            }

            // Multiply first Jones matrix with source coherency matrix.
            float4c c_a = station_i[t];
            oskar_cudaf_mul_mat2c_mat2h_f(c_a, c_b);

            // Multiply result with second (Hermitian transposed) Jones matrix.
            c_b = station_j[t];
            oskar_cudaf_mul_mat2c_mat2c_conj_trans_f(c_a, c_b);

            // Multiply result by bandwidth-smearing term.
            sum.a.x += c_a.a.x * rb;
            sum.a.y += c_a.a.y * rb;
            sum.b.x += c_a.b.x * rb;
            sum.b.y += c_a.b.y * rb;
            sum.c.x += c_a.c.x * rb;
            sum.c.y += c_a.c.y * rb;
            sum.d.x += c_a.d.x * rb;
            sum.d.y += c_a.d.y * rb;
        }
        smem_f[threadIdx.x] = sum;
    }
    __syncthreads();

    // Accumulate contents of shared memory.
    if (threadIdx.x == 0)
    {
        // Sum over all sources for this baseline.
        float4c sum;
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_f[i].a.x;
            sum.a.y += smem_f[i].a.y;
            sum.b.x += smem_f[i].b.x;
            sum.b.y += smem_f[i].b.y;
            sum.c.x += smem_f[i].c.x;
            sum.c.y += smem_f[i].c.y;
            sum.d.x += smem_f[i].d.x;
            sum.d.y += smem_f[i].d.y;
        }

        // Determine 1D index.
        int idx = AJ*(na-1) - (AJ-1)*AJ/2 + AI - AJ - 1;

        // Modify existing visibility.
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}

// Double precision.
__global__
void oskar_cudak_correlator_d(const int ns, const int na,
        const double4c* jones, const double* source_I, const double* source_Q,
        const double* source_U, const double* source_V, const double* u,
        const double* v, const double* l, const double* m,
        const double lambda_bandwidth, double4c* vis)
{
    // Return immediately if we're in the lower triangular half of the
    // visibility matrix.
    if (AJ >= AI) return;

    // Common things per thread block.
    __device__ __shared__ double uu, vv;
    if (threadIdx.x == 0)
    {
        // Determine UV-distance for baseline (common per thread block).
        uu = ONE_OVER_2C * lambda_bandwidth * (u[AI] - u[AJ]);
        vv = ONE_OVER_2C * lambda_bandwidth * (v[AI] - v[AJ]);
    }
    __syncthreads();

    // Get pointers to both source vectors for station i and j.
    const double4c* station_i = &jones[ns * AI];
    const double4c* station_j = &jones[ns * AJ];

    // Each thread loops over a subset of the sources.
    {
        double4c sum; // Partial sum per thread.
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int t = threadIdx.x; t < ns; t += blockDim.x)
        {
            // Compute bandwidth-smearing term first (register optimisation).
            double rb = oskar_cudaf_sinc_d(uu * l[t] + vv * m[t]);

            // Construct source brightness matrix.
            double4c c_b;
            {
                double s_I = source_I[t];
                double s_Q = source_Q[t];
                c_b.b.x = source_U[t];
                c_b.b.y = source_V[t];
                c_b.a.x = s_I + s_Q;
                c_b.d.x = s_I - s_Q;
            }

            // Multiply first Jones matrix with source coherency matrix.
            double4c c_a = station_i[t];
            oskar_cudaf_mul_mat2c_mat2h_d(c_a, c_b);

            // Multiply result with second (Hermitian transposed) Jones matrix.
            c_b = station_j[t];
            oskar_cudaf_mul_mat2c_mat2c_conj_trans_d(c_a, c_b);

            // Multiply result by bandwidth-smearing term.
            sum.a.x += c_a.a.x * rb;
            sum.a.y += c_a.a.y * rb;
            sum.b.x += c_a.b.x * rb;
            sum.b.y += c_a.b.y * rb;
            sum.c.x += c_a.c.x * rb;
            sum.c.y += c_a.c.y * rb;
            sum.d.x += c_a.d.x * rb;
            sum.d.y += c_a.d.y * rb;
        }
        smem_d[threadIdx.x] = sum;
    }
    __syncthreads();

    // Accumulate contents of shared memory.
    if (threadIdx.x == 0)
    {
        // Sum over all sources for this baseline.
        double4c sum;
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_d[i].a.x;
            sum.a.y += smem_d[i].a.y;
            sum.b.x += smem_d[i].b.x;
            sum.b.y += smem_d[i].b.y;
            sum.c.x += smem_d[i].c.x;
            sum.c.y += smem_d[i].c.y;
            sum.d.x += smem_d[i].d.x;
            sum.d.y += smem_d[i].d.y;
        }

        // Determine 1D index.
        int idx = AJ*(na-1) - (AJ-1)*AJ/2 + AI - AJ - 1;

        // Modify existing visibility.
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}
