#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "interferometry/cudak/oskar_cudak_correlator_time_smearing.h"
#include "math/cudak/oskar_cudaf_mul_mat2c_mat2c.h"
#include "math/cudak/oskar_cudaf_mul_mat2c_mat2h.h"
#include "math/cudak/oskar_cudaf_mul_mat2c_mat2c_conj_trans.h"
#include "math/cudak/oskar_cudaf_sinc.h"
#include <math.h>

#define ONE_OVER_2PI  0.159154943091895335768884   // 1 / (2 * pi)
#define ONE_OVER_2PIf 0.159154943091895335768884f  // 1 / (2 * pi)

#define OMEGA_EARTH  7.272205217e-5  // radians/sec
#define OMEGA_EARTHf 7.272205217e-5f // radians/sec

// Indices into the visibility/baseline matrix.
#define SI blockIdx.x // Column index.
#define SJ blockIdx.y // Row index.

extern __shared__ float4c  smem_f[];
extern __shared__ double4c smem_d[];

// Single precision.
__global__
void oskar_cudak_correlator_time_smearing_f(const int num_sources,
        const int num_stations, const float4c* jones, const float* source_I,
        const float* source_Q, const float* source_U, const float* source_V,
        const float* source_l, const float* source_m, const float* source_n,
        const float* station_u, const float* station_v,
        const float* station_x, const float* station_y, const float freq_hz,
        const float bandwidth_hz, const float time_int_sec,
        const float gha0_rad, const float dec0_rad, float4c* vis)
{
    // Return immediately if we're in the lower triangular half of the
    // visibility matrix.
    if (SJ >= SI) return;

    // Common things per thread block.
    __device__ __shared__ float uu, vv;
    __device__ __shared__ float du_dt, dv_dt, dw_dt;
    if (threadIdx.x == 0)
    {
        float xx, yy, rot_angle, temp;
        float fractional_bandwidth, sin_HA, cos_HA, sin_Dec, cos_Dec;

        // Baseline distances, in wavelengths.
        fractional_bandwidth = bandwidth_hz / freq_hz;
        uu = (station_u[SI] - station_u[SJ]) * 0.5f * fractional_bandwidth;
        vv = (station_v[SI] - station_v[SJ]) * 0.5f * fractional_bandwidth;
        xx = (station_x[SI] - station_x[SJ]) * ONE_OVER_2PIf;
        yy = (station_y[SI] - station_y[SJ]) * ONE_OVER_2PIf;

        // Compute the derivatives for time-average smearing.
        rot_angle = OMEGA_EARTHf * time_int_sec;
        sin_HA = sinf(gha0_rad);
        cos_HA = cosf(gha0_rad);
        sin_Dec = sinf(dec0_rad);
        cos_Dec = cosf(dec0_rad);
        temp = (xx * sin_HA + yy * cos_HA) * rot_angle;
        du_dt = (xx * cos_HA - yy * sin_HA) * rot_angle;
        dv_dt = temp * sin_Dec;
        dw_dt = -temp * cos_Dec;
    }
    __syncthreads();

    // Get pointers to both source vectors for station i and j.
    const float4c* station_i = &jones[num_sources * SI];
    const float4c* station_j = &jones[num_sources * SJ];

    // Each thread loops over a subset of the sources.
    {
        float4c sum; // Partial sum per thread.
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int t = threadIdx.x; t < num_sources; t += blockDim.x)
        {
            // Get source direction cosines.
            float l = source_l[t];
            float m = source_m[t];
            float n = source_n[t];

            // Compute bandwidth-smearing term first (register optimisation).
            float rb = oskar_cudaf_sinc_f(uu * l + vv * m);

            // Compute time-smearing term.
            float rt = oskar_cudaf_sinc_f(du_dt * l + dv_dt * m + dw_dt * n);

            rb *= rt;

            // Construct source brightness matrix.
            float4c c_b;
            {
                float s_I = source_I[t];
                float s_Q = source_Q[t];
                c_b.b.x = source_U[t];
                c_b.b.y = source_V[t];
                c_b.a.x = s_I + s_Q;
                c_b.d.x = s_I - s_Q;
            }

            // Multiply first Jones matrix with source coherency matrix.
            float4c c_a = station_i[t];
            oskar_cudaf_mul_mat2c_mat2h_f(c_a, c_b);

            // Multiply result with second (Hermitian transposed) Jones matrix.
            c_b = station_j[t];
            oskar_cudaf_mul_mat2c_mat2c_conj_trans_f(c_a, c_b);

            // Multiply result by smearing term.
            sum.a.x += c_a.a.x * rb;
            sum.a.y += c_a.a.y * rb;
            sum.b.x += c_a.b.x * rb;
            sum.b.y += c_a.b.y * rb;
            sum.c.x += c_a.c.x * rb;
            sum.c.y += c_a.c.y * rb;
            sum.d.x += c_a.d.x * rb;
            sum.d.y += c_a.d.y * rb;
        }
        smem_f[threadIdx.x] = sum;
    }
    __syncthreads();

    // Accumulate contents of shared memory.
    if (threadIdx.x == 0)
    {
        // Sum over all sources for this baseline.
        float4c sum;
        sum.a = make_float2(0.0f, 0.0f);
        sum.b = make_float2(0.0f, 0.0f);
        sum.c = make_float2(0.0f, 0.0f);
        sum.d = make_float2(0.0f, 0.0f);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_f[i].a.x;
            sum.a.y += smem_f[i].a.y;
            sum.b.x += smem_f[i].b.x;
            sum.b.y += smem_f[i].b.y;
            sum.c.x += smem_f[i].c.x;
            sum.c.y += smem_f[i].c.y;
            sum.d.x += smem_f[i].d.x;
            sum.d.y += smem_f[i].d.y;
        }

        // Determine 1D index.
        int idx = SJ*(num_stations-1) - (SJ-1)*SJ/2 + SI - SJ - 1;

        // Modify existing visibility.
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}

// Double precision.
__global__
void oskar_cudak_correlator_time_smearing_d(const int num_sources,
        const int num_stations, const double4c* jones, const double* source_I,
        const double* source_Q, const double* source_U, const double* source_V,
        const double* source_l, const double* source_m, const double* source_n,
        const double* station_u, const double* station_v,
        const double* station_x, const double* station_y, const double freq_hz,
        const double bandwidth_hz, const double time_int_sec,
        const double gha0_rad, const double dec0_rad, double4c* vis)
{
    // Return immediately if we're in the lower triangular half of the
    // visibility matrix.
    if (SJ >= SI) return;

    // Common things per thread block.
    __device__ __shared__ double uu, vv;
    __device__ __shared__ double du_dt, dv_dt, dw_dt;
    if (threadIdx.x == 0)
    {
        double xx, yy, rot_angle, temp;
        double fractional_bandwidth, sin_HA, cos_HA, sin_Dec, cos_Dec;

        // Baseline distances, in wavelengths.
        fractional_bandwidth = bandwidth_hz / freq_hz;
        uu = (station_u[SI] - station_u[SJ]) * 0.5 * fractional_bandwidth;
        vv = (station_v[SI] - station_v[SJ]) * 0.5 * fractional_bandwidth;
        xx = (station_x[SI] - station_x[SJ]) * ONE_OVER_2PI;
        yy = (station_y[SI] - station_y[SJ]) * ONE_OVER_2PI;

        // Compute the derivatives for time-average smearing.
        rot_angle = OMEGA_EARTH * time_int_sec;
        sin_HA = sin(gha0_rad);
        cos_HA = cos(gha0_rad);
        sin_Dec = sin(dec0_rad);
        cos_Dec = cos(dec0_rad);
        temp = (xx * sin_HA + yy * cos_HA) * rot_angle;
        du_dt = (xx * cos_HA - yy * sin_HA) * rot_angle;
        dv_dt = temp * sin_Dec;
        dw_dt = -temp * cos_Dec;
    }
    __syncthreads();

    // Get pointers to both source vectors for station i and j.
    const double4c* station_i = &jones[num_sources * SI];
    const double4c* station_j = &jones[num_sources * SJ];

    // Each thread loops over a subset of the sources.
    {
        double4c sum; // Partial sum per thread.
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int t = threadIdx.x; t < num_sources; t += blockDim.x)
        {
            // Get source direction cosines.
            double l = source_l[t];
            double m = source_m[t];
            double n = source_n[t];

            // Compute bandwidth-smearing term first (register optimisation).
            double rb = oskar_cudaf_sinc_d(uu * l + vv * m);

            // Compute time-smearing term.
            double rt = oskar_cudaf_sinc_d(du_dt * l + dv_dt * m + dw_dt * n);

            rb *= rt;

            // Construct source brightness matrix.
            double4c c_b;
            {
                double s_I = source_I[t];
                double s_Q = source_Q[t];
                c_b.b.x = source_U[t];
                c_b.b.y = source_V[t];
                c_b.a.x = s_I + s_Q;
                c_b.d.x = s_I - s_Q;
            }

            // Multiply first Jones matrix with source coherency matrix.
            double4c c_a = station_i[t];
            oskar_cudaf_mul_mat2c_mat2h_d(c_a, c_b);

            // Multiply result with second (Hermitian transposed) Jones matrix.
            c_b = station_j[t];
            oskar_cudaf_mul_mat2c_mat2c_conj_trans_d(c_a, c_b);

            // Multiply result by smearing term.
            sum.a.x += c_a.a.x * rb;
            sum.a.y += c_a.a.y * rb;
            sum.b.x += c_a.b.x * rb;
            sum.b.y += c_a.b.y * rb;
            sum.c.x += c_a.c.x * rb;
            sum.c.y += c_a.c.y * rb;
            sum.d.x += c_a.d.x * rb;
            sum.d.y += c_a.d.y * rb;
        }
        smem_d[threadIdx.x] = sum;
    }
    __syncthreads();

    // Accumulate contents of shared memory.
    if (threadIdx.x == 0)
    {
        // Sum over all sources for this baseline.
        double4c sum;
        sum.a = make_double2(0.0, 0.0);
        sum.b = make_double2(0.0, 0.0);
        sum.c = make_double2(0.0, 0.0);
        sum.d = make_double2(0.0, 0.0);
        for (int i = 0; i < blockDim.x; ++i)
        {
            sum.a.x += smem_d[i].a.x;
            sum.a.y += smem_d[i].a.y;
            sum.b.x += smem_d[i].b.x;
            sum.b.y += smem_d[i].b.y;
            sum.c.x += smem_d[i].c.x;
            sum.c.y += smem_d[i].c.y;
            sum.d.x += smem_d[i].d.x;
            sum.d.y += smem_d[i].d.y;
        }

        // Determine 1D index.
        int idx = SJ*(num_stations-1) - (SJ-1)*SJ/2 + SI - SJ - 1;

        // Modify existing visibility.
        vis[idx].a.x += sum.a.x;
        vis[idx].a.y += sum.a.y;
        vis[idx].b.x += sum.b.x;
        vis[idx].b.y += sum.b.y;
        vis[idx].c.x += sum.c.x;
        vis[idx].c.y += sum.c.y;
        vis[idx].d.x += sum.d.x;
        vis[idx].d.y += sum.d.y;
    }
}
