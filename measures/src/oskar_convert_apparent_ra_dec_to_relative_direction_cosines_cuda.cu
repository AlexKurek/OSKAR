#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cuda.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

/* Single precision. */
void oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cuda_f(
        int num_points, const float* d_ra, const float* d_dec, float ra0,
        float dec0, float* d_l, float* d_m, float* d_n)
{
    float cosDec0, sinDec0;
    int num_blocks, num_threads = 256;

    /* Compute direction-cosines of RA, Dec relative to reference point. */
    num_blocks = (num_points + num_threads - 1) / num_threads;
    cosDec0 = (float) cos(dec0);
    sinDec0 = (float) sin(dec0);

    oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cudak_f
        OSKAR_CUDAK_CONF(num_blocks, num_threads)
        (num_points, d_ra, d_dec, ra0, cosDec0, sinDec0, d_l, d_m, d_n);
}

/* Double precision. */
void oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cuda_d(
        int num_points, const double* d_ra, const double* d_dec, double ra0,
        double dec0, double* d_l, double* d_m, double* d_n)
{
    double cosDec0, sinDec0;
    int num_blocks, num_threads = 256;

    /* Compute direction-cosines of RA, Dec relative to reference point. */
    num_blocks = (num_points + num_threads - 1) / num_threads;
    cosDec0 = cos(dec0);
    sinDec0 = sin(dec0);

    oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cudak_d
        OSKAR_CUDAK_CONF(num_blocks, num_threads)
        (num_points, d_ra, d_dec, ra0, cosDec0, sinDec0, d_l, d_m, d_n);
}


/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cudak_f(
        const int num_points, const float* ra, const float* dec,
        const float ra0, const float cosDec0, const float sinDec0,
        float* l, float* m, float* n)
{
    float cosLat, sinLat, sinLon, cosLon, relLon, pLat, l_, m_, n_;

    /* Get the position ID that this thread is working on. */
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_points) return;

    /* Copy the input data from global memory. */
    relLon = ra[i];
    pLat = dec[i];

    /* Convert from spherical to tangent-plane. */
    relLon -= ra0;
    sincosf(relLon, &sinLon, &cosLon);
    sincosf(pLat, &sinLat, &cosLat);
    l_ = cosLat * sinLon;
    m_ = cosDec0 * sinLat - sinDec0 * cosLat * cosLon;
    n_ = sinDec0 * sinLat + cosDec0 * cosLat * cosLon;

    /* Store output data. */
    l[i] = l_;
    m[i] = m_;
    n[i] = n_;
}

/* Double precision. */
__global__
void oskar_convert_apparent_ra_dec_to_relative_direction_cosines_cudak_d(
        const int num_points, const double* ra, const double* dec,
        const double ra0, const double cosDec0, const double sinDec0,
        double* l, double* m, double* n)
{
    double cosLat, sinLat, sinLon, cosLon, relLon, pLat, l_, m_, n_;

    /* Get the position ID that this thread is working on. */
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= num_points) return;

    /* Copy the input data from global memory. */
    relLon = ra[i];
    pLat = dec[i];

    /* Convert from spherical to tangent-plane. */
    relLon -= ra0;
    sincos(relLon, &sinLon, &cosLon);
    sincos(pLat, &sinLat, &cosLat);
    l_ = cosLat * sinLon;
    m_ = cosDec0 * sinLat - sinDec0 * cosLat * cosLon;
    n_ = sinDec0 * sinLat + cosDec0 * cosLat * cosLon;

    /* Store output data. */
    l[i] = l_;
    m[i] = m_;
    n[i] = n_;
}

#ifdef __cplusplus
}
#endif
