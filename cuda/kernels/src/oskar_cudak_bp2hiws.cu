#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/kernels/oskar_cudak_bp2hiws.h"
#include "math/core/phase.h"

// Shared memory pointer used by the kernel.
extern __shared__ float2 smem[];

__global__
void oskar_cudak_bp2hiws(const int na, const float* ax, const float* ay,
        const float2* weights, const float2* signals, const int sigStride,
        const int ns, const float* saz, const float* sel, const float k,
        const int maxAntennasPerBlock, float2* image)
{
    // Get the pixel (source position) ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;

    // Get the source position.
    // (NB. Cannot exit on index condition, as all threads are needed later).
    float az = 0.0f, el = 0.0f, sinAz, cosAz, cosEl;
    if (s < ns) {
        az = saz[s];
        el = sel[s];
    }
    cosEl = cosf(el);
    sincosf(az, &sinAz, &cosAz);

    // Initialise shared memory caches.
    // Antenna positions are cached as float2 for speed increase.
    float2 cpx = make_float2(0.0f, 0.0f); // Clear pixel value.
    float2* cwt = smem; // Cached antenna weights.
    float2* csg = cwt + maxAntennasPerBlock; // Cached antenna signals.
    float2* cap = csg + maxAntennasPerBlock; // Cached antenna positions.

    // Cache a block of antenna positions and weights into shared memory.
    for (int as = 0; as < na; as += maxAntennasPerBlock) {
        int antennasInBlock = na - as;
        if (antennasInBlock > maxAntennasPerBlock)
            antennasInBlock = maxAntennasPerBlock;

        // There are blockDim.x threads available - need to copy
        // antennasInBlock pieces of data from global memory.
        for (int t = threadIdx.x; t < antennasInBlock; t += blockDim.x) {
            const int ag = as + t; // Global antenna index.
            cwt[t] = weights[ag];
            csg[t] = signals[ag * sigStride + s];
            cap[t].x = ax[ag];
            cap[t].y = ay[ag];
        }

        // Must synchronise before computing the signal for these antennas.
        __syncthreads();

        // Loop over antennas in block.
        for (int a = 0; a < antennasInBlock; ++a) {
            // Get the signal and the weight.
            float2 signal = csg[a], w = cwt[a];

            float2 srcSig;
            float phaseSrc = GEOMETRIC_PHASE_2D_HORIZONTAL(cap[a].x,
                    cap[a].y, cosEl, sinAz, cosAz, k);
            __sincosf(phaseSrc, &srcSig.y, &srcSig.x);

            float2 signalNew;
            signalNew.x = (signal.x * srcSig.x - signal.y * srcSig.y);
            signalNew.y = (signal.y * srcSig.x + signal.x * srcSig.y);

            // Perform complex multiply-accumulate.
            cpx.x += (signalNew.x * w.x - signalNew.y * w.y);
            cpx.y += (signalNew.y * w.x + signalNew.x * w.y);
        }

        // Must synchronise again before loading in a new block of antennas.
        __syncthreads();
    }

    // Copy result into global memory.
    if (s < ns)
        image[s] = cpx;
}
