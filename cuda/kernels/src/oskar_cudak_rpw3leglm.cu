#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/kernels/oskar_cudak_rpw3leglm.h"
#include "math/core/phase.h"

// Single precision.

// Shared memory pointer used by the kernel.
extern __shared__ float smem[];

__global__
void oskar_cudakf_rpw3leglm(const int na, const float* uvw, const int ns,
        const float* l, const float* m, const float* n, const float k,
        float2* weights)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int s = blockDim.x * blockIdx.x + tx; // Source index.
    const int a = blockDim.y * blockIdx.y + ty; // Antenna index.

    // Cache source and antenna data from global memory.
    float* cl = smem;
    float* cm = &cl[blockDim.x];
    float* cn = &cm[blockDim.x];
    float* cu = &cn[blockDim.x];
    float* cv = &cu[blockDim.y];
    float* cw = &cv[blockDim.y];
    if (s < ns && ty == 0) {
        cl[tx] = l[s];
        cm[tx] = m[s];
        cn[tx] = n[s];
    }
    if (a < na && tx == 0) {
        cu[ty] = uvw[a];
        cv[ty] = uvw[a + na];
        cw[ty] = uvw[a + 2*na];
    }
    __syncthreads();

    float arg = k * (cu[ty] * cl[tx] + cv[ty] * cm[tx] + cw[ty] * cn[tx]);
    float2 weight;
    sincosf(arg, &weight.y, &weight.x);

    // Write result to global memory.
    if (s < ns && a < na) {
        const int w = s + ns * a;
        weights[w] = weight;
    }
}

// Double precision.

// Shared memory pointer used by the kernel.
extern __shared__ double smemd[];

__global__
void oskar_cudakd_rpw3leglm(const int na, const double* uvw, const int ns,
        const double* l, const double* m, const double* n, const double k,
        double2* weights)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int s = blockDim.x * blockIdx.x + tx; // Source index.
    const int a = blockDim.y * blockIdx.y + ty; // Antenna index.

    // Cache source and antenna data from global memory.
    double* cl = smemd;
    double* cm = &cl[blockDim.x];
    double* cn = &cm[blockDim.x];
    double* cu = &cn[blockDim.x];
    double* cv = &cu[blockDim.y];
    double* cw = &cv[blockDim.y];
    if (s < ns && ty == 0) {
        cl[tx] = l[s];
        cm[tx] = m[s];
        cn[tx] = n[s];
    }
    if (a < na && tx == 0) {
        cu[ty] = uvw[a];
        cv[ty] = uvw[a + na];
        cw[ty] = uvw[a + 2*na];
    }
    __syncthreads();

    double arg = k * (cu[ty] * cl[tx] + cv[ty] * cm[tx] + cw[ty] * cn[tx]);
    double2 weight;
    sincos(arg, &weight.y, &weight.x);

    // Write result to global memory.
    if (s < ns && a < na) {
        const int w = s + ns * a;
        weights[w] = weight;
    }
}
