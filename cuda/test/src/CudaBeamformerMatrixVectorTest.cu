#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/test/CudaBeamformerMatrixVectorTest.h"
#include "cuda/oskar_cuda_bf2hig.h"
#include "math/core/SphericalPositions.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>
#include <hipblas.h>

#ifndef M_PI
#define M_PI 3.1415926535
#endif

#define DEG2RAD (M_PI / 180.0)
#define RAD2DEG (180.0 / M_PI)
#define C_0 299792458.0

#define TIMER_ENABLE 1
#include "utility/timer.h"

// Register the test class.
CPPUNIT_TEST_SUITE_REGISTRATION(CudaBeamformerMatrixVectorTest);

/**
 * @details
 * Sets up the context before running each test method.
 */
void CudaBeamformerMatrixVectorTest::setUp()
{
}

/**
 * @details
 * Clean up routine called after each test is run.
 */
void CudaBeamformerMatrixVectorTest::tearDown()
{
}

/**
 * @details
 * Tests antenna signal generation using CUDA.
 */
void CudaBeamformerMatrixVectorTest::test_basicMatrixVector()
{
    unsigned na = 3;
    unsigned nb = 2;

    // Allocate memory for signals, weights and beams.
    float* signals = (float*)calloc(na * 2, sizeof(float));
    float* beams   = (float*)calloc(nb * 2, sizeof(float));
    float* weights = (float*)calloc(na * nb * 2, sizeof(float));

    // Fill signal and weights arrays.
    for (unsigned i = 0; i < na * 2; i += 2) signals[i] = i + 1;
    for (unsigned i = 0; i < na * nb * 2; i += 2) weights[i] = i + 2;

    // Perform matrix-matrix multiply.
    // Initialise cuBLAS.
    cublasInit();

    // Allocate memory for antenna signals and beamforming weights
    // on the device.
    float2 *signalsd, *weightsd, *beamsd;
    hipMalloc((void**)&signalsd, na * sizeof(float2));
    hipMalloc((void**)&beamsd, nb * sizeof(float2));
    hipMalloc((void**)&weightsd, na * nb * sizeof(float2));

    // Copy antenna signals and beamforming weights to the device.
    hipMemcpy(signalsd, signals, na * sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(weightsd, weights, na * nb * sizeof(float2), hipMemcpyHostToDevice);

    // Call cuBLAS function to perform the matrix-vector multiplication.
    // Note that cuBLAS calls use Fortran-ordering (column major) for their
    // matrices, so we use the transpose here.
    hipblasCgemv('t', na, nb, make_float2(1.0, 0.0),
            weightsd, na, signalsd, 1, make_float2(0.0, 0.0), beamsd, 1);

    // Copy result from device memory to host memory.
    hipMemcpy(beams, beamsd, nb * sizeof(float2), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(signalsd);
    hipFree(weightsd);
    hipFree(beamsd);

    // Shut down cuBLAS.
    cublasShutdown();

    CPPUNIT_ASSERT_DOUBLES_EQUAL(44.0, beams[0], 1e-5);
    CPPUNIT_ASSERT_DOUBLES_EQUAL(0.0,  beams[1], 1e-5);
    CPPUNIT_ASSERT_DOUBLES_EQUAL(98.0, beams[2], 1e-5);
    CPPUNIT_ASSERT_DOUBLES_EQUAL(0.0,  beams[3], 1e-5);

    // Free host memory.
    free(signals);
    free(beams);
    free(weights);
}

/**
 * @details
 * Tests antenna signal generation using CUDA.
 */
void CudaBeamformerMatrixVectorTest::test_method()
{
    // Generate square array of antenna positions.
    const int na = 100;
    const float sep = 0.15; // Antenna separation, metres.
    const float halfArraySize = (na - 1) * sep / 2.0;
    std::vector<float> ax(na * na), ay(na * na); // Antenna (x,y) positions.
    for (int x = 0; x < na; ++x) {
        for (int y = 0; y < na; ++y) {
            int i = y + x * na;
            ax[i] = x * sep - halfArraySize;
            ay[i] = y * sep - halfArraySize;
        }
    }

    // Generate some source positions.
    float centreAz = 0;  // Beam azimuth.
    float centreEl = 50; // Beam elevation.
    SphericalPositions<float> posSrc (
            centreAz * DEG2RAD, centreEl * DEG2RAD, // Centre.
            20 * DEG2RAD, 20 * DEG2RAD, // Half-widths.
            10 * DEG2RAD, 10 * DEG2RAD); // Spacings.
    unsigned ns = posSrc.generate(0, 0); // No. of sources.
    std::vector<float> slon(ns), slat(ns);
    posSrc.generate(&slon[0], &slat[0]);

    // Generate source amplitudes.
    std::vector<float> samp(ns, 1.0);

    // Generate some beam positions.
    SphericalPositions<float> posBeam (
            centreAz * DEG2RAD, centreEl * DEG2RAD, // Centre.
            30 * DEG2RAD, 30 * DEG2RAD, // Half-widths.
            0.2 * DEG2RAD, 0.2 * DEG2RAD); // Spacings.
    unsigned nb = posBeam.generate(0, 0); // No. of beams.
    std::vector<float> blon(nb), blat(nb);
    posBeam.generate(&blon[0], &blat[0]);

    // Call CUDA beamformer.
    float freq = 1e9; // Observing frequency, Hertz.
    std::vector<float> beams(nb * 2); // Beam real & imaginary values.
    TIMER_START
    oskar_cudaf_bf2hig(na*na, &ax[0], &ay[0], ns, &samp[0],
            &slon[0], &slat[0], nb, &blon[0], &blat[0],
            2 * M_PI * (freq / C_0), &beams[0]);
    TIMER_STOP("Finished beamforming "
            "(%d antennas, %d sources, %d beams)", na*na, ns, nb);

    // Write beam data to file.
    FILE* file = fopen("beams.dat", "w");
    for (unsigned b = 0; b < nb; ++b) {
        fprintf(file, "%12.3f%12.3f%16.4e%16.4e\n",
                blon[b] * RAD2DEG, blat[b] * RAD2DEG, beams[2*b], beams[2*b+1]);
    }
    fclose(file);
}
