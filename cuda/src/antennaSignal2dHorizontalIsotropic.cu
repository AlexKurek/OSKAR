#include "hip/hip_runtime.h"
#include "cuda/antennaSignal2dHorizontalIsotropic.h"
#include "cuda/_antennaSignal2dHorizontalIsotropic.h"

/**
 * @details
 * Computes antenna signals using CUDA.
 *
 * The function must be supplied with the antenna x- and y-positions, the
 * source amplitudes, longitude and latitude positions, and the wavenumber.
 *
 * The computed antenna signals are returned in the \p signals array, which
 * must be pre-sized to length 2*na. The values in the \p signals array
 * are alternate (real, imag) pairs for each antenna.
 *
 * @param[in] na The number of antennas.
 * @param[in] ax The antenna x-positions in metres.
 * @param[in] ay The antenna y-positions in metres.
 * @param[in] ns The number of source positions.
 * @param[in] samp The source amplitudes.
 * @param[in] slon The source longitude coordinates in radians.
 * @param[in] slat The source latitude coordinates in radians.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] signals The computed antenna signals (see note, above).
 */
void antennaSignal2dHorizontalIsotropic(const int na, const float* ax,
        const float* ay, const int ns, const float* samp, const float* slon,
        const float* slat, const float k, float* signals)
{
    // Allocate memory for antenna positions, source positions
    // and antenna signals on the device.
    float *axd, *ayd, *slond, *slatd, *sampd;
    float2 *sig;
    hipMalloc((void**)&axd, na * sizeof(float));
    hipMalloc((void**)&ayd, na * sizeof(float));
    hipMalloc((void**)&sampd, ns * sizeof(float));
    hipMalloc((void**)&slond, ns * sizeof(float));
    hipMalloc((void**)&slatd, ns * sizeof(float));
    hipMalloc((void**)&sig, ns * sizeof(float2));

    // Copy antenna positions and test source positions to device.
    hipMemcpy(axd, ax, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(slond, slon, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(slatd, slat, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sampd, samp, ns * sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernel to compute antenna signals on the device.
    int threadsPerBlock = 384;
    int blocks = (ns + threadsPerBlock - 1) / threadsPerBlock;
//    _antennaSignal2dHorizontalIsotropic <<<blocks, threadsPerBlock>>> (na, axd, ayd,
//            ns, sampd, slond, slatd, k, sig);

    // Copy result from device memory to host memory.
    hipMemcpy(signals, sig, ns * sizeof(float2), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(sampd);
    hipFree(slond);
    hipFree(slatd);
    hipFree(sig);
}
