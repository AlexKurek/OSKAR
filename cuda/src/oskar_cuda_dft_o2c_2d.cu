#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/oskar_cuda_dft_o2c_2d.h"
#include "math/cudak/oskar_cudak_dftw_o2c_2d.h"
#include "math/cudak/oskar_cudak_dftw_2d_seq_in.h"

#ifdef __cplusplus
extern "C" {
#endif

// Single precision.

int oskar_cudaf_dft_o2c_2d(int n_in, const float* x_in, const float* y_in,
        int n_out, const float* x_out, const float* y_out, float x_centre,
        float y_centre, float* work, float* output)
{
    // Initialise.
    hipError_t errCuda;
    const int max_in_chunk = 896; // Should be multiple of 16.
    const int max_out_chunk = 65536; // Manageable output chunk size.
    float2* weights = (float2*)work;
    float2* out = (float2*)output;

    // Set up thread blocks.
    const dim3 wThd(256, 1); // Weights generator (input, output).
    const dim3 wBlk((n_in + wThd.x - 1) / wThd.x, 1);
    const size_t wSmem = sizeof(float2) * (wThd.x + 1);
    const int bThd = 256; // DFT kernel (output positions).
    size_t bSmem = 2 * max_in_chunk * sizeof(float2);

    // Invoke kernel to compute unnormalised DFT weights.
    oskar_cudak_dftw_2d_seq_in_f <<< wBlk, wThd, wSmem >>>
            (n_in, x_in, y_in, 1, (const float*)&x_centre,
                    (const float*)&y_centre, weights);
    hipDeviceSynchronize();
    errCuda = hipPeekAtLastError();
    if (errCuda != hipSuccess) return errCuda;

    // Loop over output chunks.
    for (int start = 0; start < n_out; start += max_out_chunk)
    {
        int chunk_size = n_out - start;
        if (chunk_size > max_out_chunk) chunk_size = max_out_chunk;

        // Invoke kernel to compute the (partial) DFT on the device.
        int bBlk = (chunk_size + bThd - 1) / bThd;
        oskar_cudak_dftw_o2c_2d_f <<< bBlk, bThd, bSmem >>>
                (n_in, x_in, y_in, weights, chunk_size, x_out + start,
                        y_out + start, max_in_chunk, out + start);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;
    }

    return 0;
}

// Double precision.

int oskar_cudad_dft_o2c_2d(int n_in, const double* x_in, const double* y_in,
        int n_out, const double* x_out, const double* y_out, double x_centre,
        double y_centre, double* work, double* output)
{
    // Initialise.
    hipError_t errCuda;
    const int max_in_chunk = 448; // Should be multiple of 16.
    const int max_out_chunk = 32768; // Manageable output chunk size.
    double2* weights = (double2*)work;
    double2* out = (double2*)output;

    // Set up thread blocks.
    const dim3 wThd(256, 1); // Weights generator (input, output).
    const dim3 wBlk((n_in + wThd.x - 1) / wThd.x, 1);
    const size_t wSmem = sizeof(double2) * (wThd.x + 1);
    const int bThd = 256; // DFT kernel (output positions).
    size_t bSmem = 2 * max_in_chunk * sizeof(double2);

    // Invoke kernel to compute unnormalised DFT weights.
    oskar_cudak_dftw_2d_seq_in_d <<< wBlk, wThd, wSmem >>>
            (n_in, x_in, y_in, 1, (const double*)&x_centre,
                    (const double*)&y_centre, weights);
    hipDeviceSynchronize();
    errCuda = hipPeekAtLastError();
    if (errCuda != hipSuccess) return errCuda;

    // Loop over output chunks.
    for (int start = 0; start < n_out; start += max_out_chunk)
    {
        int chunk_size = n_out - start;
        if (chunk_size > max_out_chunk) chunk_size = max_out_chunk;

        // Invoke kernel to compute the (partial) DFT on the device.
        int bBlk = (chunk_size + bThd - 1) / bThd;
        oskar_cudak_dftw_o2c_2d_d <<< bBlk, bThd, bSmem >>>
                (n_in, x_in, y_in, weights, chunk_size, x_out + start,
                        y_out + start, max_in_chunk, out + start);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) return errCuda;
    }

    return 0;
}

#ifdef __cplusplus
}
#endif
