#include "hip/hip_runtime.h"
#include "cuda/vectorAdd.h"
#include <cstdio>
#include <cstdlib>

// Functions prototypes
__global__ void VecAdd(const float* A, const float* B, float* C, int N);

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void vectorAddMethod()
{
    printf("Vector addition\n");
    int N = 50000;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Allocate vectors in device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    VecAdd <<<blocksPerGrid, threadsPerBlock>>> (d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");

    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
}





__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

