#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/vectorAdd.h"

#include "cuda/CudaEclipse.h"

#include <cstdio>
#include <cstdlib>

// Function prototypes.
__global__ void _vectorAdd(const float* A, const float* B, float* C, int N);

// Tests vector addition.
void vectorAdd()
{
    printf("Vector addition\n");
    int N = 50000;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory.
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Fill input vectors with random numbers.
    for (int i = 0; i < N; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate vectors in device memory.
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy input vectors from host memory to device memory.
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel.
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    _vectorAdd <<<blocksPerGrid, threadsPerBlock>>> (d_A, d_B, d_C, N);

    // Copy result from device memory to host memory.
    // h_C contains the result in host memory.
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result.
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");

    // Free device and host memory.
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);
    if (h_A) free(h_A);
    if (h_B) free(h_B);
    if (h_C) free(h_C);
}

// Vector addition kernel.
__global__ void _vectorAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

