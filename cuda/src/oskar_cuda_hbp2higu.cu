#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/oskar_cuda_hbp2higu.h"
#include "cuda/kernels/oskar_cudak_bp2hiw.h"
#include "cuda/kernels/oskar_cudak_bp2hiws.h"
#include "cuda/kernels/oskar_cudak_wt2hg.h"
#include <stdio.h>

#include "cuda/CudaEclipse.h"

#ifdef __cplusplus
extern "C" {
#endif

void oskar_cuda_hbp2higu(int n2, int* n1, const float* x1, const float* y1,
        const float* x2, const float* y2, int ns, const float* sa,
        const float* se, float ba1, float be1, float ba2, float be2,
        float k, float* image)
{
    // Find total number of antennas.
    int na = 0, maxTileSize = 0;
    for (int i = 0; i < n2; ++i) {
        na += n1[i];
        maxTileSize = (maxTileSize > n1[i]) ? maxTileSize : n1[i];
    }
    int maxWeights = (maxTileSize > n2) ? maxTileSize : n2;

    // Precompute.
    float3 trig1 = make_float3(cos(ba1), sin(ba1), cos(be1));
    float3 trig2 = make_float3(cos(ba2), sin(ba2), cos(be2));

    // Allocate memory for antenna positions, antenna weights,
    // test source positions and pixel values on the device.
    float *x1d, *y1d, *x2d, *y2d, *sad, *sed;
    float2 *weights, *imaged, *signalsd;
    float3 *trig1d, *trig2d;
    hipMalloc((void**)&x1d, na * sizeof(float));
    hipMalloc((void**)&y1d, na * sizeof(float));
    hipMalloc((void**)&x2d, n2 * sizeof(float));
    hipMalloc((void**)&y2d, n2 * sizeof(float));
    hipMalloc((void**)&weights, maxWeights * sizeof(float2));
    hipMalloc((void**)&trig1d, 1 * sizeof(float3));
    hipMalloc((void**)&trig2d, 1 * sizeof(float3));

    // Copy antenna positions and beam geometry to device.
    hipMemcpy(x1d, x1, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y1d, y1, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x2d, x2, n2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y2d, y2, n2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(trig1d, &trig1, 1 * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(trig2d, &trig2, 1 * sizeof(float3), hipMemcpyHostToDevice);

    // Divide up the source (pixel) list into manageable chunks.
    int nsMax = 100000;
    int chunk = 0, chunks = (ns + nsMax - 1) / nsMax;

    // Allocate memory for source position chunk on the device.
    hipMalloc((void**)&sad, nsMax * sizeof(float));
    hipMalloc((void**)&sed, nsMax * sizeof(float));
    hipMalloc((void**)&imaged, nsMax * sizeof(float2));
    hipMalloc((void**)&signalsd, n2 * nsMax * sizeof(float2));
    hipDeviceSynchronize();
    hipError_t err = hipPeekAtLastError();
    if (err != hipSuccess)
        printf("CUDA Error (malloc): %s\n", hipGetErrorString(err));

    // Loop over pixel chunks.
    for (chunk = 0; chunk < chunks; ++chunk) {

        const int srcStart = chunk * nsMax;
        int srcInBlock = ns - srcStart;
        if (srcInBlock > nsMax) srcInBlock = nsMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(sad, sa + srcStart, srcInBlock * sizeof(float),
                hipMemcpyHostToDevice);
        hipMemcpy(sed, se + srcStart, srcInBlock * sizeof(float),
                hipMemcpyHostToDevice);

        // Loop over tiles.
        int antCount = 0;
        for (int t = 0; t < n2; ++t) {

            // Get number of antennas in the tile.
            int nat = n1[t];
            float* x1td = x1d + antCount;
            float* y1td = y1d + antCount;
            float2* signalstd = signalsd + t * srcInBlock;

            // Invoke kernel to compute antenna weights on the device.
            dim3 wThreads(256, 1);
            dim3 wBlocks((nat + wThreads.x - 1) / wThreads.x, 1);
            size_t wSharedMem = wThreads.x * sizeof(float2) + sizeof(float3);
            oskar_cudak_wt2hg <<<wBlocks, wThreads, wSharedMem>>> (
                    nat, x1td, y1td, 1, trig1d, k, weights);
            hipDeviceSynchronize();
            hipError_t err = hipPeekAtLastError();
            if (err != hipSuccess)
                printf("CUDA Error (weights): %s\n", hipGetErrorString(err));

            // Invoke kernel to compute the (partial) beam pattern on the device.
            int threadsPerBlock = 256;
            int blocks = (srcInBlock + threadsPerBlock - 1) / threadsPerBlock;
            int maxAntennasPerBlock = 864; // Should be multiple of 16.
            size_t sharedMem = 2 * maxAntennasPerBlock * sizeof(float2);
            oskar_cudak_bp2hiw <<<blocks, threadsPerBlock, sharedMem>>>
                    (nat, x1td, y1td, weights, srcInBlock, sad, sed, k,
                            maxAntennasPerBlock, signalstd);
            hipDeviceSynchronize();
            err = hipPeekAtLastError();
            if (err != hipSuccess)
                printf("CUDA Error (bp): %s\n", hipGetErrorString(err));

            // Increment by number of antennas in tile.
            antCount += nat;
        }

        // Invoke kernel to compute tile weights on the device.
        dim3 wThreads(256, 1);
        dim3 wBlocks((n2 + wThreads.x - 1) / wThreads.x, 1);
        size_t wSharedMem = wThreads.x * sizeof(float2) + sizeof(float3);
        oskar_cudak_wt2hg <<<wBlocks, wThreads, wSharedMem>>> (
                n2, x2d, y2d, 1, trig2d, k, weights);
        hipDeviceSynchronize();

        // Beam pattern kernel that takes tile signals.
        // Invoke kernel to compute the (partial) beam pattern on the device.
        int threadsPerBlock = 256;
        int blocks = (srcInBlock + threadsPerBlock - 1) / threadsPerBlock;
        int maxAntennasPerBlock = 512; // Should be multiple of 16.
        size_t sharedMem = 3 * maxAntennasPerBlock * sizeof(float2);
        oskar_cudak_bp2hiws <<<blocks, threadsPerBlock, sharedMem>>>
                (n2, x2d, y2d, weights, signalsd, srcInBlock, srcInBlock, sad, sed, k,
                        maxAntennasPerBlock, imaged);
        hipDeviceSynchronize();
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess)
            printf("CUDA Error: %s\n", hipGetErrorString(err));

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + 2 * srcStart, imaged, srcInBlock * sizeof(float2),
                hipMemcpyDeviceToHost);
    }

    // Free device memory.
    hipFree(x1d);
    hipFree(y1d);
    hipFree(x2d);
    hipFree(y2d);
    hipFree(weights);
    hipFree(sad);
    hipFree(sed);
    hipFree(imaged);
    hipFree(trig1d);
    hipFree(trig2d);
    hipFree(signalsd);
}

#ifdef __cplusplus
}
#endif
