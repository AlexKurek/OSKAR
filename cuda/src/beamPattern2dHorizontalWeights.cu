#include "hip/hip_runtime.h"
#include "cuda/beamPattern2dHorizontalWeights.h"
#include "cuda/_beamPattern2dHorizontalWeights.h"
#include "cuda/_weights2dHorizontalGeometric.h"
#include <cstdio>

/**
 * @details
 * Computes a beam pattern using CUDA, generating the beamforming weights
 * separately.
 *
 * The function must be supplied with the antenna x- and y-positions, the
 * test source longitude and latitude positions, the beam direction, and
 * the wavenumber.
 *
 * The computed beam pattern is returned in the \p image array, which
 * must be pre-sized to length 2*ns. The values in the \p image array
 * are alternate (real, imag) pairs for each position of the test source.
 *
 * @param[in] na The number of antennas.
 * @param[in] ax The antenna x-positions in metres.
 * @param[in] ay The antenna y-positions in metres.
 * @param[in] ns The number of test source positions.
 * @param[in] slon The longitude coordinates of the test source.
 * @param[in] slat The latitude coordinates of the test source.
 * @param[in] ba The beam azimuth direction in radians
 * @param[in] be The beam elevation direction in radians.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] image The computed beam pattern (see note, above).
 */
void beamPattern2dHorizontalWeights(const int na, const float* ax,
        const float* ay, const int ns, const float* slon, const float* slat,
        const float ba, const float be, const float k, float* image)
{
    // Precompute.
    float sinBeamAz = sin(ba);
    float cosBeamAz = cos(ba);
    float cosBeamEl = cos(be);

    // Allocate memory for antenna positions, antenna weights,
    // test source positions and pixel values on the device.
    float *axd, *ayd, *slond, *slatd, *sbad, *cbad, *cbed;
    float2 *weights, *pix;
    hipMalloc((void**)&axd, na * sizeof(float));
    hipMalloc((void**)&ayd, na * sizeof(float));
    hipMalloc((void**)&weights, na * sizeof(float2));
    hipMalloc((void**)&sbad, 1 * sizeof(float));
    hipMalloc((void**)&cbad, 1 * sizeof(float));
    hipMalloc((void**)&cbed, 1 * sizeof(float));

    // Copy antenna positions and test source positions to device.
    hipMemcpy(axd, ax, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sbad, &sinBeamAz, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cbad, &cosBeamAz, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cbed, &cosBeamEl, 1 * sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernel to compute antenna weights on the device.
    int wThreadsPerBlock = 256;
    int wBlocks = (na + wThreadsPerBlock - 1) / wThreadsPerBlock;
    _weights2dHorizontalGeometric <<<wBlocks, wThreadsPerBlock>>> (
            na, axd, ayd, 1, cbed, cbad, sbad, k, weights);
    hipDeviceSynchronize();

    // Divide up the source (pixel) list into manageable chunks.
    int nsMax = 100000;
    int chunks = (ns + nsMax - 1) / nsMax;

    // Allocate memory for source position chunk on the device.
    hipMalloc((void**)&slond, nsMax * sizeof(float));
    hipMalloc((void**)&slatd, nsMax * sizeof(float));
    hipMalloc((void**)&pix, nsMax * sizeof(float2));

    // Loop over pixel chunks.
    for (int chunk = 0; chunk < chunks; ++chunk) {
        const int srcStart = chunk * nsMax;
        int srcInBlock = ns - srcStart;
        if (srcInBlock > nsMax) srcInBlock = nsMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(slond, slon + srcStart, srcInBlock * sizeof(float),
                hipMemcpyHostToDevice);
        hipMemcpy(slatd, slat + srcStart, srcInBlock * sizeof(float),
                hipMemcpyHostToDevice);

        // Invoke kernel to compute the (partial) beam pattern on the device.
        int threadsPerBlock = 256;
        int blocks = (srcInBlock + threadsPerBlock - 1) / threadsPerBlock;
        int maxAntennasPerBlock = 864; // Should be multiple of 16.
        size_t sharedMem = (threadsPerBlock + 2 * maxAntennasPerBlock)
                    * sizeof(float2);
        _beamPattern2dHorizontalWeights <<<blocks, threadsPerBlock, sharedMem>>>
                (na, axd, ayd, weights, srcInBlock, slond, slatd, k,
                        maxAntennasPerBlock, pix);
        hipDeviceSynchronize();
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess)
            printf("CUDA Error: %s\n", hipGetErrorString(err));

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + 2 * srcStart, pix, srcInBlock * sizeof(float2),
                hipMemcpyDeviceToHost);
    }

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(weights);
    hipFree(slond);
    hipFree(slatd);
    hipFree(pix);
    hipFree(sbad);
    hipFree(cbad);
    hipFree(cbed);
}
