#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/beamPattern2dHorizontalWeights.h"
#include "cuda/_beamPattern2dHorizontalWeights.h"
#include "cuda/_weights2dHorizontalGeometric.h"
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

/**
 * @details
 * Computes a beam pattern using CUDA, generating the beamforming weights
 * separately.
 *
 * The function must be supplied with the antenna x- and y-positions, the
 * test source longitude and latitude positions, the beam direction, and
 * the wavenumber.
 *
 * The computed beam pattern is returned in the \p image array, which
 * must be pre-sized to length 2*ns. The values in the \p image array
 * are alternate (real, imag) pairs for each position of the test source.
 *
 * @param[in] na The number of antennas.
 * @param[in] ax The antenna x-positions in metres.
 * @param[in] ay The antenna y-positions in metres.
 * @param[in] ns The number of test source positions.
 * @param[in] slon The longitude coordinates of the test source.
 * @param[in] slat The latitude coordinates of the test source.
 * @param[in] ba The beam azimuth direction in radians
 * @param[in] be The beam elevation direction in radians.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] image The computed beam pattern (see note, above).
 */
void beamPattern2dHorizontalWeights(const int na, const float* ax,
        const float* ay, const int ns, const float* slon, const float* slat,
        const float ba, const float be, const float k, float* image)
{
    // Precompute.
    float sinBeamAz = sin(ba);
    float cosBeamAz = cos(ba);
    float cosBeamEl = cos(be);

    // Allocate memory for antenna positions, antenna weights,
    // test source positions and pixel values on the device.
    float *axd, *ayd, *slond, *slatd, *sbad, *cbad, *cbed;
    float2 *weights, *pix;
    hipMalloc((void**)&axd, na * sizeof(float));
    hipMalloc((void**)&ayd, na * sizeof(float));
    hipMalloc((void**)&weights, na * sizeof(float2));
    hipMalloc((void**)&sbad, 1 * sizeof(float));
    hipMalloc((void**)&cbad, 1 * sizeof(float));
    hipMalloc((void**)&cbed, 1 * sizeof(float));

    // Copy antenna positions and test source positions to device.
    hipMemcpy(axd, ax, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sbad, &sinBeamAz, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cbad, &cosBeamAz, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cbed, &cosBeamEl, 1 * sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernel to compute antenna weights on the device.
    int wThreadsPerBlock = 256;
    int wBlocks = (na + wThreadsPerBlock - 1) / wThreadsPerBlock;
    _weights2dHorizontalGeometric <<<wBlocks, wThreadsPerBlock>>> (
            na, axd, ayd, 1, cbed, cbad, sbad, k, weights);
    hipDeviceSynchronize();

    // Divide up the source (pixel) list into manageable chunks.
    int nsMax = 100000;
    int chunk = 0, chunks = (ns + nsMax - 1) / nsMax;

    // Allocate memory for source position chunk on the device.
    hipMalloc((void**)&slond, nsMax * sizeof(float));
    hipMalloc((void**)&slatd, nsMax * sizeof(float));
    hipMalloc((void**)&pix, nsMax * sizeof(float2));

    // Loop over pixel chunks.
    for (chunk = 0; chunk < chunks; ++chunk) {
        const int srcStart = chunk * nsMax;
        int srcInBlock = ns - srcStart;
        if (srcInBlock > nsMax) srcInBlock = nsMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(slond, slon + srcStart, srcInBlock * sizeof(float),
                hipMemcpyHostToDevice);
        hipMemcpy(slatd, slat + srcStart, srcInBlock * sizeof(float),
                hipMemcpyHostToDevice);

        // Invoke kernel to compute the (partial) beam pattern on the device.
        int threadsPerBlock = 256;
        int blocks = (srcInBlock + threadsPerBlock - 1) / threadsPerBlock;
        int maxAntennasPerBlock = 864; // Should be multiple of 16.
        size_t sharedMem = (threadsPerBlock + 2 * maxAntennasPerBlock)
                    * sizeof(float2);
        _beamPattern2dHorizontalWeights <<<blocks, threadsPerBlock, sharedMem>>>
                (na, axd, ayd, weights, srcInBlock, slond, slatd, k,
                        maxAntennasPerBlock, pix);
        hipDeviceSynchronize();
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess)
            printf("CUDA Error: %s\n", hipGetErrorString(err));

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + 2 * srcStart, pix, srcInBlock * sizeof(float2),
                hipMemcpyDeviceToHost);
    }

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(weights);
    hipFree(slond);
    hipFree(slatd);
    hipFree(pix);
    hipFree(sbad);
    hipFree(cbad);
    hipFree(cbed);
}

#ifdef __cplusplus
}
#endif
