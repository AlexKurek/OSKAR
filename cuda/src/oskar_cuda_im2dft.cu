#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/oskar_cuda_im2dft.h"
#include "cuda/kernels/oskar_cudak_im2dft.h"
#include <stdio.h>
#include <math.h>

#include "cuda/CudaEclipse.h"


#ifdef __cplusplus
extern "C" {
#endif

#ifndef M_PI
#define M_PI 3.1415926535f
#endif

void oskar_cuda_im2dft(int nv, const float* u, const float* v,
        const float* vis, int nl, int nm, float dl, float dm,
        float sl, float sm, float* image)
{
    // Get the centre pixel in L and M.
    const int centreL = (int)floor(nl / 2.0f);
    const int centreM = (int)floor(nm / 2.0f);

    // Create and allocate memory for the pixel positions.
    const int np = nl * nm; // Number of pixels in image.
    float* pl = (float*)malloc(np * sizeof(float));
    float* pm = (float*)malloc(np * sizeof(float));
    int i, j, k; // Indices.
    float l, m; // Pixel coordinates.
    for (j = 0; j < nm; ++j) {
        // Image m-coordinate.
        m = 2.0f * (j - centreM) * dm * sm / M_PI;

        for (i = 0; i < nl; ++i) {
            // Image l-coordinate.
            l = 2.0f * (i - centreL) * dl * sl / M_PI;

            // Image pixel index.
            k = i + j * nl;
            pl[k] = l;
            pm[k] = m;
        }
    }

    // Allocate memory for visibilities and u,v-coordinates on the device.
    float *ud, *vd, *pld, *pmd, *pix;
    float2 *visd;
    hipMalloc((void**)&ud, nv * sizeof(float));
    hipMalloc((void**)&vd, nv * sizeof(float));
    hipMalloc((void**)&visd, nv * sizeof(float2));

    // Copy visibilities and u,v-coordinates to device.
    hipMemcpy(ud, u, nv * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vd, v, nv * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(visd, vis, nv * sizeof(float2), hipMemcpyHostToDevice);

    // Divide up the pixel list into manageable chunks.
    int npMax = 100000;
    int chunk = 0, chunks = (np + npMax - 1) / npMax;

    // Allocate memory for pixel position chunk on the device.
    hipMalloc((void**)&pld, npMax * sizeof(float));
    hipMalloc((void**)&pmd, npMax * sizeof(float));
    hipMalloc((void**)&pix, npMax * sizeof(float));

    // Loop over pixel chunks.
    for (chunk = 0; chunk < chunks; ++chunk) {
        const int pixStart = chunk * npMax;
        int pixInBlock = np - pixStart;
        if (pixInBlock > npMax) pixInBlock = npMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(pld, pl + pixStart, pixInBlock * sizeof(float),
                hipMemcpyHostToDevice);
        hipMemcpy(pmd, pm + pixStart, pixInBlock * sizeof(float),
                hipMemcpyHostToDevice);

        // Invoke kernel to compute the (partial) image on the device.
        int threadsPerBlock = 384;
        int blocks = (pixInBlock + threadsPerBlock - 1) / threadsPerBlock;
        int maxVisPerBlock = 896; // Should be multiple of 16.
        size_t sharedMem = 2 * maxVisPerBlock * sizeof(float2);
        oskar_cudak_im2dft <<<blocks, threadsPerBlock, sharedMem>>>
        		(nv, ud, vd, visd, pixInBlock, pld, pmd, maxVisPerBlock, pix);
        hipDeviceSynchronize();
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess)
            printf("CUDA Error: %s\n", hipGetErrorString(err));

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + pixStart, pix, pixInBlock * sizeof(float),
                hipMemcpyDeviceToHost);
    }

    // Free device memory.
    hipFree(ud);
    hipFree(vd);
    hipFree(visd);
    hipFree(pld);
    hipFree(pmd);
    hipFree(pix);

    // Free host memory.
    free(pm);
    free(pl);
}

#ifdef __cplusplus
}
#endif
