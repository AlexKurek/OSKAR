#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/oskar_cuda_bp2hc.h"
#include "cuda/kernels/oskar_cudak_antenna.h"
#include "cuda/kernels/oskar_cudak_apodisation.h"
#include "cuda/kernels/oskar_cudak_bp2hiw.h"
#include "cuda/kernels/oskar_cudak_wt2hg.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "cuda/CudaEclipse.h"

#ifdef __cplusplus
extern "C" {
#endif

// Single precision.

int oskar_cudaf_bp2hc(int na, const float* ax, const float* ay,
        int ns, const float* sa, const float* se, float ba, float be, float k,
        int apfn, int atype, float gain, float fwhm, float* image)
{
    // Initialise.
    hipError_t errCuda = hipSuccess;
    int i, a, csize, retVal = 0;
    const int nb = 1; // Number of beams is 1, since this is a beam pattern.
    const int naMax = 864; // Should be multiple of 16.
    const int nsMax = 100000; // Maximum number of sources per iteration.
    float r, rmax = 0.0f; // Station radius.

    // Set up thread blocks.
    const dim3 wThd(256, 1); // Weights generator (antennas, beams).
    const dim3 wBlk((na + wThd.x - 1) / wThd.x, 1);
    const size_t wSmem = wThd.x * sizeof(float2) + sizeof(float3);
    const int aThd = 256; // Apodisation function generator (antennas).
    const int aBlk = (na + aThd - 1) / aThd;
    const int bThd = 256; // Beam pattern generator (source positions).
    int bBlk = 0; // Number of thread blocks for beam pattern computed later.
    size_t bSmem = 2 * naMax * sizeof(float2);

    // Precompute beam geometry.
    const float3 trig = make_float3(cos(ba), sin(ba), cos(be));

    // Allocate memory for antenna positions, antenna weights,
    // test source position chunk and pixel value chunk on the device.
    float *axd, *ayd, *sad, *sed;
    float2 *wts, *imaged;
    float3 *trigd;
    hipMalloc((void**)&axd, na * sizeof(float));
    hipMalloc((void**)&ayd, na * sizeof(float));
    hipMalloc((void**)&wts, na * sizeof(float2));
    hipMalloc((void**)&trigd, nb * sizeof(float3));
    hipMalloc((void**)&sad, nsMax * sizeof(float));
    hipMalloc((void**)&sed, nsMax * sizeof(float));
    hipMalloc((void**)&imaged, nsMax * sizeof(float2));

    // Copy antenna positions and beam geometry to device.
    hipMemcpy(axd, ax, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(trigd, &trig, nb * sizeof(float3), hipMemcpyHostToDevice);

    // Invoke kernel to compute unnormalised, geometric antenna weights.
    oskar_cudakf_wt2hg <<< wBlk, wThd, wSmem >>>
            (na, axd, ayd, nb, trigd, k, wts);
    hipDeviceSynchronize();
    errCuda = hipPeekAtLastError();
    if (errCuda != hipSuccess) goto stop;

    // Weights apodisation.
    if (apfn != apfn_none)
    {
        // Determine station radius.
        for (a = 0; a < na; ++a)
        {
            r = sqrtf(ax[a] * ax[a] + ay[a] * ay[a]);
            if (r > rmax) rmax = r;
        }

        // Switch on selected apodisation function.
        switch (apfn)
        {
        case apfn_hann:
            oskar_cudakf_apodisation_hann <<< aBlk, aThd >>> (
                    na, axd, ayd, nb, rmax, wts);
            break;
        default:
            printf("ERROR: Unknown apodisation function.\n");
            retVal = EXIT_FAILURE;
            goto stop;
        }
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) goto stop;
    }

    // Iterate over pixel chunks.
    for (i = 0; i < ns; i += nsMax)
    {
        csize = ns - i; // Chunk size.
        if (csize > nsMax) csize = nsMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(sad, sa + i, csize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(sed, se + i, csize * sizeof(float), hipMemcpyHostToDevice);

        // Invoke kernel to compute the (partial) beam pattern on the device.
        bBlk = (csize + bThd - 1) / bThd;
        oskar_cudakf_bp2hiw <<< bBlk, bThd, bSmem >>>
                (na, axd, ayd, wts, csize, sad, sed, k, naMax, imaged);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) goto stop;

        // Antenna type.
        if (atype != atype_isotropic)
        {
            // Switch on selected antenna type.
            switch (atype)
            {
            case atype_sin_elevation:
                oskar_cudakf_antenna_sine <<< bBlk, bThd >>> (
                        csize, sed, imaged);
                break;
            case atype_sin2_elevation:
                oskar_cudakf_antenna_sine_squared <<< bBlk, bThd >>> (
                        csize, sed, imaged);
                break;
            case atype_gaussian:
                oskar_cudakf_antenna_gaussian <<< bBlk, bThd >>> (
                        csize, sed, gain, fwhm, imaged);
                break;
            default:
                printf("ERROR: Unknown antenna type.\n");
                retVal = EXIT_FAILURE;
                goto stop;
            }
            hipDeviceSynchronize();
            errCuda = hipPeekAtLastError();
            if (errCuda != hipSuccess) goto stop;
        }

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + 2 * i, imaged, csize * sizeof(float2),
                hipMemcpyDeviceToHost);
    }

    // Clean up before exit.
    stop:
    if (errCuda != hipSuccess)
    {
        retVal = errCuda;
        printf("CUDA Error: %s\n", hipGetErrorString(errCuda));
    }

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(wts);
    hipFree(sad);
    hipFree(sed);
    hipFree(imaged);
    hipFree(trigd);

    return retVal;
}

// Double precision.

int oskar_cudad_bp2hc(int na, const double* ax, const double* ay,
        int ns, const double* sa, const double* se, double ba, double be,
        double k, int apfn, int atype, double gain, double fwhm, double* image)
{
    // Initialise.
    hipError_t errCuda = hipSuccess;
    int i, a, csize, retVal = 0;
    const int nb = 1; // Number of beams is 1, since this is a beam pattern.
    const int naMax = 432; // Should be multiple of 16.
    const int nsMax = 100000; // Maximum number of sources per iteration.
    double r, rmax = 0.0; // Station radius.

    // Set up thread blocks.
    const dim3 wThd(256, 1); // Weights generator (antennas, beams).
    const dim3 wBlk((na + wThd.x - 1) / wThd.x, 1);
    const size_t wSmem = wThd.x * sizeof(double2) + sizeof(double3);
    const int aThd = 256; // Apodisation function generator (antennas).
    const int aBlk = (na + aThd - 1) / aThd;
    const int bThd = 256; // Beam pattern generator (source positions).
    int bBlk = 0; // Number of thread blocks for beam pattern computed later.
    size_t bSmem = 2 * naMax * sizeof(double2);

    // Precompute beam geometry.
    const double3 trig = make_double3(cos(ba), sin(ba), cos(be));

    // Allocate memory for antenna positions, antenna weights,
    // test source position chunk and pixel value chunk on the device.
    double *axd, *ayd, *sad, *sed;
    double2 *wts, *imaged;
    double3 *trigd;
    hipMalloc((void**)&axd, na * sizeof(double));
    hipMalloc((void**)&ayd, na * sizeof(double));
    hipMalloc((void**)&wts, na * sizeof(double2));
    hipMalloc((void**)&trigd, nb * sizeof(double3));
    hipMalloc((void**)&sad, nsMax * sizeof(double));
    hipMalloc((void**)&sed, nsMax * sizeof(double));
    hipMalloc((void**)&imaged, nsMax * sizeof(double2));

    // Copy antenna positions and beam geometry to device.
    hipMemcpy(axd, ax, na * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(trigd, &trig, nb * sizeof(double3), hipMemcpyHostToDevice);

    // Invoke kernel to compute unnormalised, geometric antenna weights.
    oskar_cudakd_wt2hg <<< wBlk, wThd, wSmem >>>
            (na, axd, ayd, nb, trigd, k, wts);
    hipDeviceSynchronize();
    errCuda = hipPeekAtLastError();
    if (errCuda != hipSuccess) goto stop;

    // Weights apodisation.
    if (apfn != apfn_none)
    {
        // Determine station radius.
        for (a = 0; a < na; ++a)
        {
            r = sqrt(ax[a] * ax[a] + ay[a] * ay[a]);
            if (r > rmax) rmax = r;
        }

        // Switch on selected apodisation function.
        switch (apfn)
        {
        case apfn_hann:
            oskar_cudakd_apodisation_hann <<< aBlk, aThd >>> (
                    na, axd, ayd, nb, rmax, wts);
            break;
        default:
            printf("ERROR: Unknown apodisation function.\n");
            retVal = EXIT_FAILURE;
            goto stop;
        }
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) goto stop;
    }

    // Iterate over pixel chunks.
    for (i = 0; i < ns; i += nsMax)
    {
        csize = ns - i; // Chunk size.
        if (csize > nsMax) csize = nsMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(sad, sa + i, csize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(sed, se + i, csize * sizeof(double), hipMemcpyHostToDevice);

        // Invoke kernel to compute the (partial) beam pattern on the device.
        bBlk = (csize + bThd - 1) / bThd;
        oskar_cudakd_bp2hiw <<< bBlk, bThd, bSmem >>>
                (na, axd, ayd, wts, csize, sad, sed, k, naMax, imaged);
        hipDeviceSynchronize();
        errCuda = hipPeekAtLastError();
        if (errCuda != hipSuccess) goto stop;

        // Antenna type.
        if (atype != atype_isotropic)
        {
            // Switch on selected antenna type.
            switch (atype)
            {
            case atype_sin_elevation:
                oskar_cudakd_antenna_sine <<< bBlk, bThd >>> (
                        csize, sed, imaged);
                break;
            case atype_sin2_elevation:
                oskar_cudakd_antenna_sine_squared <<< bBlk, bThd >>> (
                        csize, sed, imaged);
                break;
            case atype_gaussian:
                oskar_cudakd_antenna_gaussian <<< bBlk, bThd >>> (
                        csize, sed, gain, fwhm, imaged);
                break;
            default:
                printf("ERROR: Unknown antenna type.\n");
                retVal = EXIT_FAILURE;
                goto stop;
            }
            hipDeviceSynchronize();
            errCuda = hipPeekAtLastError();
            if (errCuda != hipSuccess) goto stop;
        }

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + 2 * i, imaged, csize * sizeof(double2),
                hipMemcpyDeviceToHost);
    }

    // Clean up before exit.
    stop:
    if (errCuda != hipSuccess)
    {
        retVal = errCuda;
        printf("CUDA Error: %s\n", hipGetErrorString(errCuda));
    }

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(wts);
    hipFree(sad);
    hipFree(sed);
    hipFree(imaged);
    hipFree(trigd);

    return retVal;
}

#ifdef __cplusplus
}
#endif
