#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/oskar_cudad_im2dft.h"
#include "cuda/kernels/oskar_cudakd_im2dft.h"
#include <stdio.h>

#include "cuda/CudaEclipse.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifndef M_PI
#define M_PI 3.1415926535
#endif

void oskar_cudad_im2dft(int nv, const double* u, const double* v,
        const double* vis, int nl, int nm, double dl, double dm,
        double sl, double sm, double* image)
{
    // Get the centre pixel in L and M.
    const int centreL = floor(nl / 2.0f);
    const int centreM = floor(nm / 2.0f);

    // Create and allocate memory for the pixel positions.
    const int np = nl * nm; // Number of pixels in image.
    double* pl = (double*)malloc(np * sizeof(double));
    double* pm = (double*)malloc(np * sizeof(double));
    int i, j, k; // Indices.
    double l, m; // Pixel coordinates.
    for (j = 0; j < nm; ++j) {
        // Image m-coordinate.
        m = 2.0 * (j - centreM) * dm * sm / M_PI;

        for (i = 0; i < nl; ++i) {
            // Image l-coordinate.
            l = 2.0 * (i - centreL) * dl * sl / M_PI;

            // Image pixel index.
            k = i + j * nl;
            pl[k] = l;
            pm[k] = m;
        }
    }

    // Allocate memory for visibilities and u,v-coordinates on the device.
    double *ud, *vd, *pld, *pmd, *pix;
    double2 *visd;
    hipMalloc((void**)&ud, nv * sizeof(double));
    hipMalloc((void**)&vd, nv * sizeof(double));
    hipMalloc((void**)&visd, nv * sizeof(double2));

    // Copy visibilities and u,v-coordinates to device.
    hipMemcpy(ud, u, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(vd, v, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(visd, vis, nv * sizeof(double2), hipMemcpyHostToDevice);

    // Divide up the pixel list into manageable chunks.
    int npMax = 100000;
    int chunk = 0, chunks = (np + npMax - 1) / npMax;

    // Allocate memory for pixel position chunk on the device.
    hipMalloc((void**)&pld, npMax * sizeof(double));
    hipMalloc((void**)&pmd, npMax * sizeof(double));
    hipMalloc((void**)&pix, npMax * sizeof(double));

    // Loop over pixel chunks.
    for (chunk = 0; chunk < chunks; ++chunk) {
        const int pixStart = chunk * npMax;
        int pixInBlock = np - pixStart;
        if (pixInBlock > npMax) pixInBlock = npMax;

        // Copy test source positions for this chunk to the device.
        hipMemcpy(pld, pl + pixStart, pixInBlock * sizeof(double),
                hipMemcpyHostToDevice);
        hipMemcpy(pmd, pm + pixStart, pixInBlock * sizeof(double),
                hipMemcpyHostToDevice);

        // Invoke kernel to compute the (partial) image on the device.
        int threadsPerBlock = 384;
        int blocks = (pixInBlock + threadsPerBlock - 1) / threadsPerBlock;
        int maxVisPerBlock = 896; // Should be multiple of 16.
        size_t sharedMem = 2 * maxVisPerBlock * sizeof(double2);
        oskar_cudakd_im2dft <<<blocks, threadsPerBlock, sharedMem>>>
                (nv, ud, vd, visd, pixInBlock, pld, pmd, maxVisPerBlock, pix);
        hipDeviceSynchronize();
        hipError_t err = hipPeekAtLastError();
        if (err != hipSuccess)
            printf("CUDA Error: %s\n", hipGetErrorString(err));

        // Copy (partial) result from device memory to host memory.
        hipMemcpy(image + pixStart, pix, pixInBlock * sizeof(double),
                hipMemcpyDeviceToHost);
    }

    // Free device memory.
    hipFree(ud);
    hipFree(vd);
    hipFree(visd);
    hipFree(pld);
    hipFree(pmd);
    hipFree(pix);

    // Free host memory.
    free(pm);
    free(pl);
}

#ifdef __cplusplus
}
#endif
