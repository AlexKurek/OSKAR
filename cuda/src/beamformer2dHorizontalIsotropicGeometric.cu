#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda/beamformer2dHorizontalIsotropicGeometric.h"
#include "cuda/_precompute2dHorizontalTrig.h"
#include "cuda/_antennaSignal2dHorizontalIsotropic.h"
#include "cuda/_weights2dHorizontalGeometric.h"

#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>

#include "cuda/CudaEclipse.h"
#include "cuda/CudaTypes.h"

#define TIMER_ENABLE 1
#include "utility/timer.h"

#ifdef __cplusplus
extern "C" {
#endif

/**
 * @details
 * Computes beams using CUDA.
 *
 * The computed beams are returned in the \p beams array, which
 * must be pre-sized to length 2*nb. The values in the \p beams array
 * are alternate (real, imag) pairs for each beam.
 *
 * @param[in] na The number of antennas.
 * @param[in] ax The antenna x-positions in metres.
 * @param[in] ay The antenna y-positions in metres.
 * @param[in] ns The number of sources.
 * @param[in] samp The source amplitudes.
 * @param[in] slon The source longitude coordinates in radians.
 * @param[in] slat The source latitude coordinates in radians.
 * @param[in] nb The number of beams to form.
 * @param[in] blon The source longitude coordinates in radians.
 * @param[in] blat The source latitude coordinates in radians.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] beams The complex vector of output beams (length nb).
 */
void beamformer2dHorizontalIsotropicGeometric(const int na,
        const float* ax, const float* ay, const int ns, const float* samp,
        const float* slon, const float* slat, const int nb,
        const float* blon, const float* blat, const float k, float* beams)
{
    // Initialise cuBLAS.
    cublasInit();

    // Create source and beam position pairs in host memory.
    float2* spos = (float2*)calloc(ns, sizeof(float2));
    float2* bpos = (float2*)calloc(nb, sizeof(float2));
    int i;
    for (i = 0; i < ns; ++i)
        spos[i] = make_float2(slon[i], slat[i]);
    for (i = 0; i < nb; ++i)
        bpos[i] = make_float2(blon[i], blat[i]);

    // Allocate memory for antenna positions, source positions,
    // beam positions,
    // antenna signals, beamforming weights on the device.
    float *axd, *ayd, *sampd;
    float2 *sposd, *bposd, *signalsd, *weightsd, *beamsd;
    float3 *strigd, *btrigd;
    hipMalloc((void**)&axd, na * sizeof(float));
    hipMalloc((void**)&ayd, na * sizeof(float));
    hipMalloc((void**)&sampd, ns * sizeof(float));
    hipMalloc((void**)&sposd, ns * sizeof(float2));
    hipMalloc((void**)&bposd, nb * sizeof(float2));
    hipMalloc((void**)&strigd, ns * sizeof(float3));
    hipMalloc((void**)&signalsd, na * sizeof(float2));

    // Copy antenna positions, source positions and beam positions to device.
    hipMemcpy(axd, ax, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sampd, samp, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sposd, &spos[0], ns * sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(bposd, &bpos[0], nb * sizeof(float2), hipMemcpyHostToDevice);

    // Set the maximum number of beams the device can compute at once.
    const int maxBeams = 1000;

    // Allocate enough memory for the beams and weights blocks.
    hipMalloc((void**)&weightsd, na * maxBeams * sizeof(float2));
    hipMalloc((void**)&btrigd, maxBeams * sizeof(float3));
    hipMalloc((void**)&beamsd, maxBeams * sizeof(float2));

    // Set threads per block.
    int threadsPerBlock = 384;

    // Invoke kernel to precompute source positions on the device.
    int sBlocks = (ns + threadsPerBlock - 1) / threadsPerBlock;
    _precompute2dHorizontalTrig <<<sBlocks, threadsPerBlock>>>
            (ns, sposd, strigd);

    // Invoke kernel to compute antenna signals on the device.
    int aBlocks = (na + threadsPerBlock - 1) / threadsPerBlock;
    int maxSourcesPerBlock = 384;
    size_t aSharedMem = threadsPerBlock * sizeof(float2)
            + maxSourcesPerBlock * sizeof(float4);
    _antennaSignal2dHorizontalIsotropic <<<aBlocks,
            threadsPerBlock, aSharedMem>>>
            (na, axd, ayd, ns, sampd, strigd, k, maxSourcesPerBlock, signalsd);

    // Start beamforming loop.
    // There may not be enough memory to allocate a weights matrix big enough,
    // so we divide it up and only compute (at most) maxBeams at once.
    int block = 0, blocks = (nb + maxBeams - 1) / maxBeams;
    for (block = 0; block < blocks; ++block) {
        const int beamStart = block * maxBeams;
        int beamsInBlock = nb - beamStart;
        if (beamsInBlock > maxBeams) {
            beamsInBlock = maxBeams;
        }

        // Invoke kernel to precompute the beam positions on the device.
        int bBlocks = (beamsInBlock + threadsPerBlock - 1) / threadsPerBlock;
        _precompute2dHorizontalTrig <<<bBlocks, threadsPerBlock>>>
                (beamsInBlock, &bposd[beamStart], btrigd);

        // Invoke kernel to compute beamforming weights on the device.
        int wBlocks = (na*beamsInBlock + threadsPerBlock - 1) / threadsPerBlock;
        TIMER_START
        _weights2dHorizontalGeometric <<<wBlocks, threadsPerBlock>>> (
                na, axd, ayd, beamsInBlock, btrigd, k, weightsd);
        hipDeviceSynchronize();
        TIMER_STOP("Finished weights")

        // Call cuBLAS function to perform the matrix-vector multiplication.
        // Note that cuBLAS calls use Fortran-ordering (column major) for their
        // matrices, so we use the transpose here.
        TIMER_START
        hipblasCgemv('t', na, beamsInBlock, make_float2(1.0, 0.0),
                weightsd, na, signalsd, 1, make_float2(0.0, 0.0), beamsd, 1);
        hipDeviceSynchronize();
        TIMER_STOP("Finished matrix-vector")

        // Copy result from device memory to host memory.
        hipMemcpy(&beams[2*beamStart], beamsd, beamsInBlock * sizeof(float2),
                hipMemcpyDeviceToHost);
    }

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(sampd);
    hipFree(sposd);
    hipFree(strigd);
    hipFree(btrigd);
    hipFree(signalsd);
    hipFree(weightsd);
    hipFree(beamsd);

    // Free host memory.
    free(spos);
    free(bpos);

    // Shut down cuBLAS.
    cublasShutdown();
}

#ifdef __cplusplus
}
#endif
