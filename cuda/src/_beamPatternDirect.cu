#include "hip/hip_runtime.h"
#include "cuda/_beamPatternDirect.h"
#include "math/core/phase.h"

/**
 * @details
 * This CUDA kernel evaluates the beam pattern for the given antenna
 * positions and beam direction, using the supplied positions of the test
 * source.
 *
 * Each thread evaluates a single pixel of the beam pattern, looping over
 * all the antennas while performing a complex multiply-accumulate with the
 * required beamforming weights, which are computed by the kernel for the
 * given beam direction. In almost all cases, this is faster than looking up
 * the weights in global memory.
 *
 * The computed beam pattern is returned in the \p image array, which
 * must be pre-sized to length 2*ns. The values in the \p image array
 * are alternate (real, imag) pairs for each test source position.
 *
 * @param[in] na Number of antennas.
 * @param[in] ax Array of antenna x positions.
 * @param[in] ay Array of antenna y positions.
 * @param[in] weights Array of complex antenna weights (length na).
 * @param[in] ns The number of test source positions.
 * @param[in] slon The longitude coordinates of the test source.
 * @param[in] slat The latitude coordinates of the test source.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] image The computed beam pattern (see note, above).
 */
__global__
void _beamPatternDirect(const int na, const float* ax, const float* ay,
        const float cosBeamEl, const float cosBeamAz,  const float sinBeamAz,
        const int ns, const float* slon, const float* slat,
        const float k, float2* image)
{
    // Get the pixel (source position) ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;
    if (s >= ns) return; // Return if the index is out of range.

    // Get the source position.
    const float az = slon[s];
    const float el = slat[s];
    const float cosEl = cosf(el);
    const float sinAz = sinf(az);
    const float cosAz = cosf(az);

    // Initialise shared memory to hold complex pixel amplitude.
    sharedMem[threadIdx.x] = make_float2(0.0, 0.0);
    float2 w, signal;
    float phaseBeam, phaseSrc, x, y;

    // Loop over all antennas.
    for (int a = 0; a < na; ++a) {
        // Get the antenna position from global memory.
        x = ax[a]; y = ay[a];

        // Calculate the geometric phase of the beam direction.
        // (Faster to recompute it here than look it up from global memory.)
        phaseBeam = -GEOMETRIC_PHASE(x, y, cosBeamEl, sinBeamAz, cosBeamAz, k);
        sincosf(phaseBeam, &w.x, &w.y);

        // Calculate the geometric phase from the source.
        phaseSrc = GEOMETRIC_PHASE(x, y, cosEl, sinAz, cosAz, k);
        sincosf(phaseSrc, &signal.x, &signal.y);

        // Perform complex multiply-accumulate.
        sharedMem[threadIdx.x].x += (signal.x * w.x - signal.y * w.y); // RE*RE - IM*IM
        sharedMem[threadIdx.x].y += (signal.y * w.x + signal.x * w.y); // IM*RE + RE*IM
    }

    // Copy shared memory back into global memory.
    image[s].x = sharedMem[threadIdx.x].x / na;
    image[s].y = sharedMem[threadIdx.x].y / na;
}
