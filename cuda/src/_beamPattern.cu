#include "hip/hip_runtime.h"
#include "cuda/_beamPattern.h"
#include "math/core/phase.h"

/**
 * @details
 * This CUDA kernel evaluates the beam pattern for the given antenna
 * positions and weights vector, using the supplied positions of the test
 * source.
 *
 * Each thread evaluates a single pixel of the beam pattern, looping over
 * all the antennas while performing a complex multiply-accumulate with the
 * required beamforming weights.
 *
 * The computed beam pattern is returned in the \p image array, which
 * must be pre-sized to length 2*ns. The values in the \p image array
 * are alternate (real, imag) pairs for each test source position.
 *
 * The number of floating-point operations performed by this kernel is:
 * \li Sines and cosines: ns * (2 * na + 3).
 * \li Multiplies: 8 * ns * na.
 * \li Additions / subtractions: 5 * ns * na.
 *
 * @param[in] na Number of antennas.
 * @param[in] ax Array of antenna x positions.
 * @param[in] ay Array of antenna y positions.
 * @param[in] weights Array of complex antenna weights (length na).
 * @param[in] ns The number of test source positions.
 * @param[in] slon The longitude coordinates of the test source.
 * @param[in] slat The latitude coordinates of the test source.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] image The computed beam pattern (see note, above).
 */
__global__
void _beamPattern(const int na, const float* ax, const float* ay,
        const float2* weights, const int ns, const float* slon, const float* slat,
        const float k, float2* image)
{
    // Get the pixel (source position) ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;
    if (s >= ns) return; // Return if the index is out of range.

    // Get the source position.
    const float az = slon[s];
    const float el = slat[s];
    const float cosEl = cosf(el);
    const float sinAz = sinf(az);
    const float cosAz = cosf(az);

    // Loop over all antennas.
    image[s] = make_float2(0.0, 0.0);
    for (int a = 0; a < na; ++a) {
        // Calculate the geometric phase from the source.
        const float phase = GEOMETRIC_PHASE(ax[a], ay[a],
                cosEl, sinAz, cosAz, k);
        const float2 signal = make_float2(cosf(phase), sinf(phase));

        // Perform complex multiply-accumulate.
        const float2 w = weights[a];
        image[s].x += (signal.x * w.x - signal.y * w.y); // RE*RE - IM*IM
        image[s].y += (signal.y * w.x + signal.x * w.y); // IM*RE + RE*IM
    }
}

__global__
void _beamPattern2(const int na, const float* ax, const float* ay,
        const float2* weights, const int ns, const float* slon, const float* slat,
        const float k, float2* image)
{
    // Get the pixel (source position) ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;
    if (s >= ns) return; // Return if the index is out of range.

    // Get the source position.
    const float az = slon[s];
    const float el = slat[s];
    const float cosEl = cosf(el);
    const float sinAz = sinf(az);
    const float cosAz = cosf(az);

    // Loop over all antennas.
    __shared__ float2 temp[384];
    temp[threadIdx.x] = make_float2(0.0, 0.0);

    for (int a = 0; a < na; ++a) {
        // Calculate the geometric phase from the source.
        const float phase = GEOMETRIC_PHASE(ax[a], ay[a],
                cosEl, sinAz, cosAz, k);
        const float2 signal = make_float2(cosf(phase), sinf(phase));

        // Perform complex multiply-accumulate.
        const float2 w = weights[a];
        temp[threadIdx.x].x += (signal.x * w.x - signal.y * w.y); // RE*RE - IM*IM
        temp[threadIdx.x].y += (signal.y * w.x + signal.x * w.y); // IM*RE + RE*IM
    }

    image[s] = temp[threadIdx.x];
}


__global__
void _beamPattern3(const int na, const float* ax, const float* ay,
        const float cosBeamEl, const float cosBeamAz,  const float sinBeamAz,
        const int ns, const float* slon, const float* slat,
        const float k, float2* image)
{
    // Get the pixel (source position) ID that this thread is working on.
    const int s = blockDim.x * blockIdx.x + threadIdx.x;
    if (s >= ns) return; // Return if the index is out of range.

    // Get the source position.
    const float az = slon[s];
    const float el = slat[s];
    const float cosEl = cosf(el);
    const float sinAz = sinf(az);
    const float cosAz = cosf(az);

    // Loop over all antennas.
    sharedMem[threadIdx.x] = make_float2(0.0, 0.0);

    for (int a = 0; a < na; ++a) {
        // Calculate the geometric phase of the beam direction.
        const float phaseBeam = -GEOMETRIC_PHASE(ax[a], ay[a],
                cosBeamEl, sinBeamAz, cosBeamAz, k);
        float2 w;
        sincosf(phaseBeam, &w.x, &w.y);

        // Calculate the geometric phase from the source.
        const float phase = GEOMETRIC_PHASE(ax[a], ay[a],
                cosEl, sinAz, cosAz, k);
        float2 signal;
        sincosf(phase, &signal.x, &signal.y);

        // Perform complex multiply-accumulate.
        sharedMem[threadIdx.x].x += (signal.x * w.x - signal.y * w.y); // RE*RE - IM*IM
        sharedMem[threadIdx.x].y += (signal.y * w.x + signal.x * w.y); // IM*RE + RE*IM
    }

    image[s].x = sharedMem[threadIdx.x].x / na;
    image[s].y = sharedMem[threadIdx.x].y / na;
}

