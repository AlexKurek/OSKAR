#include "hip/hip_runtime.h"
#include "cuda/beamPattern.h"
#include "cuda/_beamPattern.h"
#include "cuda/_generateWeights.h"

#define TIMER_ENABLE 1
#include "utility/timer.h"

/**
 * @details
 * Computes a beam pattern using CUDA.
 *
 * The function must be supplied with the antenna x- and y-positions, the
 * test source longitude and latitude positions, the beam direction, and
 * the wavenumber.
 *
 * The computed beam pattern is returned in the \p image array, which
 * must be pre-sized to length 2*ns. The values in the \p image array
 * are alternate (real, imag) pairs for each position of the test source.
 *
 * @param[in] na The number of antennas.
 * @param[in] ax The antenna x-positions in metres.
 * @param[in] ay The antenna y-positions in metres.
 * @param[in] ns The number of test source positions.
 * @param[in] slon The longitude coordinates of the test source.
 * @param[in] slat The latitude coordinates of the test source.
 * @param[in] ba The beam azimuth direction in radians
 * @param[in] be The beam elevation direction in radians.
 * @param[in] k The wavenumber (rad / m).
 * @param[out] image The computed beam pattern (see note, above).
 */
void beamPattern(const int na, const float* ax, const float* ay,
        const int ns, const float* slon, const float* slat,
        const float ba, const float be, const float k,
        float* image)
{
    // Precompute.
    float sinBeamAz = sin(ba);
    float cosBeamAz = cos(ba);
    float cosBeamEl = cos(be);

    // Allocate memory for antenna positions, antenna weights,
    // test source positions and pixel values on the device.
    float *axd, *ayd, *slond, *slatd;
    float2 *weights, *pix;
    hipMalloc((void**)&axd, na * sizeof(float));
    hipMalloc((void**)&ayd, na * sizeof(float));
    hipMalloc((void**)&weights, na * sizeof(float2));
    hipMalloc((void**)&slond, ns * sizeof(float));
    hipMalloc((void**)&slatd, ns * sizeof(float));
    hipMalloc((void**)&pix, ns * sizeof(float2));

    // Copy antenna positions and test source positions to device.
    hipMemcpy(axd, ax, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ayd, ay, na * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(slond, slon, ns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(slatd, slat, ns * sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernel to compute antenna weights on the device.
//    int wThreadsPerBlock = 256;
//    int wBlocks = (na + wThreadsPerBlock - 1) / wThreadsPerBlock;
//    _generateWeights <<<wBlocks, wThreadsPerBlock>>> (
//            na, axd, ayd, weights, cosBeamEl, cosBeamAz, sinBeamAz, k);

    // Invoke kernel to compute the beam pattern on the device.
    int threadsPerBlock = 384;
    int blocks = (ns + threadsPerBlock - 1) / threadsPerBlock;
    TIMER_START
//    _beamPattern2 <<<blocks, threadsPerBlock>>> (na, axd, ayd, weights,
//            ns, slond, slatd, k, pix);
    _beamPattern3 <<<blocks, threadsPerBlock, threadsPerBlock * sizeof(float2)>>> (na, axd, ayd, cosBeamEl, cosBeamAz, sinBeamAz,
            ns, slond, slatd, k, pix);
    hipError_t err = hipPeekAtLastError();
    const char* msg = hipGetErrorString(err);
    printf("Error message: %s\n", msg);
    hipDeviceSynchronize();
    TIMER_STOP("Generated beam pattern")

    // Copy result from device memory to host memory.
    hipMemcpy(image, pix, ns * sizeof(float2), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(axd);
    hipFree(ayd);
    hipFree(weights);
    hipFree(slond);
    hipFree(slatd);
    hipFree(pix);
}
