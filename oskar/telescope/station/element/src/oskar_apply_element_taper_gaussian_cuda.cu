#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-2018, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "telescope/station/element/oskar_apply_element_taper_gaussian_cuda.h"


/* Kernels. ================================================================ */

/* Single precision. */
__global__
void oskar_apply_element_taper_gaussian_scalar_cudak_f(const int num_sources,
        const float inv_2sigma_sq, const float* theta, float2* jones)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_sources) return;
    float theta_sq = theta[i];
    theta_sq *= theta_sq;
    const float f = expf(-theta_sq * inv_2sigma_sq);
    jones[i].x *= f;
    jones[i].y *= f;
}

__global__
void oskar_apply_element_taper_gaussian_matrix_cudak_f(const int num_sources,
        const float inv_2sigma_sq, const float* theta, float4c* jones)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_sources) return;
    float theta_sq = theta[i];
    theta_sq *= theta_sq;
    const float f = expf(-theta_sq * inv_2sigma_sq);
    jones[i].a.x *= f;
    jones[i].a.y *= f;
    jones[i].b.x *= f;
    jones[i].b.y *= f;
    jones[i].c.x *= f;
    jones[i].c.y *= f;
    jones[i].d.x *= f;
    jones[i].d.y *= f;
}

/* Double precision. */
__global__
void oskar_apply_element_taper_gaussian_scalar_cudak_d(const int num_sources,
        const double inv_2sigma_sq, const double* theta, double2* jones)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_sources) return;
    double theta_sq = theta[i];
    theta_sq *= theta_sq;
    const double f = exp(-theta_sq * inv_2sigma_sq);
    jones[i].x *= f;
    jones[i].y *= f;
}

__global__
void oskar_apply_element_taper_gaussian_matrix_cudak_d(const int num_sources,
        const double inv_2sigma_sq, const double* theta, double4c* jones)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_sources) return;
    double theta_sq = theta[i];
    theta_sq *= theta_sq;
    const double f = exp(-theta_sq * inv_2sigma_sq);
    jones[i].a.x *= f;
    jones[i].a.y *= f;
    jones[i].b.x *= f;
    jones[i].b.y *= f;
    jones[i].c.x *= f;
    jones[i].c.y *= f;
    jones[i].d.x *= f;
    jones[i].d.y *= f;
}

#ifdef __cplusplus
extern "C" {
#endif

/* Kernel wrappers. ======================================================== */

void oskar_apply_element_taper_gaussian_scalar_cuda_f(int num_sources,
        float inv_2sigma_sq, const float* d_theta, float2* d_jones)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_apply_element_taper_gaussian_scalar_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (
            num_sources, inv_2sigma_sq, d_theta, d_jones);
}

void oskar_apply_element_taper_gaussian_matrix_cuda_f(int num_sources,
        float inv_2sigma_sq, const float* d_theta, float4c* d_jones)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_apply_element_taper_gaussian_matrix_cudak_f
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (
            num_sources, inv_2sigma_sq, d_theta, d_jones);
}

void oskar_apply_element_taper_gaussian_scalar_cuda_d(int num_sources,
        double inv_2sigma_sq, const double* d_theta, double2* d_jones)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_apply_element_taper_gaussian_scalar_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (
            num_sources, inv_2sigma_sq, d_theta, d_jones);
}

void oskar_apply_element_taper_gaussian_matrix_cuda_d(int num_sources,
        double inv_2sigma_sq, const double* d_theta, double4c* d_jones)
{
    int num_blocks, num_threads = 256;
    num_blocks = (num_sources + num_threads - 1) / num_threads;
    oskar_apply_element_taper_gaussian_matrix_cudak_d
    OSKAR_CUDAK_CONF(num_blocks, num_threads) (
            num_sources, inv_2sigma_sq, d_theta, d_jones);
}

#ifdef __cplusplus
}
#endif
