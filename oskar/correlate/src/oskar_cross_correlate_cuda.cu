#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011-2018, The University of Oxford
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 3. Neither the name of the University of Oxford nor the names of its
 *    contributors may be used to endorse or promote products derived from this
 *    software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include "correlate/private_correlate_functions_inline.h"
#include "correlate/oskar_cross_correlate_cuda.h"
#include "math/oskar_add_inline.h"
#include "utility/oskar_device_utils.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstring>

// Indices into the visibility/baseline matrix.
#define SP blockIdx.x /* Column index. */
#define SQ blockIdx.y /* Row index. */

enum { VER_OLD = 1, VER_NON_SM = 2, VER_SM = 3 };
static int ver_ = 0;
static int correlate_version(void);

template
<
// Compile-time parameters.
bool BANDWIDTH_SMEARING, bool TIME_SMEARING, bool GAUSSIAN,
typename REAL, typename REAL2, typename REAL8
>
__global__
void oskar_xcorr_cudak(
        const int                   num_sources,
        const int                   num_stations,
        const REAL8* const restrict jones,
        const REAL*  const restrict source_I,
        const REAL*  const restrict source_Q,
        const REAL*  const restrict source_U,
        const REAL*  const restrict source_V,
        const REAL*  const restrict source_l,
        const REAL*  const restrict source_m,
        const REAL*  const restrict source_n,
        const REAL*  const restrict source_a,
        const REAL*  const restrict source_b,
        const REAL*  const restrict source_c,
        const REAL*  const restrict station_u,
        const REAL*  const restrict station_v,
        const REAL*  const restrict station_w,
        const REAL*  const restrict station_x,
        const REAL*  const restrict station_y,
        const REAL                  uv_min_lambda,
        const REAL                  uv_max_lambda,
        const REAL                  inv_wavelength,
        const REAL                  frac_bandwidth,
        const REAL                  time_int_sec,
        const REAL                  gha0_rad,
        const REAL                  dec0_rad,
        REAL8*             restrict vis)
{
    extern __shared__ __align__(sizeof(double4c)) unsigned char my_smem[];
    __shared__ REAL uv_len, uu, vv, ww, uu2, vv2, uuvv, du, dv, dw;
    REAL8 m1, m2, sum; // Partial sum per thread.
    REAL8* smem = reinterpret_cast<REAL8*>(my_smem); // Allows template.

    // Return immediately if in the wrong half of the visibility matrix.
    if (SQ >= SP) return;

    // Get common baseline values per thread block.
    if (threadIdx.x == 0)
    {
        OSKAR_BASELINE_TERMS(REAL, station_u[SP], station_u[SQ],
                station_v[SP], station_v[SQ], station_w[SP], station_w[SQ],
                uu, vv, ww, uu2, vv2, uuvv, uv_len);

        if (TIME_SMEARING)
            OSKAR_BASELINE_DELTAS(REAL, station_x[SP], station_x[SQ],
                    station_y[SP], station_y[SQ], du, dv, dw);
    }
    __syncthreads();

    // Apply the baseline length filter.
    if (uv_len < uv_min_lambda || uv_len > uv_max_lambda) return;

    // Get pointers to source vectors for both stations.
    const REAL8* const restrict station_p = &jones[num_sources * SP];
    const REAL8* const restrict station_q = &jones[num_sources * SQ];

    // Each thread loops over a subset of the sources.
    OSKAR_CLEAR_COMPLEX_MATRIX(REAL, sum)
    for (int i = threadIdx.x; i < num_sources; i += blockDim.x)
    {
        REAL smearing;
        if (GAUSSIAN)
        {
            const REAL t = source_a[i] * uu2 + source_b[i] * uuvv +
                    source_c[i] * vv2;
            smearing = exp((REAL) -t);
        }
        else
        {
            smearing = (REAL) 1;
        }
        if (BANDWIDTH_SMEARING || TIME_SMEARING)
        {
            const REAL l = source_l[i];
            const REAL m = source_m[i];
            const REAL n = source_n[i] - (REAL) 1;
            if (BANDWIDTH_SMEARING)
            {
                const REAL t = uu * l + vv * m + ww * n;
                smearing *= oskar_sinc<REAL>(t);
            }
            if (TIME_SMEARING)
            {
                const REAL t = du * l + dv * m + dw * n;
                smearing *= oskar_sinc<REAL>(t);
            }
        }

        // Construct source brightness matrix.
        OSKAR_CONSTRUCT_B(REAL, m2,
                source_I[i], source_Q[i], source_U[i], source_V[i])

        // Multiply first Jones matrix with source brightness matrix.
        OSKAR_LOAD_MATRIX(m1, station_p[i])
        OSKAR_MUL_COMPLEX_MATRIX_HERMITIAN_IN_PLACE(REAL2, m1, m2)

        // Multiply result with second (Hermitian transposed) Jones matrix.
        OSKAR_LOAD_MATRIX(m2, station_q[i])
        OSKAR_MUL_COMPLEX_MATRIX_CONJUGATE_TRANSPOSE_IN_PLACE(REAL2, m1, m2)

        // Multiply result by smearing term and accumulate.
        OSKAR_MUL_ADD_COMPLEX_MATRIX_SCALAR(sum, m1, smearing)
    }

    // Store partial sum for the thread in shared memory.
    smem[threadIdx.x] = sum;
    __syncthreads();

    // Accumulate contents of shared memory.
    if (threadIdx.x == 0)
    {
        // Sum over all sources for this baseline.
        for (int i = 1; i < blockDim.x; ++i)
            OSKAR_ADD_COMPLEX_MATRIX_IN_PLACE(sum, smem[i]);

        // Add result of this thread block to the baseline visibility.
        int i = oskar_evaluate_baseline_index_inline(num_stations, SP, SQ);
        OSKAR_ADD_COMPLEX_MATRIX_IN_PLACE(vis[i], sum);
    }
}

#define OKN_NSOURCES 32
#define OKN_BPK 4 /* baselines per kernel */
#define WARP 32

template
<
// Compile-time parameters.
bool BANDWIDTH_SMEARING, bool TIME_SMEARING, bool GAUSSIAN,
typename REAL, typename REAL2, typename REAL8
>
__global__
void oskar_xcorr_NON_SM_cudak(
        const int                   num_sources,
        const int                   num_stations,
        const REAL8* const restrict jones,
        const REAL*  const restrict source_I,
        const REAL*  const restrict source_Q,
        const REAL*  const restrict source_U,
        const REAL*  const restrict source_V,
        const REAL*  const restrict source_l,
        const REAL*  const restrict source_m,
        const REAL*  const restrict source_n,
        const REAL*  const restrict source_a,
        const REAL*  const restrict source_b,
        const REAL*  const restrict source_c,
        const REAL*  const restrict station_u,
        const REAL*  const restrict station_v,
        const REAL*  const restrict station_w,
        const REAL*  const restrict station_x,
        const REAL*  const restrict station_y,
        const REAL                  uv_min_lambda,
        const REAL                  uv_max_lambda,
        const REAL                  inv_wavelength,
        const REAL                  frac_bandwidth,
        const REAL                  time_int_sec,
        const REAL                  gha0_rad,
        const REAL                  dec0_rad,
        REAL8*             restrict vis)
{
    __shared__ REAL uv_len[OKN_BPK], uu[OKN_BPK], vv[OKN_BPK], ww[OKN_BPK];
    __shared__ REAL uu2[OKN_BPK], vv2[OKN_BPK], uuvv[OKN_BPK];
    __shared__ REAL du[OKN_BPK], dv[OKN_BPK], dw[OKN_BPK];
    __shared__ const REAL8 *station_q[OKN_BPK];
    REAL8 m1, m2, sum;

    const int w = (threadIdx.x >> 5); // Warp ID.
    const int i = (threadIdx.x & 31); // ID within warp (local ID).

    // Return immediately if in the wrong half of the visibility matrix.
    if (OKN_BPK * SQ >= SP) return;

    // Get baseline values per warp.
    if (i == 0)
    {
        const int i_sq = OKN_BPK * SQ + w;

        // Set pointer to source vector for station q to safe position
        // so non-existence SQ >= SP does not cause problems.
        station_q[w] = &jones[0];

        if (i_sq < num_stations)
        {
            OSKAR_BASELINE_TERMS(REAL,
                    station_u[SP], station_u[i_sq],
                    station_v[SP], station_v[i_sq],
                    station_w[SP], station_w[i_sq],
                    uu[w], vv[w], ww[w], uu2[w], vv2[w], uuvv[w], uv_len[w]);

            if (TIME_SMEARING)
                OSKAR_BASELINE_DELTAS(REAL,
                        station_x[SP], station_x[i_sq],
                        station_y[SP], station_y[i_sq],
                        du[w], dv[w], dw[w]);

            // Get valid pointer to source vector for station q.
            station_q[w] = &jones[num_sources * i_sq];
        }
    }
    __syncthreads();

    // Get pointer to source vector for station p.
    const REAL8* const restrict station_p = &jones[num_sources * SP];

    // Each thread from given warp loops over a subset of the sources,
    // and each warp works with a different station q.
    OSKAR_CLEAR_COMPLEX_MATRIX(REAL, sum)
    int itemp = (num_sources >> 5) * WARP;
    for (int outer = i; outer < itemp; outer += WARP)
    {
        REAL smearing;
        if (GAUSSIAN)
        {
            const REAL t = source_a[outer] * uu2[w] +
                    source_b[outer] * uuvv[w] + source_c[outer] * vv2[w];
            smearing = exp((REAL) -t);
        }
        else
        {
            smearing = (REAL) 1;
        }
        if (BANDWIDTH_SMEARING || TIME_SMEARING)
        {
            const REAL l = source_l[outer];
            const REAL m = source_m[outer];
            const REAL n = source_n[outer] - (REAL) 1;
            if (BANDWIDTH_SMEARING)
            {
                const REAL t = uu[w] * l + vv[w] * m + ww[w] * n;
                smearing *= oskar_sinc<REAL>(t);
            }
            if (TIME_SMEARING)
            {
                const REAL t = du[w] * l + dv[w] * m + dw[w] * n;
                smearing *= oskar_sinc<REAL>(t);
            }
        }

        // Construct source brightness matrix.
        OSKAR_CONSTRUCT_B(REAL, m2, source_I[outer], source_Q[outer],
                source_U[outer], source_V[outer])

        // Multiply first Jones matrix with source brightness matrix.
        m1 = station_p[outer];
        OSKAR_MUL_COMPLEX_MATRIX_HERMITIAN_IN_PLACE(REAL2, m1, m2)

        // Multiply result with second (Hermitian transposed) Jones matrix.
        OSKAR_LOAD_MATRIX(m2, (station_q[w])[outer])
        OSKAR_MUL_COMPLEX_MATRIX_CONJUGATE_TRANSPOSE_IN_PLACE(REAL2, m1, m2)

        // Multiply result by smearing term and accumulate.
        OSKAR_MUL_ADD_COMPLEX_MATRIX_SCALAR(sum, m1, smearing)
        __syncthreads();
    }
    if ((num_sources & 31) > 0)
    {
        int outer = (num_sources >> 5) * WARP + i;
        if (outer < num_sources)
        {
            REAL smearing;
            if (GAUSSIAN)
            {
                const REAL t = source_a[outer] * uu2[w] +
                        source_b[outer] * uuvv[w] + source_c[outer] * vv2[w];
                smearing = exp((REAL) -t);
            }
            else
            {
                smearing = (REAL) 1;
            }
            if (BANDWIDTH_SMEARING || TIME_SMEARING)
            {
                const REAL l = source_l[outer];
                const REAL m = source_m[outer];
                const REAL n = source_n[outer] - (REAL) 1;
                if (BANDWIDTH_SMEARING)
                {
                    const REAL t = uu[w] * l + vv[w] * m + ww[w] * n;
                    smearing *= oskar_sinc<REAL>(t);
                }
                if (TIME_SMEARING)
                {
                    const REAL t = du[w] * l + dv[w] * m + dw[w] * n;
                    smearing *= oskar_sinc<REAL>(t);
                }
            }

            // Construct source brightness matrix.
            OSKAR_CONSTRUCT_B(REAL, m2, source_I[outer], source_Q[outer],
                    source_U[outer], source_V[outer])

            // Multiply first Jones matrix with source brightness matrix.
            m1 = station_p[outer];
            OSKAR_MUL_COMPLEX_MATRIX_HERMITIAN_IN_PLACE(REAL2, m1, m2)

            // Multiply result with second (Hermitian transposed) Jones matrix.
            OSKAR_LOAD_MATRIX(m2, (station_q[w])[outer])
            OSKAR_MUL_COMPLEX_MATRIX_CONJUGATE_TRANSPOSE_IN_PLACE(REAL2, m1, m2)

            // Multiply result by smearing term and accumulate.
            OSKAR_MUL_ADD_COMPLEX_MATRIX_SCALAR(sum, m1, smearing)
        }
    }

    // Reduce matrices within warp.
    OSKAR_WARP_REDUCE(sum.a.x);
    OSKAR_WARP_REDUCE(sum.a.y);
    OSKAR_WARP_REDUCE(sum.b.x);
    OSKAR_WARP_REDUCE(sum.b.y);
    OSKAR_WARP_REDUCE(sum.c.x);
    OSKAR_WARP_REDUCE(sum.c.y);
    OSKAR_WARP_REDUCE(sum.d.x);
    OSKAR_WARP_REDUCE(sum.d.y);

    // Add result of this warp to the baseline visibility.
    if (i == 0 && (OKN_BPK * SQ + w) < SP)
    {
        if (uv_len[w] < uv_min_lambda || uv_len[w] > uv_max_lambda) return;
        const int j = oskar_evaluate_baseline_index_inline(num_stations,
                SP, OKN_BPK * SQ + w);
        OSKAR_ADD_COMPLEX_MATRIX_IN_PLACE(vis[j], sum);
    }
}

template
<
// Compile-time parameters.
bool BANDWIDTH_SMEARING, bool TIME_SMEARING, bool GAUSSIAN,
typename REAL, typename REAL2, typename REAL8
>
__global__
void oskar_xcorr_SM_cudak(
        const int                   num_sources,
        const int                   num_stations,
        const REAL8* const restrict jones,
        const REAL*  const restrict source_I,
        const REAL*  const restrict source_Q,
        const REAL*  const restrict source_U,
        const REAL*  const restrict source_V,
        const REAL*  const restrict source_l,
        const REAL*  const restrict source_m,
        const REAL*  const restrict source_n,
        const REAL*  const restrict source_a,
        const REAL*  const restrict source_b,
        const REAL*  const restrict source_c,
        const REAL*  const restrict station_u,
        const REAL*  const restrict station_v,
        const REAL*  const restrict station_w,
        const REAL*  const restrict station_x,
        const REAL*  const restrict station_y,
        const REAL                  uv_min_lambda,
        const REAL                  uv_max_lambda,
        const REAL                  inv_wavelength,
        const REAL                  frac_bandwidth,
        const REAL                  time_int_sec,
        const REAL                  gha0_rad,
        const REAL                  dec0_rad,
        REAL8*             restrict vis)
{
    __shared__ REAL uv_len[OKN_BPK], uu[OKN_BPK], vv[OKN_BPK], ww[OKN_BPK];
    __shared__ REAL uu2[OKN_BPK], vv2[OKN_BPK], uuvv[OKN_BPK];
    __shared__ REAL du[OKN_BPK], dv[OKN_BPK], dw[OKN_BPK];
    __shared__ const REAL8 *station_q[OKN_BPK];
    __shared__ REAL   s_I[OKN_NSOURCES];
    __shared__ REAL   s_Q[OKN_NSOURCES];
    __shared__ REAL   s_U[OKN_NSOURCES];
    __shared__ REAL   s_V[OKN_NSOURCES];
    __shared__ REAL   s_l[OKN_NSOURCES];
    __shared__ REAL   s_m[OKN_NSOURCES];
    __shared__ REAL   s_n[OKN_NSOURCES];
    __shared__ REAL   s_a[OKN_NSOURCES];
    __shared__ REAL   s_b[OKN_NSOURCES];
    __shared__ REAL   s_c[OKN_NSOURCES];
    __shared__ REAL8 s_sp[OKN_NSOURCES];
    REAL8 m1, m2, sum;

    const int w = (threadIdx.x >> 5); // Warp ID.
    const int i = (threadIdx.x & 31); // ID within warp (local ID).

    // Return immediately if in the wrong half of the visibility matrix.
    if (OKN_BPK * SQ >= SP) return;

    // Get baseline values per warp.
    if (i == 0)
    {
        const int i_sq = OKN_BPK * SQ + w;

        // Set pointer to source vector for station q to safe position
        // so non-existence SQ >= SP does not cause problems.
        station_q[w] = &jones[0];

        if (i_sq < num_stations)
        {
            OSKAR_BASELINE_TERMS(REAL,
                    station_u[SP], station_u[i_sq],
                    station_v[SP], station_v[i_sq],
                    station_w[SP], station_w[i_sq],
                    uu[w], vv[w], ww[w], uu2[w], vv2[w], uuvv[w], uv_len[w]);

            if (TIME_SMEARING)
                OSKAR_BASELINE_DELTAS(REAL,
                        station_x[SP], station_x[i_sq],
                        station_y[SP], station_y[i_sq],
                        du[w], dv[w], dw[w]);

            // Get valid pointer to source vector for station q.
            station_q[w] = &jones[num_sources * i_sq];
        }
    }
    __syncthreads();

    // Get pointer to source vector for station p.
    const REAL8* const restrict station_p = &jones[num_sources * SP];

    // Each thread from given warp loops over a subset of the sources,
    // and each warp works with a different station q.
    OSKAR_CLEAR_COMPLEX_MATRIX(REAL, sum)
    int itemp = (num_sources >> 5) * WARP;
    for (int outer = i; outer < itemp; outer += WARP)
    {
        if (w == 0)
        {
            s_I[i] = source_I[outer];
            if (BANDWIDTH_SMEARING || TIME_SMEARING)
                s_l[i] = source_l[outer];
            if (GAUSSIAN)
            {
                s_a[i] = source_a[outer];
                s_b[i] = source_b[outer];
            }
        }
        if (w == 1)
        {
            s_Q[i] = source_Q[outer];
            if (BANDWIDTH_SMEARING || TIME_SMEARING)
                s_m[i] = source_m[outer];
            if (GAUSSIAN)
                s_c[i] = source_c[outer];
        }
        if (w == 2)
        {
            s_U[i] = source_U[outer];
            s_V[i] = source_V[outer];
            if (BANDWIDTH_SMEARING || TIME_SMEARING)
                s_n[i] = source_n[outer];
        }
        if (w == 3)
        {
            s_sp[i] = station_p[outer];
        }
        __syncthreads();

        REAL smearing;
        if (GAUSSIAN)
        {
            const REAL t = s_a[i] * uu2[w] +
                    s_b[i] * uuvv[w] + s_c[i] * vv2[w];
            smearing = exp((REAL) -t);
        }
        else
        {
            smearing = (REAL) 1;
        }
        if (BANDWIDTH_SMEARING || TIME_SMEARING)
        {
            const REAL l = s_l[i];
            const REAL m = s_m[i];
            const REAL n = s_n[i] - (REAL) 1;
            if (BANDWIDTH_SMEARING)
            {
                const REAL t = uu[w] * l + vv[w] * m + ww[w] * n;
                smearing *= oskar_sinc<REAL>(t);
            }
            if (TIME_SMEARING)
            {
                const REAL t = du[w] * l + dv[w] * m + dw[w] * n;
                smearing *= oskar_sinc<REAL>(t);
            }
        }

        // Construct source brightness matrix.
        OSKAR_CONSTRUCT_B(REAL, m2, s_I[i], s_Q[i], s_U[i], s_V[i])

        // Multiply first Jones matrix with source brightness matrix.
        m1 = s_sp[i];
        OSKAR_MUL_COMPLEX_MATRIX_HERMITIAN_IN_PLACE(REAL2, m1, m2)

        // Multiply result with second (Hermitian transposed) Jones matrix.
        OSKAR_LOAD_MATRIX(m2, (station_q[w])[outer])
        OSKAR_MUL_COMPLEX_MATRIX_CONJUGATE_TRANSPOSE_IN_PLACE(REAL2, m1, m2)

        // Multiply result by smearing term and accumulate.
        OSKAR_MUL_ADD_COMPLEX_MATRIX_SCALAR(sum, m1, smearing)
        __syncthreads();
    }
    if ((num_sources & 31) > 0)
    {
        int outer = (num_sources >> 5) * WARP + i;
        if (outer < num_sources)
        {
            if (w == 0)
            {
                s_I[i] = source_I[outer];
                if (BANDWIDTH_SMEARING || TIME_SMEARING)
                    s_l[i] = source_l[outer];
                if (GAUSSIAN)
                {
                    s_a[i] = source_a[outer];
                    s_b[i] = source_b[outer];
                }
            }
            if (w == 1)
            {
                s_Q[i] = source_Q[outer];
                if (BANDWIDTH_SMEARING || TIME_SMEARING)
                    s_m[i] = source_m[outer];
                if (GAUSSIAN)
                    s_c[i] = source_c[outer];
            }
            if (w == 2)
            {
                s_U[i] = source_U[outer];
                s_V[i] = source_V[outer];
                if (BANDWIDTH_SMEARING || TIME_SMEARING)
                    s_n[i] = source_n[outer];
            }
            if (w == 3)
            {
                s_sp[i] = station_p[outer];
            }
        }
        __syncthreads();
        if (outer < num_sources)
        {
            REAL smearing;
            if (GAUSSIAN)
            {
                const REAL t = s_a[i] * uu2[w] +
                        s_b[i] * uuvv[w] + s_c[i] * vv2[w];
                smearing = exp((REAL) -t);
            }
            else
            {
                smearing = (REAL) 1;
            }
            if (BANDWIDTH_SMEARING || TIME_SMEARING)
            {
                const REAL l = s_l[i];
                const REAL m = s_m[i];
                const REAL n = s_n[i] - (REAL) 1;
                if (BANDWIDTH_SMEARING)
                {
                    const REAL t = uu[w] * l + vv[w] * m + ww[w] * n;
                    smearing *= oskar_sinc<REAL>(t);
                }
                if (TIME_SMEARING)
                {
                    const REAL t = du[w] * l + dv[w] * m + dw[w] * n;
                    smearing *= oskar_sinc<REAL>(t);
                }
            }

            // Construct source brightness matrix.
            OSKAR_CONSTRUCT_B(REAL, m2, s_I[i], s_Q[i], s_U[i], s_V[i])

            // Multiply first Jones matrix with source brightness matrix.
            m1 = s_sp[i];
            OSKAR_MUL_COMPLEX_MATRIX_HERMITIAN_IN_PLACE(REAL2, m1, m2)

            // Multiply result with second (Hermitian transposed) Jones matrix.
            OSKAR_LOAD_MATRIX(m2, (station_q[w])[outer])
            OSKAR_MUL_COMPLEX_MATRIX_CONJUGATE_TRANSPOSE_IN_PLACE(REAL2, m1, m2)

            // Multiply result by smearing term and accumulate.
            OSKAR_MUL_ADD_COMPLEX_MATRIX_SCALAR(sum, m1, smearing)
        }
    }

    // Reduce matrices within warp.
    OSKAR_WARP_REDUCE(sum.a.x);
    OSKAR_WARP_REDUCE(sum.a.y);
    OSKAR_WARP_REDUCE(sum.b.x);
    OSKAR_WARP_REDUCE(sum.b.y);
    OSKAR_WARP_REDUCE(sum.c.x);
    OSKAR_WARP_REDUCE(sum.c.y);
    OSKAR_WARP_REDUCE(sum.d.x);
    OSKAR_WARP_REDUCE(sum.d.y);

    // Add result of this warp to the baseline visibility.
    if (i == 0 && (OKN_BPK * SQ + w) < SP)
    {
        if (uv_len[w] < uv_min_lambda || uv_len[w] > uv_max_lambda) return;
        const int j = oskar_evaluate_baseline_index_inline(num_stations,
                SP, OKN_BPK * SQ + w);
        OSKAR_ADD_COMPLEX_MATRIX_IN_PLACE(vis[j], sum);
    }
}


void oskar_cross_correlate_point_cuda_f(
        int num_sources, int num_stations, const float4c* d_jones,
        const float* d_I, const float* d_Q,
        const float* d_U, const float* d_V,
        const float* d_l, const float* d_m, const float* d_n,
        const float* d_station_u, const float* d_station_v,
        const float* d_station_w,
        const float* d_station_x, const float* d_station_y,
        float uv_min_lambda, float uv_max_lambda, float inv_wavelength,
        float frac_bandwidth, float time_int_sec, float gha0_rad,
        float dec0_rad, float4c* d_vis)
{
    if (correlate_version() == VER_NON_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_NON_SM_cudak<false, false, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_NON_SM_cudak<true, false, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_NON_SM_cudak<false, true, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_NON_SM_cudak<true, true, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else if (correlate_version() == VER_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_SM_cudak<false, false, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_SM_cudak<true, false, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_SM_cudak<false, true, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_SM_cudak<true, true, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, num_stations);
        const size_t shared_mem = num_threads.x * sizeof(float4c);
        if (frac_bandwidth == 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_cudak<false, false, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_cudak<true, false, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_cudak<false, true, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_cudak<true, true, false, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
}

void oskar_cross_correlate_point_cuda_d(
        int num_sources, int num_stations, const double4c* d_jones,
        const double* d_I, const double* d_Q,
        const double* d_U, const double* d_V,
        const double* d_l, const double* d_m, const double* d_n,
        const double* d_station_u, const double* d_station_v,
        const double* d_station_w,
        const double* d_station_x, const double* d_station_y,
        double uv_min_lambda, double uv_max_lambda, double inv_wavelength,
        double frac_bandwidth, double time_int_sec, double gha0_rad,
        double dec0_rad, double4c* d_vis)
{
    if (correlate_version() == VER_NON_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0 && time_int_sec == 0.0)
            oskar_xcorr_NON_SM_cudak<false, false, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec == 0.0)
            oskar_xcorr_NON_SM_cudak<true, false, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0 && time_int_sec != 0.0)
            oskar_xcorr_NON_SM_cudak<false, true, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec != 0.0)
            oskar_xcorr_NON_SM_cudak<true, true, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else if (correlate_version() == VER_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0 && time_int_sec == 0.0)
            oskar_xcorr_SM_cudak<false, false, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec == 0.0)
            oskar_xcorr_SM_cudak<true, false, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0 && time_int_sec != 0.0)
            oskar_xcorr_SM_cudak<false, true, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec != 0.0)
            oskar_xcorr_SM_cudak<true, true, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, num_stations);
        if (time_int_sec != 0.0) num_threads.x = 64;
        const size_t shared_mem = num_threads.x * sizeof(double4c);
        if (frac_bandwidth == 0.0 && time_int_sec == 0.0)
            oskar_xcorr_cudak<false, false, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec == 0.0)
            oskar_xcorr_cudak<true, false, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0 && time_int_sec != 0.0)
            oskar_xcorr_cudak<false, true, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec != 0.0)
            oskar_xcorr_cudak<true, true, false, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, 0, 0, 0,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
}

void oskar_cross_correlate_gaussian_cuda_f(
        int num_sources, int num_stations, const float4c* d_jones,
        const float* d_I, const float* d_Q,
        const float* d_U, const float* d_V,
        const float* d_l, const float* d_m, const float* d_n,
        const float* d_a, const float* d_b, const float* d_c,
        const float* d_station_u, const float* d_station_v,
        const float* d_station_w, const float* d_station_x,
        const float* d_station_y, float uv_min_lambda, float uv_max_lambda,
        float inv_wavelength, float frac_bandwidth, float time_int_sec,
        float gha0_rad, float dec0_rad, float4c* d_vis)
{
    if (correlate_version() == VER_NON_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_NON_SM_cudak<false, false, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_NON_SM_cudak<true, false, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_NON_SM_cudak<false, true, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_NON_SM_cudak<true, true, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else if (correlate_version() == VER_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_SM_cudak<false, false, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_SM_cudak<true, false, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_SM_cudak<false, true, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_SM_cudak<true, true, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, num_stations);
        const size_t shared_mem = num_threads.x * sizeof(float4c);
        if (frac_bandwidth == 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_cudak<false, false, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec == 0.0f)
            oskar_xcorr_cudak<true, false, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_cudak<false, true, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0f && time_int_sec != 0.0f)
            oskar_xcorr_cudak<true, true, true, float, float2, float4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
}

void oskar_cross_correlate_gaussian_cuda_d(
        int num_sources, int num_stations, const double4c* d_jones,
        const double* d_I, const double* d_Q,
        const double* d_U, const double* d_V,
        const double* d_l, const double* d_m, const double* d_n,
        const double* d_a, const double* d_b, const double* d_c,
        const double* d_station_u, const double* d_station_v,
        const double* d_station_w, const double* d_station_x,
        const double* d_station_y, double uv_min_lambda, double uv_max_lambda,
        double inv_wavelength, double frac_bandwidth, double time_int_sec,
        double gha0_rad, double dec0_rad, double4c* d_vis)
{
    if (correlate_version() == VER_NON_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0 && time_int_sec == 0.0)
            oskar_xcorr_NON_SM_cudak<false, false, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec == 0.0)
            oskar_xcorr_NON_SM_cudak<true, false, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0 && time_int_sec != 0.0)
            oskar_xcorr_NON_SM_cudak<false, true, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec != 0.0)
            oskar_xcorr_NON_SM_cudak<true, true, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else if (correlate_version() == VER_SM)
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, (num_stations + OKN_BPK - 1) / OKN_BPK);
        const size_t shared_mem = 0;
        if (frac_bandwidth == 0.0 && time_int_sec == 0.0)
            oskar_xcorr_SM_cudak<false, false, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec == 0.0)
            oskar_xcorr_SM_cudak<true, false, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0 && time_int_sec != 0.0)
            oskar_xcorr_SM_cudak<false, true, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec != 0.0)
            oskar_xcorr_SM_cudak<true, true, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
    else
    {
        dim3 num_threads(128, 1);
        dim3 num_blocks(num_stations, num_stations);
        if (time_int_sec != 0.0) num_threads.x = 64;
        const size_t shared_mem = num_threads.x * sizeof(double4c);
        if (frac_bandwidth == 0.0 && time_int_sec == 0.0)
            oskar_xcorr_cudak<false, false, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec == 0.0)
            oskar_xcorr_cudak<true, false, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth == 0.0 && time_int_sec != 0.0)
            oskar_xcorr_cudak<false, true, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
        else if (frac_bandwidth != 0.0 && time_int_sec != 0.0)
            oskar_xcorr_cudak<true, true, true, double, double2, double4c>
            OSKAR_CUDAK_CONF(num_blocks, num_threads, shared_mem)
            (num_sources, num_stations, d_jones, d_I, d_Q, d_U, d_V, d_l, d_m, d_n, d_a, d_b, d_c,
                    d_station_u, d_station_v, d_station_w, d_station_x, d_station_y,
                    uv_min_lambda, uv_max_lambda, inv_wavelength,
                    frac_bandwidth, time_int_sec, gha0_rad, dec0_rad, d_vis);
    }
}

int correlate_version()
{
    if (ver_ == 0)
    {
        const char* v = getenv("OSKAR_CORRELATE");
        if (v)
        {
            if (!strcmp(v, "OLD") || !strcmp(v, "old"))
                ver_ = VER_OLD;
            else if (!strcmp(v, "SM") || !strcmp(v, "sm"))
                ver_ = VER_SM;
            else if (strstr(v, "NO") || strstr(v, "no"))
                ver_ = VER_NON_SM;
        }
        if (ver_ == 0)
        {
            const int compute = oskar_device_compute_capability();
            if (compute >= 70)
                ver_ = VER_NON_SM;
            else if (compute >= 30)
                ver_ = VER_SM;
            else
                ver_ = VER_OLD;
        }
    }
    return ver_;
}
